
#include <hip/hip_runtime.h>
//===- kernels.cu ---------------------------------------------*--- C++ -*-===//
//
// Copyright (c) ByteDance Inc. All rights reserved.
// Licensed under the Apache License, Version 2.0
//
//===----------------------------------------------------------------------===//

namespace brt {
namespace cuda {
namespace external_kernels {

template <typename T>
__global__ void add_kernel(const T *input_1, const T *input_2, T *output,
                           int n) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  if (idx < n) {
    output[idx] = input_1[idx] + input_2[idx];
  }
}

// instantiate
template __global__ void add_kernel<float>(const float *, const float *,
                                           float *, int);
template __global__ void add_kernel<int>(const int *, const int *, int *, int);

} // namespace external_kernels
} // namespace cuda
} // namespace brt
