
#include <hip/hip_runtime.h>
//===- test_kernels.cu ----------------------------------------*--- C++ -*-===//
//
// Copyright (c) ByteDance Inc. All rights reserved.
// Licensed under the Apache License, Version 2.0
//
//===----------------------------------------------------------------------===//

namespace brt {
namespace test {
// TODO move this kernel to another separate file
__global__ void test_kernel(const float *input, float *output, int n,
                            float val) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) {
    output[i] = input[i] + val;
  }
}
} // namespace test
} // namespace brt
