#include "hip/hip_runtime.h"
/*
 * Author: Xiaoying Jia, Changyi Wan
 * Project: Faster Transformer Training
 * Department: ByteDance Data-AML
 * Email: {jiaxiaoying, wanchangyi}@bytedance.com
 */
#include "fastertransformer_v4/includes/fuse_attention.h"
#include "fastertransformer_v4/includes/reduce.h"
#include "fastertransformer_v4/includes/utils.h"
using namespace std;

#include <mma.h>
using namespace nvcuda;

namespace fastertransformerv4 {
#define SKEW_HALF 8 // offset for avoding bank conflict
#define WMMA_M 16
#define WMMA_N 16
#define WMMA_K 16

// template<OperationType OpType>
// void Attention<OpType>::set_shared_memory()
// {
//     hipFuncSetAttribute(reinterpret_cast<const void*>(wmma_attention_long_backward_kernel<256), 64>,
//     hipFuncAttributeMaxDynamicSharedMemorySize, 64 * 1024)
// }

template <const int max_seq_len, const int size_per_head>
__global__ void wmma_attention_backward_kernel(
    const __half *grad_out, const __half *softmax_output, const half2 *q,
    const half2 *k, const half2 *v, half2 *grad_q, half2 *grad_k, half2 *grad_v,
    const int seq_len, const half2 scaler, const float scale,
    const uint8_t *dropout_mask, const __half *softmax_dropout_output) {
  extern __shared__ __half base[];
  __half(*s_softmax)[max_seq_len + SKEW_HALF] =
      (__half(*)[max_seq_len + SKEW_HALF]) base;
  __half(*s_grad)[size_per_head + SKEW_HALF] =
      (__half(*)[size_per_head + SKEW_HALF])(
          base + max_seq_len * (max_seq_len + SKEW_HALF));
  __half(*s_qkv)[size_per_head + SKEW_HALF] =
      (__half(*)[size_per_head + SKEW_HALF])(
          base +
          max_seq_len * (max_seq_len + SKEW_HALF + size_per_head + SKEW_HALF));
  __half(*s_softmax_out)[max_seq_len + SKEW_HALF] =
      (__half(*)[max_seq_len + SKEW_HALF])(
          base + max_seq_len * (max_seq_len + SKEW_HALF));

  const int warpNums = (blockDim.x >> 5);
  const int warpId = (threadIdx.x >> 5);
  const int warp_tid = (threadIdx.x & 0x1f);
  const int half_hidden_dim = gridDim.x * (size_per_head / 2);
  const int thread_offset = blockIdx.x * (size_per_head / 2) + warp_tid;
  const int batch_seq_offset = blockIdx.y * seq_len;
  const int from_size = max_seq_len / 16;
  const int to_size = max_seq_len / 16;

  // loading grad_out & s_softmax
  const __half *tmp_ptr =
      scale > 1.0f ? softmax_dropout_output : softmax_output;
  for (int from_id = warpId; from_id < seq_len; from_id += warpNums) {
    int pos = (batch_seq_offset + from_id) * half_hidden_dim + thread_offset;
    int offset = from_id * (size_per_head + SKEW_HALF) + (warp_tid << 1);
    *(__half2 *)(*s_grad + offset) = __ldg(&((const __half2 *)grad_out)[pos]);

    for (int to_id = warp_tid; to_id < seq_len; to_id += 32) {
      int softmax_offset =
          ((blockIdx.y * gridDim.x + blockIdx.x) * seq_len + from_id) *
              seq_len +
          to_id;
      s_softmax[from_id][to_id] = __ldg(&tmp_ptr[softmax_offset]);
    }
  }

  // K dim clear 0
  for (int seq_id = seq_len + warpId; seq_id < max_seq_len;
       seq_id += warpNums) {
    ((float *)(s_grad[seq_id]))[warp_tid] = 0.0f;
    for (int to_id = warp_tid; to_id < (seq_len + 1) / 2; to_id += 32)
      ((float *)(s_softmax[seq_id]))[to_id] = 0.0f;
  }

  __syncthreads();

  // compute s_qkv = s_softmax.T * s_grad
  if (warpId < (from_size << 2)) {
    wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half,
                   wmma::col_major>
        softmax_mat;
    wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half,
                   wmma::row_major>
        gradout_mat;
    wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, half> gradV_mat;
    wmma::fill_fragment(gradV_mat, 0.0f);
    const int warp_from_offset = (warpId >> 2) << 4;
    const int warp_to_offset = (warpId & 0x3) * WMMA_K;

#pragma unroll
    for (int k = 0; k < to_size; k++) {
      wmma::load_matrix_sync(softmax_mat,
                             s_softmax[k * WMMA_K] + warp_from_offset,
                             max_seq_len + SKEW_HALF);
      wmma::load_matrix_sync(gradout_mat, s_grad[k * WMMA_K] + warp_to_offset,
                             size_per_head + SKEW_HALF);
      wmma::mma_sync(gradV_mat, softmax_mat, gradout_mat, gradV_mat);
    }
    wmma::store_matrix_sync(s_qkv[warp_from_offset] + warp_to_offset, gradV_mat,
                            size_per_head + SKEW_HALF, wmma::mem_row_major);
  }

  __syncthreads();

  // saving s_qkv -> grad_V & loading Value -> s_qkv
  for (int from_id = warpId; from_id < seq_len; from_id += warpNums) {
    int pos = (batch_seq_offset + from_id) * half_hidden_dim + thread_offset;
    grad_v[pos] = ((__half2 *)(s_qkv[from_id]))[warp_tid];
    ((__half2 *)(s_qkv[from_id]))[warp_tid] = __ldg(&v[pos]);
  }

  __syncthreads();

  // compute grad_softmax_out = s_grad * v.T
  if (warpId < from_size * to_size) {
    wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half,
                   wmma::row_major>
        gradout_mat;
    wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half,
                   wmma::col_major>
        V_mat;
    wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, half>
        gradsoftmax_mat;
    wmma::fill_fragment(gradsoftmax_mat, 0.0f);
    const int warp_from_offset = (warpId / to_size) << 4;
    const int warp_to_offset = (warpId % to_size) << 4;

#pragma unroll
    for (int k = 0; k < 4; k++) {
      wmma::load_matrix_sync(gradout_mat, s_grad[warp_from_offset] + k * WMMA_K,
                             size_per_head + SKEW_HALF);
      wmma::load_matrix_sync(V_mat, s_qkv[warp_to_offset] + k * WMMA_K,
                             size_per_head + SKEW_HALF);
      wmma::mma_sync(gradsoftmax_mat, gradout_mat, V_mat, gradsoftmax_mat);
    }
    wmma::store_matrix_sync(s_softmax[warp_from_offset] + warp_to_offset,
                            gradsoftmax_mat, max_seq_len + SKEW_HALF,
                            wmma::mem_row_major);
  }

  __syncthreads();

  // softmax
  for (int from_id = warpId; from_id < seq_len; from_id += warpNums) {
    float sum = 0.0f;
    for (int to_id = warp_tid; to_id < seq_len; to_id += 32) {
      int pos_id = ((blockIdx.y * gridDim.x + blockIdx.x) * seq_len + from_id) *
                       seq_len +
                   to_id;
      float softmax_out = (float)__ldg(&softmax_output[pos_id]);
      s_softmax_out[from_id][to_id] = softmax_out;

      float grad_softmax = (float)s_softmax[from_id][to_id];
      // compute dropout grad
      if (scale > 1.0f) {
        grad_softmax = dropout_bw(grad_softmax, scale, pos_id, dropout_mask);
        s_softmax[from_id][to_id] = (__half)grad_softmax;
      }

      sum += softmax_out * grad_softmax;
    }

    // compute softmax grad & scale
    sum = warpReduceSum(sum);
    for (int to_id = warp_tid; to_id < seq_len; to_id += 32)
      s_softmax[from_id][to_id] =
          (__half)(((float)s_softmax[from_id][to_id] - sum) *
                   (float)s_softmax_out[from_id][to_id] * (float)scaler.x);
  }

  // K dim clear 0
  // for(int seq_id = seq_len + warpId; seq_id < max_seq_len; seq_id +=
  // warpNums)
  //     ((float *)(s_qkv[seq_id]))[warp_tid] = 0.0f;

  __syncthreads();

  // loading Query
  for (int seq_id = warpId; seq_id < seq_len; seq_id += warpNums) {
    int pos = (batch_seq_offset + seq_id) * half_hidden_dim + thread_offset;
    int offset = seq_id * (size_per_head + SKEW_HALF) + (warp_tid << 1);
    *(__half2 *)(*s_qkv + offset) = __ldg(&q[pos]);
  }

  // K dim clear 0
  // for(int seq_id = seq_len + warpId; seq_id < max_seq_len; seq_id +=
  // warpNums)
  //     ((float *)(s_qkv[seq_id]))[warp_tid] = 0.0f;

  __syncthreads();

  // compute grad_K = s_grad.T * q
  if (warpId < (from_size << 2)) {
    wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half,
                   wmma::col_major>
        gradout_mat;
    wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half,
                   wmma::row_major>
        Q_mat;
    wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, half> gradK_mat;
    wmma::fill_fragment(gradK_mat, 0.0f);
    const int warp_from_offset = (warpId >> 2) << 4;
    const int warp_to_offset = (warpId & 0x3) * WMMA_K;

#pragma unroll
    for (int k = 0; k < from_size; k++) {
      wmma::load_matrix_sync(gradout_mat,
                             s_softmax[k * WMMA_K] + warp_from_offset,
                             max_seq_len + SKEW_HALF);
      wmma::load_matrix_sync(Q_mat, s_qkv[k * WMMA_K] + warp_to_offset,
                             size_per_head + SKEW_HALF);
      wmma::mma_sync(gradK_mat, gradout_mat, Q_mat, gradK_mat);
    }
    wmma::store_matrix_sync(s_grad[warp_from_offset] + warp_to_offset,
                            gradK_mat, size_per_head + SKEW_HALF,
                            wmma::mem_row_major);
  }

  __syncthreads();

  // saving -> grad_k and loading Key
  for (int from_id = warpId; from_id < seq_len; from_id += warpNums) {
    int pos = (batch_seq_offset + from_id) * half_hidden_dim + thread_offset;
    grad_k[pos] = ((__half2 *)(s_grad[from_id]))[warp_tid];
    ((__half2 *)(s_qkv[from_id]))[warp_tid] = __ldg(&k[pos]);
  }

  // K dim clear 0
  // for(int seq_id = seq_len + warpId; seq_id < max_seq_len; seq_id +=
  // warpNums)
  //     ((float *)(s_qkv[seq_id]))[warp_tid] = 0.0f;

  __syncthreads();

  // compute grad_Q = s_grad * k
  if (warpId < (from_size << 2)) {
    wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half,
                   wmma::row_major>
        gradout_mat;
    wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half,
                   wmma::row_major>
        K_mat;
    wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, half> gradQ_mat;
    wmma::fill_fragment(gradQ_mat, 0.0f);
    const int warp_from_offset = (warpId >> 2) << 4;
    const int warp_to_offset = (warpId & 0x3) * WMMA_K;

#pragma unroll
    for (int k = 0; k < to_size; k++) {
      wmma::load_matrix_sync(gradout_mat,
                             s_softmax[warp_from_offset] + k * WMMA_K,
                             max_seq_len + SKEW_HALF);
      wmma::load_matrix_sync(K_mat, s_qkv[k * WMMA_K] + warp_to_offset,
                             size_per_head + SKEW_HALF);
      wmma::mma_sync(gradQ_mat, gradout_mat, K_mat, gradQ_mat);
    }
    wmma::store_matrix_sync(s_grad[warp_from_offset] + warp_to_offset,
                            gradQ_mat, size_per_head + SKEW_HALF,
                            wmma::mem_row_major);
  }

  __syncthreads();

  // saving grad_Q -> grad_q
  for (int from_id = warpId; from_id < seq_len; from_id += warpNums) {
    int pos = (batch_seq_offset + from_id) * half_hidden_dim + thread_offset;
    grad_q[pos] = ((__half2 *)(s_grad[from_id]))[warp_tid];
  }
}

template <const int max_seq_len, const int size_per_head>
__global__ void wmma_attention_long_backward_kernel(
    const __half *grad_out, const __half *softmax_output, const half2 *q,
    const half2 *k, const half2 *v, half2 *grad_q, half2 *grad_k, half2 *grad_v,
    const int seq_len, const half2 scaler, const float scale,
    const uint8_t *dropout_mask, __half *softmax_dropout_output) {
  // const int split_seq_len = 64;

  // extern __shared__ __half base[];
  // __half (*s_qkv)[size_per_head    + SKEW_HALF] = (__half
  // (*)[size_per_head + SKEW_HALF])base;
  // __half (*s_qkv)[size_per_head + SKEW_HALF] = (__half (*)[size_per_head
  // + SKEW_HALF])(base + (seq_len) * (size_per_head + SKEW_HALF));
  // __half (*s_logits)[max_seq_len  + SKEW_HALF] = (__half (*)[max_seq_len
  // + SKEW_HALF])(base + (split_seq_len + seq_len) * (size_per_head +
  // SKEW_HALF));

  // const int warpNums = (blockDim.x  >> 5);
  // const int warpId   = (threadIdx.x >> 5);
  // const int warp_tid = (threadIdx.x & 0x1f);
  // const int half_hidden_dim = gridDim.x * (size_per_head / 2);
  // const int thread_offset = blockIdx.x * (size_per_head / 2) + warp_tid;
  // const int batch_seq_offset = blockIdx.z * seq_len;

  // load grad_out & transpose
  // for(int from_id = warpId; from_id < split_seq_len; from_id += warpNums)
  // {
  //     int pos = (batch_seq_offset + blockIdx.y * split_seq_len + from_id) *
  //     half_hidden_dim + thread_offset;
  //     ((__half2 *)(attention_output))[pos] = ((__half2
  //     *)(s_qkv[from_id]))[warp_tid];
  // }

  __syncthreads();
}

#define WMMA_ATTENTION_BACKWARD(SEQ_LEN, SIZE_PER_HEAD)                        \
  wmma_attention_backward_kernel<SEQ_LEN, SIZE_PER_HEAD>                       \
      <<<grid, block, shared_memory_size, param.stream>>>(                     \
          (__half *)param.grad_out, (__half *)param.softmax_output, q_ptr,     \
          k_ptr, v_ptr, grad_q_ptr, grad_k_ptr, grad_v_ptr, param.seq_len,     \
          scaler, dropout_scale, param.dropout_mask,                           \
          (__half *)param.softmax_dropout_output)

template <OperationType OpType>
void FuseAttention<OpType>::backward(FuseAttentionBackwardParam param) {
  if (OpType == OperationType::HALF) {
    const half2 *q_ptr = (const half2 *)param.input_q;
    const half2 *k_ptr = (const half2 *)param.input_k;
    const half2 *v_ptr = (const half2 *)param.input_v;
    half2 *grad_q_ptr = (half2 *)param.grad_q;
    half2 *grad_k_ptr = (half2 *)param.grad_k;
    half2 *grad_v_ptr = (half2 *)param.grad_v;

    float scale = 1.0f / sqrtf(param.size_per_head);
    half2 scaler(scale, scale);

    float dropout_scale = 1.0f / (1.0f - param.dropout_rate);
    if (param.seq_len == 128) {
      // todo: set one time
      hipFuncSetAttribute(reinterpret_cast<const void*>(wmma_attention_long_backward_kernel<128), 64>,
                           hipFuncAttributeMaxDynamicSharedMemorySize,
                           64 * 1024);

      const int split_k = 128 / 64;
      dim3 grid(param.head_num, split_k, param.batch_size), block;
      block.x = 32 * (4 * 8);
      wmma_attention_long_backward_kernel<128, 64>
          <<<grid, block, 64 * 1024, param.stream>>>(
              (__half *)param.grad_out, (__half *)param.softmax_output, q_ptr,
              k_ptr, v_ptr, grad_q_ptr, grad_k_ptr, grad_v_ptr, param.seq_len,
              scaler, dropout_scale, param.dropout_mask,
              (__half *)param.softmax_dropout_output);
    } else if (param.seq_len <= 80) {
      int max_seq_len = (param.seq_len + 15) / 16 * 16;
      int softmax_size = max_seq_len * (max_seq_len + SKEW_HALF);
      int shared_memory_size =
          softmax_size +
          max(softmax_size,
              2 * (max_seq_len * (param.size_per_head + SKEW_HALF)));
      shared_memory_size *= sizeof(DataType_);

      dim3 grid(param.head_num, param.batch_size), block;
      block.x = 32 * ((param.seq_len + 15) / 16) *
                max(((param.seq_len + 15) / 16), 64 / 16);
      if (param.seq_len <= 16)
        WMMA_ATTENTION_BACKWARD(16, 64);
      else if (param.seq_len <= 32)
        WMMA_ATTENTION_BACKWARD(32, 64);
      else if (param.seq_len <= 48)
        WMMA_ATTENTION_BACKWARD(48, 64);
      else if (param.seq_len <= 64)
        WMMA_ATTENTION_BACKWARD(64, 64);
      else if (param.seq_len <= 80)
        WMMA_ATTENTION_BACKWARD(80, 64);
    }
  } else {
    printf("FP32 fuse_attention backward op is not supported\n");
  }
}

template void
FuseAttention<OperationType::FP32>::backward(FuseAttentionBackwardParam param);
template void
FuseAttention<OperationType::HALF>::backward(FuseAttentionBackwardParam param);
} // namespace fastertransformerv4