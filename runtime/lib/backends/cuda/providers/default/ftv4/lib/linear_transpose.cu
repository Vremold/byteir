#include "hip/hip_runtime.h"
/*
 * Author: Xiaoying Jia, Changyi Wan
 * Project: Faster Transformer Training
 * Department: ByteDance Data-AML
 * Email: {jiaxiaoying, wanchangyi}@bytedance.com
 */
#include "fastertransformer_v4/includes/gemm.h"
#include "fastertransformer_v4/includes/linear_transpose.h"
#include "fastertransformer_v4/includes/reduce.h"
#include "fastertransformer_v4/includes/utils.h"
using namespace std;

namespace fastertransformerv4 {
template <transposeType transpose_type>
__global__ void add_bias_transpose(const float *input, float *output,
                                   const float *bias) {
  int offset = threadIdx.y * blockDim.x + threadIdx.x;
  int input_offset =
      (blockIdx.y * gridDim.x + blockIdx.x) * (blockDim.y * blockDim.x) +
      offset;

  float result = __ldg(&input[input_offset]) + __ldg(&bias[offset]);

  int out_offset =
      transpose3d<transpose_type>(gridDim.y, gridDim.x, blockDim.y, blockIdx.y,
                                  blockIdx.x, threadIdx.y) *
          blockDim.x +
      threadIdx.x;
  output[out_offset] = result;
}

template <transposeType transpose_type>
__global__ void add_bias_transpose(const __half *input, __half *output,
                                   const __half *bias) {
  int offset = threadIdx.y * blockDim.x + threadIdx.x;
  int input_offset =
      (blockIdx.y * gridDim.x + blockIdx.x) * (blockDim.y * blockDim.x) +
      offset;

  half2 result = __hadd2(__ldg(&((half2 *)input)[input_offset]),
                         __ldg(&((half2 *)bias)[offset]));

  int out_offset =
      transpose3d<transpose_type>(gridDim.y, gridDim.x, blockDim.y, blockIdx.y,
                                  blockIdx.x, threadIdx.y) *
          blockDim.x +
      threadIdx.x;
  ((half2 *)output)[out_offset] = result;
}

template <transposeType transpose_type>
__global__ void
linear_transpose_bw_dbias_sum(const float *dout, float *transpose_grad_in,
                              float *bias_buf, const int batch_size,
                              const int seq_len, const int head_num,
                              const int size_per_head) {
  int head_id = threadIdx.x / size_per_head;
  int id = threadIdx.x % size_per_head;

  float bias_sum = 0.0f;
  for (int row = blockIdx.x; row < batch_size * seq_len; row += gridDim.x) {
    int batch_id = row / seq_len;
    int seq_id = row % seq_len;

    int grad_out_offset =
        transpose3d<transpose_type>(batch_size, seq_len, head_num, batch_id,
                                    seq_id, head_id) *
            size_per_head +
        id;
    float grad_out = __ldg(&dout[grad_out_offset]);

    int grad_in_offset = row * blockDim.x + threadIdx.x;
    transpose_grad_in[grad_in_offset] = grad_out;

    bias_sum += grad_out;
  }
  bias_buf[blockIdx.x * blockDim.x + threadIdx.x] = bias_sum;
}

template <transposeType transpose_type>
__global__ void
linear_transpose_bw_dbias_sum(const __half *dout, __half *transpose_grad_in,
                              float *bias_buf, const int batch_size,
                              const int seq_len, const int head_num,
                              const int half_size_per_head) {
  int head_id = threadIdx.x / half_size_per_head;
  int id = threadIdx.x % half_size_per_head;

  half2 bias_sum(0.0f, 0.0f);
  for (int row = blockIdx.x; row < batch_size * seq_len; row += gridDim.x) {
    int batch_id = row / seq_len;
    int seq_id = row % seq_len;

    int grad_out_offset =
        transpose3d<transpose_type>(batch_size, seq_len, head_num, batch_id,
                                    seq_id, head_id) *
            half_size_per_head +
        id;
    half2 grad_out = __ldg(&((half2 *)dout)[grad_out_offset]);

    int grad_in_offset = row * blockDim.x + threadIdx.x;
    ((half2 *)transpose_grad_in)[grad_in_offset] = grad_out;

    bias_sum = __hadd2(bias_sum, grad_out);
  }
  ((float2 *)bias_buf)[blockIdx.x * blockDim.x + threadIdx.x] =
      __half22float2(bias_sum);
}

template <typename T>
__global__ void linear_transpose_bw_dbias_reduce(const float *bias_buf,
                                                 T *grad_bias, int hidden_dim,
                                                 int block_count) {
  __shared__ float s_bias[32][32 + 1];

  int warp_id = threadIdx.x >> 5;
  int warp_tid = threadIdx.x & 0x1F;

  int offset = blockIdx.x * 32 + warp_tid;
  const float *bias = bias_buf + offset;

  float sum_bias = 0.0f;
  for (int row = warp_id; row < block_count; row += 32)
    sum_bias += *(bias + row * hidden_dim);

  s_bias[warp_tid][warp_id] = sum_bias;

  __syncthreads();

  float d_bias = warpReduceSum(s_bias[warp_id][warp_tid]);

  if (warp_tid == 0)
    grad_bias[blockIdx.x * 32 + warp_id] = (T)d_bias;
}

template <OperationType OpType>
void LinearTranspose<OpType>::forward(LinearTransposeForwardParam param) {
  const int rows = param.batch_size * seq_len_;
  dense_layer_kernel_launcher(
      param.input, param_.weight, (DataType_ *)param.buf, rows,
      from_hidden_dim_, to_hidden_dim_, HIPBLAS_OP_N,
      transposed_weight_ ? HIPBLAS_OP_T : HIPBLAS_OP_N, (DataType_)1.0f,
      (DataType_)0.0f, param.cublas_handle);

  const int hidden_dim =
      (OpType == OperationType::HALF) ? to_hidden_dim_ / 2 : to_hidden_dim_;
  dim3 grid, block;
  grid.y = param.batch_size, grid.x = seq_len_;
  block.y = head_num_, block.x = hidden_dim / head_num_;
  switch (param.transpose_type) {
  case TRANSPOSE0213:
    add_bias_transpose<TRANSPOSE0213><<<grid, block, 0, param.stream>>>(
        (DataType_ *)param.buf, param.output, param_.bias);
    break;
  case TRANSPOSE2013:
    add_bias_transpose<TRANSPOSE2013><<<grid, block, 0, param.stream>>>(
        (DataType_ *)param.buf, param.output, param_.bias);
    break;
  case TRANSPOSE1203:
    add_bias_transpose<TRANSPOSE1203><<<grid, block, 0, param.stream>>>(
        (DataType_ *)param.buf, param.output, param_.bias);
    break;
  }
}

template <OperationType OpType>
void LinearTranspose<OpType>::backward(LinearTransposeBackwardParam param) {
  float *bias_buf = (float *)param.buf;
  DataType_ *transpose_grad_in_buf =
      (DataType_ *)(bias_buf + block_count_ * to_hidden_dim_);

  const int hidden_dim =
      (OpType == OperationType::HALF) ? to_hidden_dim_ / 2 : to_hidden_dim_;
  dim3 grid, block;

  grid.x = block_count_, block.x = hidden_dim;
  switch (param.transpose_type) {
  case TRANSPOSE0213:
    linear_transpose_bw_dbias_sum<TRANSPOSE0213>
        <<<grid, block, 0, param.stream>>>(
            param.grad_out, transpose_grad_in_buf, bias_buf, param.batch_size,
            seq_len_, head_num_, hidden_dim / head_num_);
    break;
  case TRANSPOSE1203:
    linear_transpose_bw_dbias_sum<TRANSPOSE1203>
        <<<grid, block, 0, param.stream>>>(
            param.grad_out, transpose_grad_in_buf, bias_buf, param.batch_size,
            seq_len_, head_num_, hidden_dim / head_num_);
    break;
  case TRANSPOSE2013:
    linear_transpose_bw_dbias_sum<TRANSPOSE2013>
        <<<grid, block, 0, param.stream>>>(
            param.grad_out, transpose_grad_in_buf, bias_buf, param.batch_size,
            seq_len_, head_num_, hidden_dim / head_num_);
    break;
  }

  grid.x = to_hidden_dim_ / 32, block.x = 1024;
  linear_transpose_bw_dbias_reduce<<<grid, block, 0, param.stream>>>(
      bias_buf, param.grad_bias, to_hidden_dim_, block_count_);

  const int rows = param.batch_size * seq_len_;
  dense_layer_kernel_launcher(
      transpose_grad_in_buf, param_.weight, param.grad_in, rows, to_hidden_dim_,
      from_hidden_dim_, HIPBLAS_OP_N,
      transposed_weight_ ? HIPBLAS_OP_N : HIPBLAS_OP_T, (DataType_)1.0f,
      (DataType_)0.0f, param.cublas_handle);

  if (transposed_weight_)
    dense_layer_kernel_launcher(
        transpose_grad_in_buf, param.input, param.grad_weight, to_hidden_dim_,
        rows, from_hidden_dim_, HIPBLAS_OP_T, HIPBLAS_OP_N, (DataType_)1.0f,
        (DataType_)0.0f, param.cublas_handle);
  else
    dense_layer_kernel_launcher(
        param.input, transpose_grad_in_buf, param.grad_weight, from_hidden_dim_,
        rows, to_hidden_dim_, HIPBLAS_OP_T, HIPBLAS_OP_N, (DataType_)1.0f,
        (DataType_)0.0f, param.cublas_handle);
}

template void LinearTranspose<OperationType::FP32>::forward(
    LinearTransposeForwardParam param);
template void LinearTranspose<OperationType::HALF>::forward(
    LinearTransposeForwardParam param);

template void LinearTranspose<OperationType::FP32>::backward(
    LinearTransposeBackwardParam param);
template void LinearTranspose<OperationType::HALF>::backward(
    LinearTransposeBackwardParam param);
} // namespace fastertransformerv4