/*
 * Author: Xiaoying Jia, Changyi Wan
 * Project: Faster Transformer Training
 * Department: ByteDance Data-AML
 * Email: {jiaxiaoying, wanchangyi}@bytedance.com
 */
#include "fastertransformer_v4/includes/gemm.h"
#include "fastertransformer_v4/includes/matmul.h"
using namespace std;

namespace fastertransformerv4 {
template <OperationType OpType>
void MatMul<OpType>::forward(MatMulForwardParam param) {
  hipblasOperation_t trans_A = param.A_T ? HIPBLAS_OP_T : HIPBLAS_OP_N;
  hipblasOperation_t trans_B = param.B_T ? HIPBLAS_OP_T : HIPBLAS_OP_N;

  // if(param.batch_count > 1)
  // {
  cublas_Gemm_Strided_Batched(param.input_A, param.input_B, param.output,
                              param.M, param.K, param.N, param.batch_count,
                              trans_A, trans_B, (DataType_)param.scale,
                              (DataType_)0.0f, param.cublas_handle);
  // }
  // else
  // {
  //     dense_layer_kernel_launcher(
  //         param.input_A, param.input_B, param.output,
  //         param.M, param.K, param.N, trans_A, trans_B,
  //         (DataType_)param.scale, (DataType_)0.0f, param.cublas_handle);
  // }
}

template <OperationType OpType>
void MatMul<OpType>::backward(MatMulBackwardParam param) {
  // hipblasOperation_t trans_A, trans_B;

  // if(param.batch_count > 1)
  // {
  if (param.A_T == false && param.B_T == false) {
    cublas_Gemm_Strided_Batched(
        param.grad_out, param.input_B, param.grad_A, param.M, param.N, param.K,
        param.batch_count, HIPBLAS_OP_N, HIPBLAS_OP_T, (DataType_)param.scale,
        (DataType_)0.0f, param.cublas_handle);

    cublas_Gemm_Strided_Batched(
        param.input_A, param.grad_out, param.grad_B, param.K, param.M, param.N,
        param.batch_count, HIPBLAS_OP_T, HIPBLAS_OP_N, (DataType_)param.scale,
        (DataType_)0.0f, param.cublas_handle);
  } else if (param.A_T == false && param.B_T == true) {
    cublas_Gemm_Strided_Batched(
        param.grad_out, param.input_B, param.grad_A, param.M, param.N, param.K,
        param.batch_count, HIPBLAS_OP_N, HIPBLAS_OP_N, (DataType_)param.scale,
        (DataType_)0.0f, param.cublas_handle);

    cublas_Gemm_Strided_Batched(
        param.grad_out, param.input_A, param.grad_B, param.N, param.M, param.K,
        param.batch_count, HIPBLAS_OP_T, HIPBLAS_OP_N, (DataType_)param.scale,
        (DataType_)0.0f, param.cublas_handle);
  } else if (param.A_T == true && param.B_T == false) {
    cublas_Gemm_Strided_Batched(
        param.input_B, param.grad_out, param.grad_A, param.K, param.N, param.M,
        param.batch_count, HIPBLAS_OP_N, HIPBLAS_OP_T, (DataType_)param.scale,
        (DataType_)0.0f, param.cublas_handle);

    cublas_Gemm_Strided_Batched(
        param.input_A, param.grad_out, param.grad_B, param.K, param.M, param.N,
        param.batch_count, HIPBLAS_OP_N, HIPBLAS_OP_N, (DataType_)param.scale,
        (DataType_)0.0f, param.cublas_handle);
  } else if (param.A_T == true && param.B_T == true) {
    cublas_Gemm_Strided_Batched(
        param.input_B, param.grad_out, param.grad_A, param.K, param.N, param.M,
        param.batch_count, HIPBLAS_OP_T, HIPBLAS_OP_T, (DataType_)param.scale,
        (DataType_)0.0f, param.cublas_handle);

    cublas_Gemm_Strided_Batched(
        param.grad_out, param.input_A, param.grad_B, param.N, param.M, param.K,
        param.batch_count, HIPBLAS_OP_T, HIPBLAS_OP_T, (DataType_)param.scale,
        (DataType_)0.0f, param.cublas_handle);
  }
  // }
  // else
  // {

  // }
}

template void MatMul<OperationType::FP32>::forward(MatMulForwardParam param);
template void MatMul<OperationType::HALF>::forward(MatMulForwardParam param);

template void MatMul<OperationType::FP32>::backward(MatMulBackwardParam param);
template void MatMul<OperationType::HALF>::backward(MatMulBackwardParam param);
} // namespace fastertransformerv4