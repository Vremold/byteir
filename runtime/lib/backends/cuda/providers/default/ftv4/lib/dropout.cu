#include "hip/hip_runtime.h"
/*
 * Author: Xiaoying Jia, Changyi Wan
 * Project: Faster Transformer Training
 * Department: ByteDance Data-AML
 * Email: {jiaxiaoying, wanchangyi}@bytedance.com
 */
#include "fastertransformer_v4/includes/dropout.h"
#include "fastertransformer_v4/includes/utils.h"

using namespace std;

namespace fastertransformerv4 {
template <typename T>
__global__ void dropout_fw_kernel(const T *in, uchar4 *mask, T *out,
                                  const int N, const float ratio,
                                  const int seed) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid < N) {
    float4 in4 = load_vector(in + tid * 4);
    float4 out4 = dropout_fw(in4, ratio, seed, tid, mask);
    store_vector(out + tid * 4, out4);
  }
}

template <typename T>
__global__ void dropout_bw_kernel(const T *in, const uchar4 *mask, T *out,
                                  const int N, const float scale) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid * 4 < N) {
    float4 in4 = load_vector(in + tid * 4);
    float4 out4 = dropout_bw(in4, scale, tid, mask);
    store_vector(out + tid * 4, out4);
  }
}

template <OperationType OpType>
void Dropout<OpType>::forward(DropoutForwardParam param) {
  const int seed = generate_random_seed();
  dim3 grid, block;
  block.x = 1024;
  grid.x = (param.N + (block.x * 4) - 1) / (block.x * 4);
  dropout_fw_kernel<<<grid, block, 0, param.stream>>>(
      param.dropout_in, param.dropout_mask, param.dropout_out, param.N, ratio_,
      seed);
}

template <OperationType OpType>
void Dropout<OpType>::backward(DropoutBackwardParam param) {
  dim3 grid, block;
  block.x = 1024;
  grid.x = (param.N + (block.x * 4) - 1) / (block.x * 4);
  dropout_bw_kernel<<<grid, block, 0, param.stream>>>(
      param.grad_out, param.dropout_mask, param.grad_in, param.N,
      1.0f / (1.0f - ratio_));
}

template void Dropout<OperationType::FP32>::forward(DropoutForwardParam param);
template void Dropout<OperationType::HALF>::forward(DropoutForwardParam param);

template void
Dropout<OperationType::FP32>::backward(DropoutBackwardParam param);
template void
Dropout<OperationType::HALF>::backward(DropoutBackwardParam param);
} // namespace fastertransformerv4