#include "hip/hip_runtime.h"
/*
 * Author: Xiaoying Jia, Changyi Wan
 * Project: Faster Transformer Training
 * Department: ByteDance Data-AML
 * Email: {jiaxiaoying, wanchangyi}@bytedance.com
 */
#include "fastertransformer_v4/includes/reduce.h"
#include "fastertransformer_v4/includes/search_ctr_mha.h"
#include "fastertransformer_v4/includes/utils.h"
using namespace std;

namespace fastertransformerv4 {
template <typename T, const int MAX_SEQ_LEN, const int SIZE_PER_HEAD>
__global__ void fuse_attention_forward_kernel(
    const T *input_q, const T *input_k, const T *input_v, const T *mask,
    T *softmax_out, T *attention_out, const int batch_size, const int seq_len,
    const int head_num, const float scaler) {
  __shared__ T s_buf1[MAX_SEQ_LEN][SIZE_PER_HEAD + 1]; // for grad_out
  __shared__ T s_buf2[MAX_SEQ_LEN][SIZE_PER_HEAD + 1]; // loading q/k/v tensor
  __shared__ T s_buf3[MAX_SEQ_LEN][MAX_SEQ_LEN + 1];   // softmax_out

  int batch_id = blockIdx.x / head_num;
  int head_id = blockIdx.x % head_num;

  // s_buf1 loading Q/K
  for (int tid = threadIdx.x; tid < seq_len * SIZE_PER_HEAD;
       tid += blockDim.x) {
    int seq_id = tid / SIZE_PER_HEAD;
    int dim_id = tid % SIZE_PER_HEAD;

    int id = batch_id * seq_len * head_num * SIZE_PER_HEAD +
             seq_id * head_num * SIZE_PER_HEAD + head_id * SIZE_PER_HEAD +
             dim_id;

    s_buf1[seq_id][dim_id] = __ldg(&input_q[id]);
    s_buf2[seq_id][dim_id] = __ldg(&input_k[id]);
  }
  __syncthreads();

  for (int tid = threadIdx.x; tid < seq_len * seq_len; tid += blockDim.x) {
    int seq_id1 = tid / seq_len;
    int seq_id2 = tid % seq_len;
    T qk_val = 0.0f;
    for (int dim_id = 0; dim_id < SIZE_PER_HEAD; dim_id++)
      qk_val += s_buf1[seq_id1][dim_id] * s_buf2[seq_id2][dim_id];
    qk_val *= scaler;
    qk_val += ((T)1.0f - mask[batch_id * seq_len * seq_len + tid]) * (T)-1e4f;
    s_buf3[seq_id1][seq_id2] = qk_val;
  }
  __syncthreads();
  // s_buf3 qk [seq_len][seq_len]
  // softmax
  const int WARP_SIZE = 32;
  int total_warps = blockDim.x / WARP_SIZE;
  int local_tid = threadIdx.x % WARP_SIZE;
  for (int seq_id = threadIdx.x / WARP_SIZE; seq_id < seq_len;
       seq_id += total_warps) {
    float max_val = -1e20f, exp_sum = 0.0f;
    for (int tid = local_tid; tid < seq_len; tid += WARP_SIZE)
      max_val = (float)s_buf3[seq_id][tid] > max_val
                    ? (float)s_buf3[seq_id][tid]
                    : max_val;
    max_val = warpReduceMax<float>(max_val);
    for (int tid = local_tid; tid < seq_len; tid += WARP_SIZE) {
      float exp_val = __expf((float)s_buf3[seq_id][tid] - max_val);
      s_buf3[seq_id][tid] = exp_val;
      exp_sum += exp_val;
    }

    exp_sum = warpReduceSum<float>(exp_sum) + 1e-6f;
    exp_sum = __fdividef(1.0f, exp_sum);
    for (int tid = local_tid; tid < seq_len; tid += WARP_SIZE) {
      T out_val = (T)((float)s_buf3[seq_id][tid] * exp_sum);
      s_buf3[seq_id][tid] = out_val;
      int id = blockIdx.x * seq_len * seq_len + seq_id * seq_len + tid;
      softmax_out[id] = out_val;
    }
  }
  // s_buf1 loading Q/K/V
  for (int tid = threadIdx.x; tid < seq_len * SIZE_PER_HEAD;
       tid += blockDim.x) {
    int seq_id = tid / SIZE_PER_HEAD;
    int dim_id = tid % SIZE_PER_HEAD;

    int id = batch_id * seq_len * head_num * SIZE_PER_HEAD +
             seq_id * head_num * SIZE_PER_HEAD + head_id * SIZE_PER_HEAD +
             dim_id;
    s_buf1[seq_id][dim_id] = __ldg(&input_v[id]);
  }
  __syncthreads();

  // softmax * V
  // s_buf3 [seq_len][seq_len]
  // s_buf1 [seq_len][size_per_head]
  for (int tid = threadIdx.x; tid < seq_len * SIZE_PER_HEAD;
       tid += blockDim.x) {
    int seq_id = tid / SIZE_PER_HEAD;
    int dim_id = tid % SIZE_PER_HEAD;
    T val = 0.0f;

    for (int id = 0; id < seq_len; id++)
      val += s_buf3[seq_id][id] * s_buf1[id][dim_id];

    int id = batch_id * seq_len * head_num * SIZE_PER_HEAD +
             seq_id * head_num * SIZE_PER_HEAD + head_id * SIZE_PER_HEAD +
             dim_id;
    attention_out[id] = val;
  }
}

// grad_out: [batch_size, seq_len, head_num, size_per_head]
// softmax_out: [batch_size, head_num, seq_len, seq_len]
// input_q/k/v: [batch_size, seq_len, head_num, size_per_head]

template <typename T, const int MAX_SEQ_LEN, const int SIZE_PER_HEAD>
__global__ void fuse_attention_backward_kernel(
    const T *grad_out, const T *softmax_output, const T *input_q,
    const T *input_k, const T *input_v, T *grad_q, T *grad_k, T *grad_v,
    const int batch_size, const int seq_len, const int head_num,
    const float scaler) {
  __shared__ T s_buf1[MAX_SEQ_LEN][SIZE_PER_HEAD + 1]; // for grad_out
  __shared__ T s_buf2[MAX_SEQ_LEN][MAX_SEQ_LEN + 1];   // softmax_out
  __shared__ T s_buf3[MAX_SEQ_LEN][MAX_SEQ_LEN + 1];   // softmax_out_grad
  __shared__ T s_buf4[MAX_SEQ_LEN][SIZE_PER_HEAD + 1]; // loading q/k/v tensor

  int batch_id = blockIdx.x / head_num;
  int head_id = blockIdx.x % head_num;

  // s_buf1 grad out
  for (int tid = threadIdx.x; tid < seq_len * SIZE_PER_HEAD;
       tid += blockDim.x) {
    int seq_id = tid / SIZE_PER_HEAD;
    int dim_id = tid % SIZE_PER_HEAD;

    int grad_id = batch_id * seq_len * head_num * SIZE_PER_HEAD +
                  seq_id * head_num * SIZE_PER_HEAD + head_id * SIZE_PER_HEAD +
                  dim_id;

    s_buf1[seq_id][dim_id] = __ldg(&grad_out[grad_id]);
    s_buf4[seq_id][dim_id] = __ldg(&input_v[grad_id]);
  }

  // s_buf2 softmax_output
  for (int tid = threadIdx.x; tid < seq_len * seq_len; tid += blockDim.x) {
    int seq_id1 = tid / seq_len;
    int seq_id2 = tid % seq_len;
    s_buf2[seq_id1][seq_id2] =
        __ldg(&softmax_output[blockIdx.x * seq_len * seq_len + tid]);
  }
  __syncthreads();

  // test_v_grad = tf.matmul(tf.transpose(logits4, perm=[0, 1, 3, 2]),
  // logits5_grad[0]) compute v_grad [batch_size, head_num, seq_len,
  // size_per_head] [seq_len, seq_len] x [seq_len, size_per_head]
  for (int tid = threadIdx.x; tid < seq_len * SIZE_PER_HEAD;
       tid += blockDim.x) {
    int seq_id = tid / SIZE_PER_HEAD;
    int dim_id = tid % SIZE_PER_HEAD;

    T v_grad_val = (T)0.0f;
    for (int id = 0; id < seq_len; id++) {
      v_grad_val += s_buf2[id][seq_id] * s_buf1[id][dim_id];
    }
    grad_v[batch_id * head_num * seq_len * SIZE_PER_HEAD +
           seq_id * head_num * SIZE_PER_HEAD + head_id * SIZE_PER_HEAD +
           dim_id] = v_grad_val;
  }

  // test_logits4_grad = tf.matmul(logits5_grad[0], tf.transpose(v, perm=[0, 1,
  // 3, 2])) s_buf1 logits5_grad [seq_len][dim] s_buf4 v [seq_len][dim] s_buf3
  // softmax_out_grad [seq_len][seq_len]
  for (int tid = threadIdx.x; tid < seq_len * seq_len; tid += blockDim.x) {
    int seq_id1 = tid / seq_len;
    int seq_id2 = tid % seq_len;

    T grad_val = (T)0.0f;
    for (int id = 0; id < SIZE_PER_HEAD; id++) {
      grad_val += s_buf1[seq_id1][id] * s_buf4[seq_id2][id];
    }
    s_buf3[seq_id1][seq_id2] = grad_val;
  }
  __syncthreads();

  // softmax input grad
  // s_buf3 softmax_output_grad [seq_len][seq_len]
  // s_buf2 softmax_out [seq_len][seq_len]
  // put in s_buf1
  const int WARP_SIZE = 32;
  int total_warps = blockDim.x / WARP_SIZE;
  int local_tid = threadIdx.x % WARP_SIZE;
  for (int seq_id = threadIdx.x / WARP_SIZE; seq_id < seq_len;
       seq_id += total_warps) {
    T sum_val = 0.0f;
    for (int tid = local_tid; tid < seq_len; tid += WARP_SIZE) {
      sum_val += s_buf2[seq_id][tid] * s_buf3[seq_id][tid];
    }
    sum_val = warpReduceSum<float>(sum_val);
    for (int tid = local_tid; tid < seq_len; tid += WARP_SIZE)
      s_buf2[seq_id][tid] =
          s_buf2[seq_id][tid] * (s_buf3[seq_id][tid] - sum_val) * (T)scaler;
  }
  // s_buf1 grad out
  for (int tid = threadIdx.x; tid < seq_len * SIZE_PER_HEAD;
       tid += blockDim.x) {
    int seq_id = tid / SIZE_PER_HEAD;
    int dim_id = tid % SIZE_PER_HEAD;

    int id = batch_id * seq_len * head_num * SIZE_PER_HEAD +
             seq_id * head_num * SIZE_PER_HEAD + head_id * SIZE_PER_HEAD +
             dim_id;

    s_buf1[seq_id][dim_id] = __ldg(&input_q[id]);
    s_buf4[seq_id][dim_id] = __ldg(&input_k[id]);
  }
  __syncthreads();

  // s_buf2 grad_in [seq_len, seq_len]
  // s_buf1 input_q [seq_len, dim]
  // s_buf4 input_k [seq_len, dim]
  for (int tid = threadIdx.x; tid < seq_len * SIZE_PER_HEAD;
       tid += blockDim.x) {
    int seq_id = tid / SIZE_PER_HEAD;
    int dim_id = tid % SIZE_PER_HEAD;

    T grad_q_val = (T)0.0f;
    T grad_k_val = (T)0.0f;
    for (int id = 0; id < seq_len; id++) {
      grad_q_val += s_buf2[seq_id][id] * s_buf4[id][dim_id];
      grad_k_val += s_buf2[id][seq_id] * s_buf1[id][dim_id];
    }

    int grad_id = batch_id * head_num * seq_len * SIZE_PER_HEAD +
                  seq_id * head_num * SIZE_PER_HEAD + head_id * SIZE_PER_HEAD +
                  dim_id;

    grad_k[grad_id] = grad_k_val;
    grad_q[grad_id] = grad_q_val;
  }
}

template <OperationType OpType>
void FuseAttentionCTR<OpType>::forward(FuseAttentionCTRForwardParam param) {
  const int batch_size = param.batch_size;
  const int seq_len = param.seq_len;
  const int head_num = param.head_num;
  // const int size_per_head = param.size_per_head;

  dim3 grid(batch_size * head_num);
  dim3 block(1024);

  fuse_attention_forward_kernel<DataType_, 64, 16>
      <<<grid, block, 0, param.stream>>>(
          param.input_q, param.input_k, param.input_v, param.mask,
          param.softmax_output, param.attention_output, batch_size, seq_len,
          head_num, param.scaler);
}

template <OperationType OpType>
void FuseAttentionCTR<OpType>::backward(FuseAttentionCTRBackwardParam param) {
  const int batch_size = param.batch_size;
  const int seq_len = param.seq_len;
  const int head_num = param.head_num;
  // const int size_per_head = param.size_per_head;

  dim3 grid(batch_size * head_num);
  dim3 block(1024);

  fuse_attention_backward_kernel<DataType_, 64, 16>
      <<<grid, block, 0, param.stream>>>(
          param.grad_out, param.softmax_output, param.input_q, param.input_k,
          param.input_v, param.grad_q, param.grad_k, param.grad_v, batch_size,
          seq_len, head_num, param.scaler);
}

template void FuseAttentionCTR<OperationType::FP32>::forward(
    FuseAttentionCTRForwardParam param);
template void FuseAttentionCTR<OperationType::HALF>::forward(
    FuseAttentionCTRForwardParam param);

template void FuseAttentionCTR<OperationType::FP32>::backward(
    FuseAttentionCTRBackwardParam param);
template void FuseAttentionCTR<OperationType::HALF>::backward(
    FuseAttentionCTRBackwardParam param);
} // namespace fastertransformerv4