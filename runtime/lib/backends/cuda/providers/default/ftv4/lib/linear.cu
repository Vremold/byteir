#include "hip/hip_runtime.h"
/*
 * Author: Xiaoying Jia, Changyi Wan
 * Project: Faster Transformer Training
 * Department: ByteDance Data-AML
 * Email: {jiaxiaoying, wanchangyi}@bytedance.com
 */
#include "fastertransformer_v4/includes/gemm.h"
#include "fastertransformer_v4/includes/linear.h"
#include "fastertransformer_v4/includes/reduce.h"
#include "fastertransformer_v4/includes/utils.h"
using namespace std;

namespace fastertransformerv4 {
template <typename T>
__global__ void add_bias_gelu_dropout(T *input, const T *bias, const int M,
                                      const int N, bool act_gelu,
                                      float dropout_rate, T *bias_out,
                                      uint8_t *dropout_mask, const float ratio,
                                      const int seed) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int offset = tid * 4;

  float4 in = load_vector(input + offset);
  float4 bias4 = load_vector(bias + threadIdx.x * 4);

  in.x += bias4.x;
  in.y += bias4.y;
  in.z += bias4.z;
  in.w += bias4.w;

  if (act_gelu) {
    store_vector(bias_out + offset, in);
    in.x = gelu_fw(in.x);
    in.y = gelu_fw(in.y);
    in.z = gelu_fw(in.z);
    in.w = gelu_fw(in.w);
  }
  if (dropout_rate > 0.0f)
    in = dropout_fw(in, ratio, seed, tid, (uchar4 *)dropout_mask);

  store_vector(input + offset, in);
}

template <typename T>
__global__ void
linear_bw_dropout_gelu_dbias_sum(const T *dout, float *bias_buf, int hidden_dim,
                                 int rows, bool act_gelu, T *bias_out,
                                 uint8_t *dropout_mask, float scale) {
  const T *dout_buf = dout + threadIdx.x * 4;

  float4 bias_sum = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
  for (int row = blockIdx.x; row < rows; row += gridDim.x) {
    int offset = row * hidden_dim;
    float4 dout4 = load_vector(dout_buf + offset);

    if (scale > 1.0f) // apply_dropout
      dout4 = dropout_bw(dout4, scale, offset / 4 + threadIdx.x,
                         (uchar4 *)dropout_mask);

    T *bias_out_buf = bias_out + offset + threadIdx.x * 4;
    if (act_gelu) {
      float4 bias_out4 = load_vector(bias_out_buf);
      dout4.x = gelu_bw(dout4.x, bias_out4.x);
      dout4.y = gelu_bw(dout4.y, bias_out4.y);
      dout4.z = gelu_bw(dout4.z, bias_out4.z);
      dout4.w = gelu_bw(dout4.w, bias_out4.w);
    }

    if (scale > 1.0f || act_gelu)
      store_vector(bias_out_buf, dout4);

    bias_sum.x += dout4.x;
    bias_sum.y += dout4.y;
    bias_sum.z += dout4.z;
    bias_sum.w += dout4.w;
  }

  store_vector(bias_buf + blockIdx.x * hidden_dim + threadIdx.x * 4, bias_sum);
}

template <typename T>
__global__ void linear_bw_dbias_reduce(const float *bias_buf, T *grad_bias,
                                       int hidden_dim, int block_count) {
  __shared__ float s_bias[32][32 + 1];

  int warp_id = threadIdx.x >> 5;
  int warp_tid = threadIdx.x & 0x1F;

  int offset = blockIdx.x * 32 + warp_tid;
  const float *bias = bias_buf + offset;

  float sum_bias = 0.0f;
  for (int row = warp_id; row < block_count; row += 32)
    sum_bias += *(bias + row * hidden_dim);

  s_bias[warp_tid][warp_id] = sum_bias;

  __syncthreads();

  float d_bias = warpReduceSum(s_bias[warp_id][warp_tid]);

  if (warp_tid == 0)
    grad_bias[blockIdx.x * 32 + warp_id] = (T)d_bias;
}

template <OperationType OpType>
void Linear<OpType>::forward(LinearForwardParam param) {
  dense_layer_kernel_launcher(
      param.input, param_.weight, param.output, param.rows, K_, N_, HIPBLAS_OP_N,
      transposed_weight_ ? HIPBLAS_OP_T : HIPBLAS_OP_N, (DataType_)1.0f,
      (DataType_)0.0f, param.cublas_handle);

  const int seed = generate_random_seed();
  add_bias_gelu_dropout<<<param.rows, N_ / 4, 0, param.stream>>>(
      param.output, param_.bias, param.rows, N_, act_gelu_, dropout_rate_,
      param.bias_out, param.dropout_mask, dropout_rate_, seed);
}

template <OperationType OpType>
void Linear<OpType>::backward(LinearBackwardParam param) {
  float *bias_buf = (float *)param.buf;

  dim3 grid, block;
  grid.x = block_count_, block.x = N_ / 4;
  linear_bw_dropout_gelu_dbias_sum<<<grid, block, 0, param.stream>>>(
      param.grad_out, bias_buf, N_, param.rows, act_gelu_, param.bias_out,
      param.dropout_mask, 1.0f / (1.0f - dropout_rate_));

  grid.x = N_ / 32, block.x = 1024;
  linear_bw_dbias_reduce<<<grid, block, 0, param.stream>>>(
      bias_buf, param.grad_bias, N_, block_count_);

  const DataType_ *grad_ptr =
      (act_gelu_ || dropout_rate_ > 0.0f) ? param.bias_out : param.grad_out;
  dense_layer_kernel_launcher(
      grad_ptr, param_.weight, param.grad_in, param.rows, N_, K_, HIPBLAS_OP_N,
      transposed_weight_ ? HIPBLAS_OP_N : HIPBLAS_OP_T, (DataType_)1.0f,
      (DataType_)0.0f, param.cublas_handle);

  if (transposed_weight_)
    dense_layer_kernel_launcher(grad_ptr, param.input, param.grad_weight, N_,
                                param.rows, K_, HIPBLAS_OP_T, HIPBLAS_OP_N,
                                (DataType_)1.0f, (DataType_)0.0f,
                                param.cublas_handle);
  else
    dense_layer_kernel_launcher(param.input, grad_ptr, param.grad_weight, K_,
                                param.rows, N_, HIPBLAS_OP_T, HIPBLAS_OP_N,
                                (DataType_)1.0f, (DataType_)0.0f,
                                param.cublas_handle);
}

template void Linear<OperationType::FP32>::forward(LinearForwardParam param);
template void Linear<OperationType::HALF>::forward(LinearForwardParam param);

template void Linear<OperationType::FP32>::backward(LinearBackwardParam param);
template void Linear<OperationType::HALF>::backward(LinearBackwardParam param);
} // namespace fastertransformerv4