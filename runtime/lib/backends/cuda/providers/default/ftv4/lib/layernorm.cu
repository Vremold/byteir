#include "hip/hip_runtime.h"
/*
 * Author: Xiaoying Jia, Changyi Wan
 * Project: Faster Transformer Training
 * Department: ByteDance Data-AML
 * Email: {jiaxiaoying, wanchangyi}@bytedance.com
 */
#include "fastertransformer_v4/includes/layernorm.h"
#include "fastertransformer_v4/includes/reduce.h"
#include "fastertransformer_v4/includes/utils.h"
using namespace std;

namespace fastertransformerv4 {
#define LN_EPSILON 1e-6f
#define WARP_SIZE 32

template <typename T>
__global__ void layernorm_fw(const T *input, const T *gamma, const T *beta,
                             const T *residual, T *mean_, T *var_rsqrt_,
                             T *layernorm_out, T *input_add_residual,
                             float r_hidden_dim) {
  int offset = (blockIdx.x * blockDim.x + threadIdx.x) * 4;

  float4 in = load_vector(input + offset);
  if (residual) {
    float4 res4 = load_vector(residual + offset);
    in.x += res4.x;
    in.y += res4.y;
    in.z += res4.z;
    in.w += res4.w;
    store_vector(input_add_residual + offset, in);
  }

  __shared__ float s_mean, s_var_rsqrt;
  float mean, var_rsqrt;
  float2 sum2;
  sum2.x = in.x + in.y + in.z + in.w;
  sum2.y = in.x * in.x + in.y * in.y + in.z * in.z + in.w * in.w;
  sum2 = blockReduceSum_2(sum2);
  if (threadIdx.x == 0) {
    mean = sum2.x * r_hidden_dim;
    s_mean = mean;
    float var = sum2.y * r_hidden_dim - mean * mean;
    s_var_rsqrt = rsqrtf(var > LN_EPSILON ? var : LN_EPSILON);
  }
  __syncthreads();
  mean = s_mean, var_rsqrt = s_var_rsqrt;

  // float sum = in.x + in.y + in.z + in.w;
  // sum = blockReduceSum<float>(sum);
  // if(threadIdx.x == 0)
  //     s_mean = sum * r_hidden_dim;
  // __syncthreads();
  // mean = s_mean;

  in.x -= mean;
  in.y -= mean;
  in.z -= mean;
  in.w -= mean;

  // float variance = in.x * in.x + in.y * in.y + in.z * in.z + in.w * in.w;
  // variance = blockReduceSum<float>(variance);
  // if(threadIdx.x == 0)
  //     s_var_rsqrt = rsqrtf(variance * r_hidden_dim + 1e-6f);
  // __syncthreads();
  // var_rsqrt = s_var_rsqrt;

  float4 gamma4 = load_vector(gamma + threadIdx.x * 4);
  float4 beta4 = load_vector(beta + threadIdx.x * 4);
  float4 out;
  out.x = in.x * var_rsqrt * gamma4.x + beta4.x;
  out.y = in.y * var_rsqrt * gamma4.y + beta4.y;
  out.z = in.z * var_rsqrt * gamma4.z + beta4.z;
  out.w = in.w * var_rsqrt * gamma4.w + beta4.w;

  store_vector(layernorm_out + offset, out);
  if (threadIdx.x == 0) {
    mean_[blockIdx.x] = (T)mean;
    var_rsqrt_[blockIdx.x] = (T)var_rsqrt;
  }
}

template <typename T>
__global__ void layernorm_fw_mini_dim(const T *input, const T *gamma_ptr,
                                      const T *beta_ptr, const T *residual,
                                      T *mean_, T *var_rsqrt_, T *layernorm_out,
                                      T *input_add_residual, float r_hidden_dim,
                                      int rows, int hidden_dim) {
  if ((blockIdx.x * blockDim.y + threadIdx.y) >= rows)
    return;
  const int max_warp_per_row = 4;

  register float s_in[max_warp_per_row];
  register float s_in2[max_warp_per_row];
  int offset = (blockIdx.x * blockDim.y + threadIdx.y) * hidden_dim;
  float2 sum2;
  sum2.x = 0.0f;
  sum2.y = 0.0f;
  for (int idx = threadIdx.x, warp_idx = 0; idx < hidden_dim;
       idx += WARP_SIZE, warp_idx++) {
    s_in[warp_idx] = (float)input[offset + idx];
    if (residual) {
      s_in[warp_idx] += (float)residual[offset + idx];
      input_add_residual[offset + idx] = (T)s_in[warp_idx];
    }
    s_in2[warp_idx] = (float)(s_in[warp_idx] * s_in[warp_idx]);
    sum2.x += s_in[warp_idx];
    sum2.y += s_in2[warp_idx];
  }
  __syncwarp();
  sum2 = warpReduceSum_2(sum2);

  float mean = sum2.x * r_hidden_dim;
  float var = sum2.y * r_hidden_dim - mean * mean;
  float var_rsqrt = rsqrtf(var > LN_EPSILON ? var : LN_EPSILON);
#pragma unroll
  for (int idx = threadIdx.x, warp_idx = 0; idx < hidden_dim;
       idx += WARP_SIZE, warp_idx++) {
    register float in = s_in[warp_idx];
    float gamma = gamma_ptr[idx];
    float beta = beta_ptr[idx];
    in -= mean;
    layernorm_out[offset + idx] = in * var_rsqrt * gamma + beta;
  }
  if (threadIdx.x == 0) {
    mean_[blockIdx.x * blockDim.y + threadIdx.y] = (T)mean;
    var_rsqrt_[blockIdx.x * blockDim.y + threadIdx.y] = (T)var_rsqrt;
  }
}

template <typename T>
__global__ void
layernorm_bw_dgamma_dbeta_sum(const T *dout, const T *input, const T *mean_,
                              const T *var_rsqrt_, float *gamma_buf,
                              float *beta_buf, int hidden_dim, int rows) {
  const T *dout_buf = dout + threadIdx.x * 4;
  const T *input_buf = input + threadIdx.x * 4;

  float4 gamma_sum = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
  float4 beta_sum = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
  for (int row = blockIdx.x; row < rows; row += gridDim.x) {
    int offset = row * hidden_dim;
    float4 dout4 = load_vector(dout_buf + offset);
    float4 input4 = load_vector(input_buf + offset);

    const float mean = (float)__ldg(&mean_[row]);
    const float var_rsqrt = (float)__ldg(&var_rsqrt_[row]);

    gamma_sum.x += dout4.x * (input4.x - mean) * var_rsqrt;
    gamma_sum.y += dout4.y * (input4.y - mean) * var_rsqrt;
    gamma_sum.z += dout4.z * (input4.z - mean) * var_rsqrt;
    gamma_sum.w += dout4.w * (input4.w - mean) * var_rsqrt;

    beta_sum.x += dout4.x;
    beta_sum.y += dout4.y;
    beta_sum.z += dout4.z;
    beta_sum.w += dout4.w;
  }

  store_vector(gamma_buf + blockIdx.x * hidden_dim + threadIdx.x * 4,
               gamma_sum);
  store_vector(beta_buf + blockIdx.x * hidden_dim + threadIdx.x * 4, beta_sum);
}

template <typename T>
__global__ void
layernorm_bw_dgamma_dbeta_reduce(const float *gamma_buf, const float *beta_buf,
                                 T *grad_gamma, T *grad_beta, int hidden_dim,
                                 int block_count) {
  __shared__ float s_gamma[32][32 + 1];
  __shared__ float s_beta[32][32 + 1];

  int warp_id = threadIdx.x >> 5;
  int warp_tid = threadIdx.x & 0x1F;

  int offset = blockIdx.x * 32 + warp_tid;
  const float *gamma = gamma_buf + offset;
  const float *beta = beta_buf + offset;

  float sum_gamma = 0.0f, sum_beta = 0.0f;
  for (int row = warp_id; row < block_count; row += 32) {
    sum_gamma += *(gamma + row * hidden_dim);
    sum_beta += *(beta + row * hidden_dim);
  }

  s_gamma[warp_tid][warp_id] = sum_gamma;
  s_beta[warp_tid][warp_id] = sum_beta;

  __syncthreads();

  float2 d_gamma_beta = warpReduceSum_2(
      make_float2(s_gamma[warp_id][warp_tid], s_beta[warp_id][warp_tid]));

  if (warp_tid == 0) {
    grad_gamma[blockIdx.x * 32 + warp_id] = (T)d_gamma_beta.x;
    grad_beta[blockIdx.x * 32 + warp_id] = (T)d_gamma_beta.y;
  }
}

template <typename T>
__global__ void layernorm_bw_dinput(const T *grad_out, const T *gamma,
                                    const T *input, const T *mean_,
                                    const T *var_rsqrt_, T *grad_in,
                                    T *grad_residual, float r_hidden_dim) {
  int offset = (blockIdx.x * blockDim.x + threadIdx.x) * 4;

  float4 xhat4;
  float4 input4 = load_vector(input + offset);
  const float mean = (float)__ldg(&mean_[blockIdx.x]);
  const float var_rsqrt = (float)__ldg(&var_rsqrt_[blockIdx.x]);
  xhat4.x = (input4.x - mean) * var_rsqrt;
  xhat4.y = (input4.y - mean) * var_rsqrt;
  xhat4.z = (input4.z - mean) * var_rsqrt;
  xhat4.w = (input4.w - mean) * var_rsqrt;

  float4 gamma4 = load_vector(gamma + threadIdx.x * 4);
  float4 dxhat = load_vector(grad_out + offset);
  dxhat.x *= gamma4.x;
  dxhat.y *= gamma4.y;
  dxhat.z *= gamma4.z;
  dxhat.w *= gamma4.w;

  float4 dxhat_xhat;
  dxhat_xhat.x = dxhat.x * xhat4.x;
  dxhat_xhat.y = dxhat.y * xhat4.y;
  dxhat_xhat.z = dxhat.z * xhat4.z;
  dxhat_xhat.w = dxhat.w * xhat4.w;

  float2 sum2;
  sum2.x = dxhat.x + dxhat.y + dxhat.z + dxhat.w;
  sum2.y = dxhat_xhat.x + dxhat_xhat.y + dxhat_xhat.z + dxhat_xhat.w;

  sum2 = blockReduceSum_2(sum2);
  __shared__ float s_dxhat_sum, s_dxhat_xhat_sum;
  if (threadIdx.x == 0) {
    s_dxhat_sum = sum2.x;
    s_dxhat_xhat_sum = sum2.y;
  }
  __syncthreads();
  float dxhat_sum = s_dxhat_sum, dxhat_xhat_sum = s_dxhat_xhat_sum;

  float4 tmp;
  tmp.x = (xhat4.x * dxhat_xhat_sum + dxhat_sum) * r_hidden_dim;
  tmp.y = (xhat4.y * dxhat_xhat_sum + dxhat_sum) * r_hidden_dim;
  tmp.z = (xhat4.z * dxhat_xhat_sum + dxhat_sum) * r_hidden_dim;
  tmp.w = (xhat4.w * dxhat_xhat_sum + dxhat_sum) * r_hidden_dim;

  float4 result;
  result.x = (dxhat.x - tmp.x) * var_rsqrt;
  result.y = (dxhat.y - tmp.y) * var_rsqrt;
  result.z = (dxhat.z - tmp.z) * var_rsqrt;
  result.w = (dxhat.w - tmp.w) * var_rsqrt;

  if (grad_residual)
    store_vector(grad_residual + offset, result);

  store_vector(grad_in + offset, result);
}

template <typename T>
__global__ void
layernorm_bw_dinput_mini_dim(const T *grad_out, const T *gamma, const T *input,
                             const T *mean_, const T *var_rsqrt_, T *grad_in,
                             T *grad_residual, float r_hidden_dim, int rows,
                             int hidden_dim) {
  if ((blockIdx.x * blockDim.y + threadIdx.y) >= rows)
    return;
  int row = blockDim.y * blockIdx.x + threadIdx.y;
  int offset = row * hidden_dim;
  const float mean = (float)__ldg(&mean_[row]);
  const float var_rsqrt = (float)__ldg(&var_rsqrt_[row]);

  const int max_warp_per_row = 4;
  float2 sum2;
  sum2.x = 0.0f;
  sum2.y = 0.0f;
  register float s_in[max_warp_per_row];
  register float s_in2[max_warp_per_row];
  for (int idx = threadIdx.x, warp_idx = 0; idx < hidden_dim;
       idx += WARP_SIZE, warp_idx++) {
    float input_num = (float)(input + offset)[idx];
    float xhat = (input_num - mean) * var_rsqrt;
    float dxhat = (float)gamma[idx] * (float)(grad_out + offset)[idx];
    float dxhat_xhat = dxhat * xhat;

    s_in[warp_idx] = xhat;
    s_in2[warp_idx] = dxhat;
    sum2.x += dxhat;
    sum2.y += dxhat_xhat;
  }
  __syncwarp();
  sum2 = warpReduceSum_2(sum2);
  for (int idx = threadIdx.x, warp_idx = 0; idx < hidden_dim;
       idx += WARP_SIZE, warp_idx++) {
    float tmp = (s_in[warp_idx] * sum2.y + sum2.x) * r_hidden_dim;
    float result = (s_in2[warp_idx] - tmp) * var_rsqrt;

    if (grad_residual)
      (grad_residual + offset)[idx] = (T)result;
    (grad_in + offset)[idx] = (T)result;
  }
}

template <OperationType OpType>
void LayerNorm<OpType>::forward(LayerNormForwardParam param) {
  if (hidden_dim_ >= 32 * 4) {
    layernorm_fw<<<param.rows, hidden_dim_ / 4, 0, param.stream>>>(
        param.input, (const DataType_ *)(param_.gamma),
        (const DataType_ *)param_.beta, param.residual, param.mean,
        param.var_rsqrt, param.layernorm_out, param.input_add_residual,
        1.0f / hidden_dim_);
  } else {
    dim3 grid, block;
    grid.x = (param.rows + 32 - 1) / 32, block.x = 32, block.y = 32;
    // one block deal with 32 lines
    // one warp deal with one line
    layernorm_fw_mini_dim<<<grid, block, 0, param.stream>>>(
        param.input, (const DataType_ *)(param_.gamma),
        (const DataType_ *)param_.beta, param.residual, param.mean,
        param.var_rsqrt, param.layernorm_out, param.input_add_residual,
        1.0f / hidden_dim_, param.rows, hidden_dim_);
  }
}

template <OperationType OpType>
void LayerNorm<OpType>::backward(LayerNormBackwardParam param) {
  float *gamma_buf = (float *)param.buf;
  float *beta_buf = gamma_buf + block_count_ * hidden_dim_;
  dim3 grid, block;

  grid.x = block_count_, block.x = hidden_dim_ / 4;
  layernorm_bw_dgamma_dbeta_sum<<<grid, block, 0, param.stream>>>(
      param.grad_out, param.input_add_residual, param.mean, param.var_rsqrt,
      gamma_buf, beta_buf, hidden_dim_, param.rows);

  grid.x = hidden_dim_ / 32, block.x = 1024;
  layernorm_bw_dgamma_dbeta_reduce<<<grid, block, 0, param.stream>>>(
      gamma_buf, beta_buf, param.grad_gamma, param.grad_beta, hidden_dim_,
      block_count_);

  if (hidden_dim_ >= 32 * 4) {
    grid.x = param.rows, block.x = hidden_dim_ / 4;
    layernorm_bw_dinput<<<grid, block, 0, param.stream>>>(
        param.grad_out, (const DataType_ *)param_.gamma,
        param.input_add_residual, param.mean, param.var_rsqrt, param.grad_in,
        param.grad_residual, 1.0f / hidden_dim_);
  } else {
    grid.x = (param.rows + 32 - 1) / 32, block.x = 32, block.y = 32;
    layernorm_bw_dinput_mini_dim<<<grid, block, 0, param.stream>>>(
        param.grad_out, (const DataType_ *)param_.gamma,
        param.input_add_residual, param.mean, param.var_rsqrt, param.grad_in,
        param.grad_residual, 1.0f / hidden_dim_, param.rows, hidden_dim_);
  }
}

template void
LayerNorm<OperationType::FP32>::forward(LayerNormForwardParam param);
template void
LayerNorm<OperationType::HALF>::forward(LayerNormForwardParam param);

template void
LayerNorm<OperationType::FP32>::backward(LayerNormBackwardParam param);
template void
LayerNorm<OperationType::HALF>::backward(LayerNormBackwardParam param);
} // namespace fastertransformerv4