#include "hip/hip_runtime.h"
/*
 * Author: Xiaoying Jia, Changyi Wan
 * Project: Faster Transformer Training
 * Department: ByteDance Data-AML
 * Email: {jiaxiaoying, wanchangyi}@bytedance.com
 */
#include "fastertransformer_v4/includes/remove_padding.h"
#include "fastertransformer_v4/includes/utils.h"
using namespace std;

namespace fastertransformerv4 {
template <typename T> __inline__ __device__ T warpPrefixSum(int id, T count) {
  for (int i = 1; i < 32; i <<= 1) {
    T val = __shfl_up_sync(0xffffffff, count, i);
    if (id >= i)
      count += val;
  }
  return count;
}

template <typename T>
__global__ void parallel_prefix(const T *atten_mask, int *word_idx,
                                const int batch_size, const int seq_len) {
  const int tid = threadIdx.x;
  const int warp_count = blockDim.x >> 5;
  int warp_id = tid >> 5;
  int warp_tid = tid & 0x1F;

  extern __shared__ int base[];
  int *valid_seq_len = base;
  int *seq_offset = base + batch_size;

  for (int wid = warp_id; wid < batch_size; wid += warp_count) {
    int count = 0;
    for (int i = warp_tid; i < (seq_len + 31) / 32 * 32; i += 32) {
      T mask = i < seq_len ? atten_mask[wid * seq_len * seq_len + i] : (T)0.0f;
      count += __popc(__ballot_sync(0xFFFFFFFF, mask > (T)0.5f));
    }
    if (warp_tid == 0)
      valid_seq_len[wid] = count;
  }

  __syncthreads();

  if (warp_id == 0) {
    int offset = 0, temp = 0;
    for (int i = warp_tid; i < ((batch_size + 31) / 32) * 32; i += 32) {
      offset = warp_tid == 0 ? temp : 0;
      int len = i < batch_size ? valid_seq_len[i] : 0;
      temp = warpPrefixSum(warp_tid, offset + len);
      if (i < batch_size)
        seq_offset[i] = temp - len;

      temp = __shfl_sync(0xffffffff, temp, 31);
    }
    if (warp_tid == 0)
      seq_offset[batch_size] = temp;
  }

  __syncthreads();

  const unsigned int t_mask = (1 << warp_tid) - 1;
  for (int wid = warp_id; wid < batch_size; wid += warp_count) {
    int offset = seq_offset[wid];
    // for(int i = warp_tid; i < valid_seq_len[wid]; i += 32)
    //     word_idx[offset + i] = wid * seq_len + i;
    for (int i = warp_tid; i < (seq_len + 31) / 32 * 32; i += 32) {
      T mask = i < seq_len ? __ldg(&atten_mask[wid * seq_len * seq_len + i])
                           : (T)0.0f;
      uint32_t active_mask = __ballot_sync(0xFFFFFFFF, mask > (T)0.5f);
      int seq_pos = __popc(active_mask & t_mask);
      if (mask > (T)0.5f)
        word_idx[offset + seq_pos] = wid * seq_len + i;
      offset += __popc(active_mask);
    }
  }

  // for(int i = tid; i <= batch_size; i += blockDim.x)
  //     batch_idx[i] = seq_offset[i];
  if (tid == 0)
    word_idx[batch_size * seq_len] = seq_offset[batch_size];
}

template <typename T>
__global__ void compress_input(const T *from_tensor, T *to_tensor,
                               const int *word_idx, int hidden_dim,
                               int valid_word_num) {
  int dst_idx = blockIdx.x * hidden_dim + threadIdx.x;
  if (blockIdx.x < valid_word_num) {
    int src_idx = __ldg(&word_idx[blockIdx.x]) * hidden_dim + threadIdx.x;
    ((float4 *)to_tensor)[dst_idx] = ((const float4 *)from_tensor)[src_idx];
  } else
    ((float4 *)to_tensor)[dst_idx] = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
}

template <typename T>
__global__ void restore_input(const T *from_tensor, T *to_tensor,
                              const int *word_idx, int hidden_dim) {
  int src_idx = blockIdx.x * hidden_dim + threadIdx.x;
  int dst_idx = __ldg(&word_idx[blockIdx.x]) * hidden_dim + threadIdx.x;
  ((float4 *)to_tensor)[dst_idx] = ((const float4 *)from_tensor)[src_idx];
}

template <OperationType OpType>
void RemovePadding<OpType>::compress(RemovePaddingParam param) {
  const int hidden_dim =
      (OpType == OperationType::HALF) ? param.hidden_dim / 2 : param.hidden_dim;
  dim3 grid((param.valid_word_num + 7) / 8 * 8);
  dim3 block(hidden_dim / 4);
  compress_input<<<grid, block, 0, param.stream>>>(
      param.input, param.output, param.word_idx, hidden_dim / 4,
      param.valid_word_num);
}

template <OperationType OpType>
void RemovePadding<OpType>::restore(RemovePaddingParam param) {
  hipMemsetAsync(param.output, 0,
                  param.batch_size * param.seq_len * param.hidden_dim *
                      sizeof(DataType_),
                  param.stream);

  const int hidden_dim =
      (OpType == OperationType::HALF) ? param.hidden_dim / 2 : param.hidden_dim;
  dim3 grid(param.valid_word_num);
  dim3 block(hidden_dim / 4);
  restore_input<<<grid, block, 0, param.stream>>>(
      param.input, param.output, param.word_idx, hidden_dim / 4);
}

template <OperationType OpType>
void RemovePadding<OpType>::get_valid_word_index(GetValidWordIndexParam param) {
  dim3 block(std::min(param.batch_size * 32, 1024)); // one warp per sequence
  parallel_prefix<<<1, block, (2 * param.batch_size + 1) * sizeof(int),
                    param.stream>>>(param.attention_mask, param.word_idx,
                                    param.batch_size, param.seq_len);
  hipMemcpyAsync(param.h_valid_word_num_ptr,
                  param.word_idx + param.batch_size * param.seq_len,
                  sizeof(int), hipMemcpyDeviceToHost, param.stream);
}

template void
RemovePadding<OperationType::FP32>::compress(RemovePaddingParam param);
template void
RemovePadding<OperationType::HALF>::compress(RemovePaddingParam param);

template void
RemovePadding<OperationType::FP32>::restore(RemovePaddingParam param);
template void
RemovePadding<OperationType::HALF>::restore(RemovePaddingParam param);

template void RemovePadding<OperationType::FP32>::get_valid_word_index(
    GetValidWordIndexParam param);
template void RemovePadding<OperationType::HALF>::get_valid_word_index(
    GetValidWordIndexParam param);
} // namespace fastertransformerv4