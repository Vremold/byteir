#include "hip/hip_runtime.h"
/*
 * Author: Xiaoying Jia, Changyi Wan
 * Project: Faster Transformer Training
 * Department: ByteDance Data-AML
 * Email: {jiaxiaoying, wanchangyi}@bytedance.com
 */
#include "fastertransformer_v4/includes/fuse_attention.h"
#include "fastertransformer_v4/includes/reduce.h"
#include "fastertransformer_v4/includes/utils.h"
using namespace std;

#include <mma.h>
using namespace nvcuda;

namespace fastertransformerv4 {
#define SKEW_HALF 8 // offset for avoding bank conflict
#define WMMA_M 16
#define WMMA_N 16
#define WMMA_K 16

// template<OperationType OpType>
// void Attention<OpType>::set_shared_memory()
// {
//     hipFuncSetAttribute(reinterpret_cast<const void*>(wmma_attention_long_forward_kernel<256), 64>,
//     hipFuncAttributeMaxDynamicSharedMemorySize, 64 * 1024)
// }

template <const int max_seq_len, const int size_per_head>
__global__ void wmma_attention_forward_kernel(
    const half2 *q, const half2 *k, const half2 *v,
    const __half *attention_mask, __half *softmax_output,
    __half *attention_output, const int seq_len, const half2 scaler,
    const float dropout_rate, const int seed, uint8_t *dropout_mask,
    __half *softmax_dropout_output) {
  __shared__ __half s_kv[max_seq_len][size_per_head + SKEW_HALF];
  __shared__ __half s_query[max_seq_len][size_per_head + SKEW_HALF];
  __shared__ __half s_logits[max_seq_len][max_seq_len + SKEW_HALF];

  const int warpNums = (blockDim.x >> 5);
  const int warpId = (threadIdx.x >> 5);
  const int warp_tid = (threadIdx.x & 0x1f);
  const int half_hidden_dim = gridDim.x * (size_per_head / 2);
  const int thread_offset = blockIdx.x * (size_per_head / 2) + warp_tid;
  const int batch_seq_offset = blockIdx.y * seq_len;
  const int from_size = max_seq_len / 16;
  const int to_size = max_seq_len / 16;

  // loading Query & Key
  for (int seq_id = warpId; seq_id < seq_len; seq_id += warpNums) {
    int pos = (batch_seq_offset + seq_id) * half_hidden_dim + thread_offset;
    int offset = seq_id * (size_per_head + SKEW_HALF) + (warp_tid << 1);
    *(__half2 *)(*s_query + offset) = __hmul2(__ldg(&q[pos]), scaler);
    *(__half2 *)(*s_kv + offset) = __ldg(&k[pos]);
  }

  __syncthreads();

  if (warpId < from_size * to_size) {
    wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half,
                   wmma::row_major>
        Q_mat;
    wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half,
                   wmma::col_major>
        K_mat;
    wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, half> QK_mat;
    wmma::fill_fragment(QK_mat, 0.0f);
    const int warp_from_offset = (warpId / to_size) << 4;
    const int warp_to_offset = (warpId % to_size) << 4;

#pragma unroll
    for (int k = 0; k < 4; k++) {
      wmma::load_matrix_sync(Q_mat, s_query[warp_from_offset] + k * WMMA_K,
                             size_per_head + SKEW_HALF);
      wmma::load_matrix_sync(K_mat, s_kv[warp_to_offset] + k * WMMA_K,
                             size_per_head + SKEW_HALF);
      wmma::mma_sync(QK_mat, Q_mat, K_mat, QK_mat);
    }
    wmma::store_matrix_sync(s_logits[warp_from_offset] + warp_to_offset, QK_mat,
                            max_seq_len + SKEW_HALF, wmma::mem_row_major);
  }

  __syncthreads();

  // softmax
  for (int from_id = warpId; from_id < seq_len; from_id += warpNums) {
    float max_val = -1e20f;

    const int n = (max_seq_len + 31) / 32;
    float logits[n];
    int to_id[n];

#pragma unroll
    for (int i = 0; i < n; i++) {
      to_id[i] = warp_tid + (i << 5);

      float mask =
          to_id[i] < seq_len
              ? (float)__ldg(
                    &attention_mask[(batch_seq_offset + from_id) * seq_len +
                                    to_id[i]])
              : 0.0f;
      mask = (1.0f - mask) * (-10000.0f);

      logits[i] = to_id[i] < seq_len
                      ? (float)(s_logits[from_id][to_id[i]]) + mask
                      : -1e20f;
      max_val = max(max_val, logits[i]);
    }

    max_val = warpReduceMax(max_val);

    float sum_val = 0.0f;
#pragma unroll
    for (int i = 0; i < n; i++) {
      logits[i] = __expf(logits[i] - max_val);
      sum_val += (to_id[i] < seq_len) ? logits[i] : 0.0f;
    }

    sum_val = warpReduceSum(sum_val) + 1e-6f;

#pragma unroll
    for (int i = 0; i < n; i++)
      if (to_id[i] < max_seq_len) {
        __half softmax_result =
            (__half)(to_id[i] < seq_len ? __fdividef(logits[i], sum_val)
                                        : 0.0f);
        if (to_id[i] < seq_len) {
          int offset =
              ((blockIdx.y * gridDim.x + blockIdx.x) * seq_len + from_id) *
                  seq_len +
              to_id[i];
          softmax_output[offset] = softmax_result;
          if (dropout_rate > 0.0f) {
            softmax_result =
                (__half)dropout_fw((float)softmax_result, dropout_rate, seed,
                                   offset, dropout_mask);
            softmax_dropout_output[offset] = softmax_result;
          }
        }
        s_logits[from_id][to_id[i]] = softmax_result;
      }
  }

  // loading Value
  for (int seq_id = warpId; seq_id < seq_len; seq_id += warpNums) {
    int pos = (batch_seq_offset + seq_id) * half_hidden_dim + thread_offset;
    ((__half2 *)(s_kv[seq_id]))[warp_tid] = __ldg(&v[pos]);
  }

  // K dim clear 0
  for (int seq_id = seq_len + warpId; seq_id < max_seq_len; seq_id += warpNums)
    ((float *)(s_kv[seq_id]))[warp_tid] = 0.0f;

  __syncthreads();

  //* V
  if (warpId < (from_size << 2)) {
    wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half,
                   wmma::row_major>
        Logits_mat;
    wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half,
                   wmma::row_major>
        V_mat;
    wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, half> QKV_mat;
    wmma::fill_fragment(QKV_mat, 0.0f);
    const int warp_from_offset = (warpId >> 2) << 4;
    const int warp_to_offset = (warpId & 0x3) * WMMA_K;

#pragma unroll
    for (int k = 0; k < to_size; k++) {
      wmma::load_matrix_sync(Logits_mat,
                             s_logits[warp_from_offset] + k * WMMA_K,
                             max_seq_len + SKEW_HALF);
      wmma::load_matrix_sync(V_mat, s_kv[k * WMMA_K] + warp_to_offset,
                             size_per_head + SKEW_HALF);
      wmma::mma_sync(QKV_mat, Logits_mat, V_mat, QKV_mat);
    }
    wmma::store_matrix_sync(s_query[warp_from_offset] + warp_to_offset, QKV_mat,
                            size_per_head + SKEW_HALF, wmma::mem_row_major);
  }

  __syncthreads();

  for (int from_id = warpId; from_id < seq_len; from_id += warpNums) {
    int pos = (batch_seq_offset + from_id) * half_hidden_dim + thread_offset;
    ((__half2 *)(attention_output))[pos] =
        ((__half2 *)(s_query[from_id]))[warp_tid];
  }
}

template <const int max_seq_len, const int size_per_head>
__global__ void wmma_attention_long_forward_kernel(
    const half2 *q, const half2 *k, const half2 *v,
    const __half *attention_mask, __half *softmax_output,
    __half *attention_output, const int seq_len, const half2 scaler,
    const float dropout_rate, const int seed, uint8_t *dropout_mask,
    __half *softmax_dropout_output) {
  const int split_seq_len = 64;

  extern __shared__ __half base[];
  __half(*s_kv)[size_per_head + SKEW_HALF] =
      (__half(*)[size_per_head + SKEW_HALF]) base;
  __half(*s_query)[size_per_head + SKEW_HALF] =
      (__half(*)[size_per_head + SKEW_HALF])(
          base + (seq_len) * (size_per_head + SKEW_HALF));
  __half(*s_logits)[max_seq_len + SKEW_HALF] =
      (__half(*)[max_seq_len + SKEW_HALF])(
          base + (split_seq_len + seq_len) * (size_per_head + SKEW_HALF));

  const int warpNums = (blockDim.x >> 5);
  const int warpId = (threadIdx.x >> 5);
  const int warp_tid = (threadIdx.x & 0x1f);
  const int half_hidden_dim = gridDim.x * (size_per_head / 2);
  const int thread_offset = blockIdx.x * (size_per_head / 2) + warp_tid;
  const int batch_seq_offset = blockIdx.z * seq_len;

  // loading Query
  for (int seq_id = warpId; seq_id < split_seq_len; seq_id += warpNums) {
    int pos = (batch_seq_offset + blockIdx.y * split_seq_len + seq_id) *
                  half_hidden_dim +
              thread_offset;
    int offset = seq_id * (size_per_head + SKEW_HALF) + (warp_tid << 1);
    *(__half2 *)(*s_query + offset) = __hmul2(__ldg(&q[pos]), scaler);
  }

  // loading Key
  for (int seq_id = warpId; seq_id < seq_len; seq_id += warpNums) {
    int pos = (batch_seq_offset + seq_id) * half_hidden_dim + thread_offset;
    int offset = seq_id * (size_per_head + SKEW_HALF) + (warp_tid << 1);
    *(__half2 *)(*s_kv + offset) = __ldg(&k[pos]);
  }

  __syncthreads();

  const int from_size = split_seq_len / 16;
  const int to_size = max_seq_len / 16;
  if (warpId < from_size * to_size) {
    wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half,
                   wmma::row_major>
        Q_mat;
    wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half,
                   wmma::col_major>
        K_mat;
    wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, half> QK_mat;
    wmma::fill_fragment(QK_mat, 0.0f);
    const int warp_from_offset = (warpId / to_size) << 4;
    const int warp_to_offset = (warpId % to_size) << 4;

#pragma unroll
    for (int k = 0; k < 4; k++) {
      wmma::load_matrix_sync(Q_mat, s_query[warp_from_offset] + k * WMMA_K,
                             size_per_head + SKEW_HALF);
      wmma::load_matrix_sync(K_mat, s_kv[warp_to_offset] + k * WMMA_K,
                             size_per_head + SKEW_HALF);
      wmma::mma_sync(QK_mat, Q_mat, K_mat, QK_mat);
    }
    wmma::store_matrix_sync(s_logits[warp_from_offset] + warp_to_offset, QK_mat,
                            max_seq_len + SKEW_HALF, wmma::mem_row_major);
  }

  __syncthreads();

  // softmax
  for (int from_id = warpId; from_id < split_seq_len; from_id += warpNums) {
    float max_val = -1e20f;

    const int n = (max_seq_len + 31) / 32;
    float logits[n];
    int to_id[n];

#pragma unroll
    for (int i = 0; i < n; i++) {
      to_id[i] = warp_tid + (i << 5);

      float mask =
          to_id[i] < seq_len
              ? (float)__ldg(
                    &attention_mask[(batch_seq_offset +
                                     blockIdx.y * split_seq_len + from_id) *
                                        seq_len +
                                    to_id[i]])
              : 0.0f;
      mask = (1.0f - mask) * (-10000.0f);
      logits[i] = to_id[i] < seq_len
                      ? (float)(s_logits[from_id][to_id[i]]) + mask
                      : -1e20f;

      max_val = max(max_val, logits[i]);
    }

    max_val = warpReduceMax(max_val);

    float sum_val = 0.0f;
#pragma unroll
    for (int i = 0; i < n; i++) {
      logits[i] = __expf(logits[i] - max_val);
      sum_val += (to_id[i] < seq_len) ? logits[i] : 0.0f;
    }

    sum_val = warpReduceSum(sum_val) + 1e-6f;

#pragma unroll
    for (int i = 0; i < n; i++)
      if (to_id[i] < max_seq_len) {
        __half softmax_result =
            (__half)(to_id[i] < seq_len ? __fdividef(logits[i], sum_val)
                                        : 0.0f);
        if (to_id[i] < seq_len) {
          int offset =
              ((blockIdx.y * gridDim.x + blockIdx.x) * seq_len + from_id) *
                  seq_len +
              to_id[i];
          softmax_output[offset] = softmax_result;
          if (dropout_rate > 0.0f) {
            softmax_result =
                (__half)dropout_fw((float)softmax_result, dropout_rate, seed,
                                   offset, dropout_mask);
            softmax_dropout_output[offset] = softmax_result;
          }
        }
        s_logits[from_id][to_id[i]] = softmax_result;
      }
  }

  // loading Value
  for (int seq_id = warpId; seq_id < seq_len; seq_id += warpNums) {
    int pos = (batch_seq_offset + seq_id) * half_hidden_dim + thread_offset;
    ((__half2 *)(s_kv[seq_id]))[warp_tid] = __ldg(&v[pos]);
  }

  // K dim clear 0
  for (int seq_id = seq_len + warpId; seq_id < max_seq_len; seq_id += warpNums)
    ((float *)(s_kv[seq_id]))[warp_tid] = 0.0f;

  __syncthreads();

  //* V
  if (warpId < (from_size << 2)) {
    wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half,
                   wmma::row_major>
        Logits_mat;
    wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half,
                   wmma::row_major>
        V_mat;
    wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, half> QKV_mat;
    wmma::fill_fragment(QKV_mat, 0.0f);
    const int warp_from_offset = (warpId >> 2) << 4;
    const int warp_to_offset = (warpId & 0x3) * WMMA_K;

#pragma unroll
    for (int k = 0; k < to_size; k++) {
      wmma::load_matrix_sync(Logits_mat,
                             s_logits[warp_from_offset] + k * WMMA_K,
                             max_seq_len + SKEW_HALF);
      wmma::load_matrix_sync(V_mat, s_kv[k * WMMA_K] + warp_to_offset,
                             size_per_head + SKEW_HALF);
      wmma::mma_sync(QKV_mat, Logits_mat, V_mat, QKV_mat);
    }
    wmma::store_matrix_sync(s_query[warp_from_offset] + warp_to_offset, QKV_mat,
                            size_per_head + SKEW_HALF, wmma::mem_row_major);
  }

  __syncthreads();

  for (int from_id = warpId; from_id < split_seq_len; from_id += warpNums) {
    int pos = (batch_seq_offset + blockIdx.y * split_seq_len + from_id) *
                  half_hidden_dim +
              thread_offset;
    ((__half2 *)(attention_output))[pos] =
        ((__half2 *)(s_query[from_id]))[warp_tid];
  }
}

#define WMMA_ATTENTION_FORWARD(SEQ_LEN, SIZE_PER_HEAD)                         \
  wmma_attention_forward_kernel<SEQ_LEN, SIZE_PER_HEAD>                        \
      <<<grid, block, 0, param.stream>>>(                                      \
          q_ptr, k_ptr, v_ptr, (__half *)param.mask,                           \
          (__half *)param.softmax_output, (__half *)param.attention_output,    \
          param.seq_len, scaler, param.dropout_rate, seed, param.dropout_mask, \
          (__half *)param.softmax_dropout_output)

template <OperationType OpType>
void FuseAttention<OpType>::forward(FuseAttentionForwardParam param) {
  if (OpType == OperationType::HALF) {
    const half2 *q_ptr = (const half2 *)param.input_q;
    const half2 *k_ptr = (const half2 *)param.input_k;
    const half2 *v_ptr = (const half2 *)param.input_v;

    const float scale = 1.0f / sqrtf(param.size_per_head);
    const half2 scaler(scale, scale);

    const int seed = param.dropout_rate > 0.0f ? generate_random_seed() : 0;

    if (param.seq_len == 128) {
      // todo: set one time
      hipFuncSetAttribute(reinterpret_cast<const void*>(wmma_attention_long_forward_kernel<128), 64>,
                           hipFuncAttributeMaxDynamicSharedMemorySize,
                           64 * 1024);

      const int split_k = 128 / 64;
      dim3 grid(param.head_num, split_k, param.batch_size), block;
      block.x = 32 * (4 * 8);
      wmma_attention_long_forward_kernel<128, 64>
          <<<grid, block, 64 * 1024, param.stream>>>(
              q_ptr, k_ptr, v_ptr, (__half *)param.mask,
              (__half *)param.softmax_output, (__half *)param.attention_output,
              param.seq_len, scaler, param.dropout_rate, seed,
              param.dropout_mask, (__half *)param.softmax_dropout_output);
    } else if (param.seq_len <= 80) {
      dim3 grid(param.head_num, param.batch_size), block;
      block.x = 32 * ((param.seq_len + 15) / 16) *
                max(((param.seq_len + 15) / 16), 64 / 16);
      if (param.seq_len <= 16)
        WMMA_ATTENTION_FORWARD(16, 64);
      else if (param.seq_len <= 32)
        WMMA_ATTENTION_FORWARD(32, 64);
      else if (param.seq_len <= 48)
        WMMA_ATTENTION_FORWARD(48, 64);
      else if (param.seq_len <= 64)
        WMMA_ATTENTION_FORWARD(64, 64);
      else if (param.seq_len <= 80)
        WMMA_ATTENTION_FORWARD(80, 64);
    }
  } else {
    printf("FP32 fuse_attention forward op is not supported\n");
  }
}

template void
FuseAttention<OperationType::FP32>::forward(FuseAttentionForwardParam param);
template void
FuseAttention<OperationType::HALF>::forward(FuseAttentionForwardParam param);
} // namespace fastertransformerv4