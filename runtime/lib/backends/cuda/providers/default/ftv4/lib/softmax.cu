#include "hip/hip_runtime.h"
/*
 * Author: Xiaoying Jia, Changyi Wan
 * Project: Faster Transformer Training
 * Department: ByteDance Data-AML
 * Email: {jiaxiaoying, wanchangyi}@bytedance.com
 */
#include "fastertransformer_v4/includes/reduce.h"
#include "fastertransformer_v4/includes/softmax.h"
#include "fastertransformer_v4/includes/utils.h"
using namespace std;

namespace fastertransformerv4 {
const int WARP_SIZE = 32;

template <typename T>
__global__ void
softmax_forward_kernel(const T *input, T *softmax_output, const int rows,
                       const int cols, const bool add_mask, const T *mask,
                       const bool batch_first, const bool apply_dropout,
                       const float dropout_rate, uint8_t *dropout_mask,
                       T *softmax_dropout_output, const int seed) {
  extern __shared__ float shmem[];
  float *s_local = shmem + (threadIdx.z * blockDim.y + threadIdx.y) * cols;

  int local_tid = threadIdx.x;
  int bs_id = blockIdx.x * blockDim.z + threadIdx.z;

  int row_id;
  if (batch_first)
    row_id = (bs_id / cols * blockDim.y + threadIdx.y) * cols + (bs_id % cols);
  else
    row_id = threadIdx.y * (gridDim.x * blockDim.z) + bs_id;

  if (row_id < rows) {
    float max_val = -1e20f, exp_sum = 0.0f;
    for (int col_id = local_tid; col_id < cols; col_id += WARP_SIZE) {
      int pos_id = row_id * cols + col_id;
      float value = (float)__ldg(&input[pos_id]);
      if (add_mask)
        value +=
            (1.0f - (float)__ldg(&mask[bs_id * cols + col_id])) * -10000.0f;
      s_local[col_id] = value;
      max_val = value > max_val ? value : max_val;
    }
    max_val = warpReduceMax<float>(max_val);

    for (int col_id = local_tid; col_id < cols; col_id += WARP_SIZE) {
      float exp_val = __expf(s_local[col_id] - max_val);
      s_local[col_id] = exp_val;
      exp_sum += exp_val;
    }
    exp_sum = warpReduceSum<float>(exp_sum) + 1e-6f;
    exp_sum = __fdividef(1.0f, exp_sum);
    for (int col_id = local_tid; col_id < cols; col_id += WARP_SIZE) {
      int pos_id = row_id * cols + col_id;
      T out_val = (T)(s_local[col_id] * exp_sum);
      softmax_output[pos_id] = out_val;
      if (apply_dropout) {
        out_val = dropout_fw(out_val, dropout_rate, seed, pos_id, dropout_mask);
        softmax_dropout_output[pos_id] = out_val;
      }
    }
  }
}

template <>
__global__ void softmax_forward_kernel(
    const __half *input, __half *softmax_output, const int rows, const int cols,
    const bool add_mask, const __half *mask, const bool batch_first,
    const bool apply_dropout, const float dropout_rate, uint8_t *dropout_mask,
    __half *softmax_dropout_output, const int seed) {
  extern __shared__ float shmem[];
  float *s_local = shmem + (threadIdx.z * blockDim.y + threadIdx.y) * cols;

  int half_cols = cols / 2;
  const half2 *input_ptr = (const half2 *)(input);
  const half2 *mask_ptr = (const half2 *)(mask);
  half2 *softmax_output_ptr = (half2 *)(softmax_output);
  half2 *softmax_dropout_output_ptr = (half2 *)(softmax_dropout_output);

  int local_tid = threadIdx.x;
  int bs_id = blockIdx.x * blockDim.z + threadIdx.z;
  int row_id;
  if (batch_first)
    row_id = (bs_id / cols * blockDim.y + threadIdx.y) * cols + (bs_id % cols);
  else
    row_id = threadIdx.y * (gridDim.x * blockDim.z) + bs_id;

  if (row_id < rows) {
    float max_val = -1e20f, exp_sum = 0.0f;
    for (int col_id = local_tid; col_id < half_cols; col_id += WARP_SIZE) {
      int pos_id = row_id * half_cols + col_id;
      float2 value = __half22float2(__ldg(&input_ptr[pos_id]));
      if (add_mask) {
        float2 mask_val =
            __half22float2(__ldg(&mask_ptr[bs_id * half_cols + col_id]));
        value.x += (1.0f - mask_val.x) * -10000.0f;
        value.y += (1.0f - mask_val.y) * -10000.0f;
      }

      s_local[col_id * 2] = value.x;
      s_local[col_id * 2 + 1] = value.y;
      max_val = value.x > max_val ? value.x : max_val;
      max_val = value.y > max_val ? value.y : max_val;
    }
    max_val = warpReduceMax<float>(max_val);

    for (int col_id = local_tid; col_id < cols; col_id += WARP_SIZE) {
      float exp_val = __expf(s_local[col_id] - max_val);
      s_local[col_id] = exp_val;
      exp_sum += exp_val;
    }
    exp_sum = warpReduceSum<float>(exp_sum) + 1e-6f;
    exp_sum = __fdividef(1.0f, exp_sum);
    for (int col_id = local_tid; col_id < half_cols; col_id += WARP_SIZE) {
      int pos_id = row_id * half_cols + col_id;

      half2 val;
      val.x = s_local[col_id * 2] * exp_sum;
      val.y = s_local[col_id * 2 + 1] * exp_sum;
      softmax_output_ptr[pos_id] = val;
      if (apply_dropout) {
        val =
            dropout_fw(val, dropout_rate, seed, pos_id, (uchar2 *)dropout_mask);
        softmax_dropout_output_ptr[pos_id] = val;
      }
    }
  }
}

template <typename T>
__global__ void
softmax_backward_kernel(const T *grad_out, const T *softmax_out, T *grad_in,
                        const int rows, const int cols,
                        const bool apply_dropout, const float scale,
                        const uint8_t *dropout_mask) {
  int total_warps = gridDim.x * blockDim.x / WARP_SIZE;
  int warp_id = (threadIdx.x + blockIdx.x * blockDim.x) / WARP_SIZE;
  int local_tid = threadIdx.x % WARP_SIZE;

  extern __shared__ unsigned char smem[];
  T *s_out = (reinterpret_cast<T *>(smem)) + threadIdx.x / WARP_SIZE * cols;
  T *s_grad = (reinterpret_cast<T *>(smem)) + threadIdx.x / WARP_SIZE * cols +
              blockDim.x / WARP_SIZE * cols;

  for (int row_id = warp_id; row_id < rows; row_id += total_warps) {
    T sum = (T)(0.0f);
    for (int col_id = local_tid; col_id < cols; col_id += WARP_SIZE) {
      int pos_id = row_id * cols + col_id;
      T out = __ldg(&softmax_out[pos_id]);
      T grad = __ldg(&grad_out[pos_id]);
      if (apply_dropout)
        grad = dropout_mask[pos_id] ? grad * scale : 0.0f;

      sum += out * grad;
      s_out[col_id] = out;
      s_grad[col_id] = grad;
    }
    sum = warpReduceSum<float>(sum);

    for (int col_id = local_tid; col_id < cols; col_id += WARP_SIZE)
      grad_in[row_id * cols + col_id] = s_out[col_id] * (s_grad[col_id] - sum);
  }
}

template <>
__global__ void
softmax_backward_kernel(const __half *grad_out, const __half *softmax_out,
                        __half *grad_in, const int rows, const int cols,
                        const bool apply_dropout, const float scale,
                        const uint8_t *dropout_mask) {
  int total_warps = gridDim.x * blockDim.x / WARP_SIZE;
  int warp_id = (threadIdx.x + blockIdx.x * blockDim.x) / WARP_SIZE;
  int local_tid = threadIdx.x % WARP_SIZE;

  extern __shared__ unsigned char smem[];

  int half2_cols = cols / 2;
  half2 *s_out =
      (reinterpret_cast<half2 *>(smem)) + threadIdx.x / WARP_SIZE * half2_cols;
  half2 *s_grad = (reinterpret_cast<half2 *>(smem)) +
                  threadIdx.x / WARP_SIZE * half2_cols +
                  blockDim.x / WARP_SIZE * half2_cols;

  const half2 *softmax_out_ptr = (const half2 *)(softmax_out);
  const half2 *grad_out_ptr = (const half2 *)(grad_out);
  half2 *grad_in_ptr = (half2 *)(grad_in);

  half2 zero_half(0.0f, 0.0f);
  for (int row_id = warp_id; row_id < rows; row_id += total_warps) {
    half2 sum = zero_half;
    for (int col_id = local_tid; col_id < half2_cols; col_id += WARP_SIZE) {
      int pos_id = row_id * half2_cols + col_id;
      half2 out = __ldg(&softmax_out_ptr[pos_id]);
      half2 grad = __ldg(&grad_out_ptr[pos_id]);
      if (apply_dropout)
        grad = dropout_bw(grad, scale, pos_id, (const uchar2 *)dropout_mask);

      sum = __hfma2(out, grad, sum);
      s_out[col_id] = out;
      s_grad[col_id] = grad;
    }
    __half reduce_sum = warpReduceSum<float>((float)(sum.x + sum.y));
    sum.x = reduce_sum;
    sum.y = reduce_sum;
    for (int col_id = local_tid; col_id < half2_cols; col_id += WARP_SIZE)
      grad_in_ptr[row_id * half2_cols + col_id] =
          __hmul2(s_out[col_id], __hsub2(s_grad[col_id], sum));
  }
}

template <OperationType OpType>
void Softmax<OpType>::forward(SoftmaxForwardParam param) {
  const int seed = generate_random_seed();

  dim3 grid, block;
  block.x = WARP_SIZE;
  block.y = param.head_num;                    // <= 12
  block.z = max(384 / (block.x * block.y), 1); // thread count <= 384 (32 * 12)
  grid.x = ((param.rows / param.head_num) + block.z - 1) / block.z;
  size_t shared_mem_size =
      param.cols * (block.x * block.y * block.z / WARP_SIZE) *
      sizeof(float); // shared memory <=48KB, support seq_len <= 1024

  softmax_forward_kernel<<<grid, block, shared_mem_size, param.stream>>>(
      param.input, param.softmax_output, param.rows, param.cols, param.add_mask,
      param.mask, param.batch_first, param.apply_dropout, param.dropout_rate,
      param.dropout_mask, param.softmax_dropout_output, seed);
}

template <OperationType OpType>
void Softmax<OpType>::backward(SoftmaxBackwardParam param) {
  float scale = 1.0f / (1.0f - param.dropout_rate);

  dim3 grid(320), block(128);
  size_t shared_mem_size = param.cols * (block.x / WARP_SIZE) *
                           sizeof(DataType_) *
                           2; // shared memory <=48KB, support seq_len <= 1024

  softmax_backward_kernel<<<grid, block, shared_mem_size, param.stream>>>(
      param.grad_out, param.out, param.grad_in, param.rows, param.cols,
      param.apply_dropout, scale, param.dropout_mask);
}

template void Softmax<OperationType::FP32>::forward(SoftmaxForwardParam param);
template void Softmax<OperationType::HALF>::forward(SoftmaxForwardParam param);

template void
Softmax<OperationType::FP32>::backward(SoftmaxBackwardParam param);
template void
Softmax<OperationType::HALF>::backward(SoftmaxBackwardParam param);
} // namespace fastertransformerv4