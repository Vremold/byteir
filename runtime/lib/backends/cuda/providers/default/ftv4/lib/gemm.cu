#include "hip/hip_runtime.h"
/*
 * Author: Xiaoying Jia, Changyi Wan
 * Project: Faster Transformer Training
 * Department: ByteDance Data-AML
 * Email: {jiaxiaoying, wanchangyi}@bytedance.com
 */
#include "fastertransformer_v4/includes/gemm.h"

namespace fastertransformerv4 {
void dense_layer_kernel_launcher(const float *in, const float *weight,
                                 float *out, const int M, const int K,
                                 const int N, hipblasOperation_t trans_A,
                                 hipblasOperation_t trans_B, float alpha,
                                 float beta, hipblasHandle_t cublas_handle,
                                 int cublasAlgo) {
  const int lda = (trans_A == HIPBLAS_OP_N) ? K : M;
  const int ldb = (trans_B == HIPBLAS_OP_N) ? N : K;

  check_cuda_error(hipblasGemmEx(cublas_handle, trans_B, trans_A, N, M, K,
                                &alpha, weight, HIP_R_32F, ldb, in, HIP_R_32F,
                                lda, &beta, out, HIP_R_32F, N, HIP_R_32F,
                                static_cast<hipblasGemmAlgo_t>(cublasAlgo)));
}

void dense_layer_kernel_launcher(const __half *in, const __half *weight,
                                 __half *out, const int M, const int K,
                                 const int N, hipblasOperation_t trans_A,
                                 hipblasOperation_t trans_B, __half alpha,
                                 __half beta, hipblasHandle_t cublas_handle,
                                 int cublasAlgo) {
  const int lda = (trans_A == HIPBLAS_OP_N) ? K : M;
  const int ldb = (trans_B == HIPBLAS_OP_N) ? N : K;

  check_cuda_error(hipblasGemmEx(cublas_handle, trans_B, trans_A, N, M, K,
                                &alpha, weight, HIP_R_16F, ldb, in, HIP_R_16F,
                                lda, &beta, out, HIP_R_16F, N, HIP_R_16F,
                                static_cast<hipblasGemmAlgo_t>(cublasAlgo)));
}

void cublas_Gemm_Strided_Batched(const float *A, const float *B, float *out,
                                 const int M, const int K, const int N,
                                 const int batch_count,
                                 hipblasOperation_t trans_A,
                                 hipblasOperation_t trans_B, float alpha,
                                 float beta, hipblasHandle_t cublas_handle,
                                 int cublasAlgo) {
  const int lda = (trans_A == HIPBLAS_OP_N) ? K : M;
  const int ldb = (trans_B == HIPBLAS_OP_N) ? N : K;

  check_cuda_error(hipblasGemmStridedBatchedEx(
      cublas_handle, trans_B, trans_A, N, M, K, &alpha, B, HIP_R_32F, ldb,
      K * N, A, HIP_R_32F, lda, M * K, &beta, out, HIP_R_32F, N, M * N,
      batch_count, HIP_R_32F, static_cast<hipblasGemmAlgo_t>(cublasAlgo)));
}

void cublas_Gemm_Strided_Batched(const __half *A, const __half *B, __half *out,
                                 const int M, const int K, const int N,
                                 const int batch_count,
                                 hipblasOperation_t trans_A,
                                 hipblasOperation_t trans_B, __half alpha,
                                 __half beta, hipblasHandle_t cublas_handle,
                                 int cublasAlgo) {
  const int lda = (trans_A == HIPBLAS_OP_N) ? K : M;
  const int ldb = (trans_B == HIPBLAS_OP_N) ? N : K;

  check_cuda_error(hipblasGemmStridedBatchedEx(
      cublas_handle, trans_B, trans_A, N, M, K, &alpha, B, HIP_R_16F, ldb,
      K * N, A, HIP_R_16F, lda, M * K, &beta, out, HIP_R_16F, N, M * N,
      batch_count, HIP_R_16F, static_cast<hipblasGemmAlgo_t>(cublasAlgo)));
}
} // namespace fastertransformerv4