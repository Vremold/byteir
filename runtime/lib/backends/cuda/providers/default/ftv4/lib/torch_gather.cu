#include "hip/hip_runtime.h"
/*
 * Author: Xiaoying Jia, Changyi Wan
 * Project: Faster Transformer Training
 * Department: ByteDance Data-AML
 * Email: {jiaxiaoying, wanchangyi}@bytedance.com
 */
#include "fastertransformer_v4/includes/reduce.h"
#include "fastertransformer_v4/includes/torch_gather.h"
#include "fastertransformer_v4/includes/utils.h"
using namespace std;

namespace fastertransformerv4 {
template <typename T, const int max_ite>
__global__ void
gather_torch_forward_kernel(const T *p2c, const T *c2p, const T *score,
                            T *final, const int batch_size, const int num_heads,
                            const int seq_len, const T scaler) {
  int bid = blockIdx.x;
  int offset = bid * seq_len * seq_len * 2;
  int score_offset = bid * seq_len * seq_len;

  const int max_len = 256;
  __shared__ float s_mem[32][max_len + 32];

  int warp_id = threadIdx.x >> 5;
  int warp_tid = threadIdx.x & 0x1F;

  for (int col_id = warp_id; col_id < (((seq_len + 31) >> 5) << 5);
       col_id += 32) // result col loop
  {
    int col_offset = col_id - warp_id;
    for (int i = 0; i < max_ite; i++) {
      if (col_id < seq_len)
        for (int tid = warp_tid + i * max_len;
             tid < min((i + 1) * max_len, seq_len) + 32; tid += 32) {
          int index = (seq_len - 1) + 32 + col_offset - tid;
          s_mem[warp_id][index % (max_len + 32)] =
              __ldg(&p2c[offset + col_id * (seq_len * 2) + index]);
        }

      __syncthreads();

      int warp_col_id = col_offset + warp_tid;
      if (warp_col_id < seq_len)
        for (int row_id = warp_id + i * max_len;
             row_id < min((i + 1) * max_len, seq_len);
             row_id += 32) // result row loop
        {
          int c2p_index = (seq_len - 1) + row_id - warp_col_id;
          T c2p_val = __ldg(&c2p[offset + row_id * (seq_len * 2) + c2p_index]);

          int p2c_index = (seq_len - 1) - row_id + warp_col_id;
          T p2c_val = s_mem[warp_tid][p2c_index % (max_len + 32)];

          int result_index = score_offset + row_id * seq_len + warp_col_id;
          final[result_index] =
              (p2c_val + c2p_val) * scaler + __ldg(&score[result_index]);
        }
      __syncthreads();
    }
  }
}

template <typename T, const int seq_len>
__global__ void gather_torch_backward_kernel(
    const T *grad_out, const T *grad_out_T, T *c2p_grad, T *p2c_grad,
    T *score_grad, const int batch_size, const int head_num, const T scaler) {
  int bid = blockIdx.x;
  int grad_offset = bid * seq_len * seq_len;

  const int ROW = 16;
  __shared__ float s_mem[ROW][seq_len + 1];
  __shared__ float s_mem2[ROW][seq_len + 1];

  for (int start_row = 0; start_row < seq_len; start_row += ROW) {
    int upper_bound = min(ROW, seq_len - start_row);
    for (int row_id = 0; row_id < upper_bound; row_id++) {
      int tid = threadIdx.x;
      int grad_id = grad_offset + (start_row + row_id) * seq_len + tid;

      T grad = __ldg(&grad_out[grad_id]);
      score_grad[grad_id] = grad;
      s_mem[row_id][tid] = grad * scaler;

      s_mem2[row_id][tid] = __ldg(&grad_out_T[grad_id]) * scaler;
    }

    __syncthreads();

    for (int row_id = 0; row_id < upper_bound; row_id++) {
      int c2p_grad_offset =
          (bid * seq_len + (start_row + row_id)) * (seq_len * 2);
      int p2c_grad_offset =
          (bid * seq_len + (start_row + row_id)) * (seq_len * 2);
      for (int tid = threadIdx.x; tid < seq_len * 2; tid += blockDim.x) {
        int col_id = (seq_len - 1) - (tid - (start_row + row_id));

        c2p_grad[c2p_grad_offset + tid] =
            (col_id >= 0 && col_id < seq_len) ? s_mem[row_id][col_id] : 0.0f;
        p2c_grad[p2c_grad_offset + tid] =
            (col_id >= 0 && col_id < seq_len) ? s_mem2[row_id][col_id] : 0.0f;
      }
    }

    __syncthreads();
  }
}

template <typename T>
__global__ void transpose_grad_kernel(const T *grad, T *gradT,
                                      const int batch_size, const int head_num,
                                      const int seq_len) {
  const int TILE_SIZE = 64;
  __shared__ T s_grad[TILE_SIZE][TILE_SIZE + 1];

  int bid = blockIdx.x;
  int offset = bid * seq_len * seq_len;

  for (int start_row = 0; start_row < seq_len; start_row += TILE_SIZE) {
    for (int start_col = 0; start_col < seq_len; start_col += TILE_SIZE) {
      for (int tid = threadIdx.x; tid < TILE_SIZE * TILE_SIZE;
           tid += blockDim.x) {
        int row_id = tid / TILE_SIZE;
        int col_id = tid % TILE_SIZE;
        s_grad[row_id][col_id] =
            __ldg(&grad[offset + (start_row + row_id) * seq_len + start_col +
                        col_id]);
      }

      __syncthreads();

      for (int tid = threadIdx.x; tid < TILE_SIZE * TILE_SIZE;
           tid += blockDim.x) {
        int row_id = tid / TILE_SIZE;
        int col_id = tid % TILE_SIZE;
        gradT[offset + (start_col + row_id) * seq_len + start_row + col_id] =
            s_grad[col_id][row_id];
      }

      __syncthreads();
    }
  }
}

template <OperationType OpType>
void TorchGather<OpType>::forward(TorchGatherForwardParam param) {
  dim3 grid(param.batch_size * param.head_num), block(1024);
  const int max_len = 256;
  const int max_ite = (param.seq_len + max_len - 1) / max_len;
  switch (max_ite) {
  case 1:
    gather_torch_forward_kernel<DataType_, 1><<<grid, block, 0, param.stream>>>(
        param.p2c_ptr, param.c2p_ptr, param.score_ptr, param.output,
        param.batch_size, param.head_num, param.seq_len,
        (DataType_)param.scaler);
    break;
  case 2:
    gather_torch_forward_kernel<DataType_, 2><<<grid, block, 0, param.stream>>>(
        param.p2c_ptr, param.c2p_ptr, param.score_ptr, param.output,
        param.batch_size, param.head_num, param.seq_len,
        (DataType_)param.scaler);
    break;
  case 3:
    gather_torch_forward_kernel<DataType_, 3><<<grid, block, 0, param.stream>>>(
        param.p2c_ptr, param.c2p_ptr, param.score_ptr, param.output,
        param.batch_size, param.head_num, param.seq_len,
        (DataType_)param.scaler);
    break;
  case 4:
    gather_torch_forward_kernel<DataType_, 4><<<grid, block, 0, param.stream>>>(
        param.p2c_ptr, param.c2p_ptr, param.score_ptr, param.output,
        param.batch_size, param.head_num, param.seq_len,
        (DataType_)param.scaler);
    break;
  default:
    printf("seq_len is larger than 1024\n");
  }
}

template <OperationType OpType>
void TorchGather<OpType>::backward(TorchGatherBackwardParam param) {
  dim3 grid(param.batch_size * param.head_num), block;

  DataType_ *gradT = param.grad_out_T;

  block.x = 512;
  transpose_grad_kernel<<<grid, block, 0, param.stream>>>(
      param.grad_out, gradT, param.batch_size, param.head_num, param.seq_len);

  block.x = param.seq_len;
  if (param.seq_len == 256)
    gather_torch_backward_kernel<DataType_, 256>
        <<<grid, block, 0, param.stream>>>(
            param.grad_out, gradT, param.c2p_grad, param.p2c_grad,
            param.score_grad, param.batch_size, param.head_num,
            (DataType_)param.scaler);
}

template void
TorchGather<OperationType::FP32>::forward(TorchGatherForwardParam param);
template void
TorchGather<OperationType::HALF>::forward(TorchGatherForwardParam param);

template void
TorchGather<OperationType::FP32>::backward(TorchGatherBackwardParam param);
template void
TorchGather<OperationType::HALF>::backward(TorchGatherBackwardParam param);
} // namespace fastertransformerv4