#include "hip/hip_runtime.h"
/*
 * Author: Xiaoying Jia, Changyi Wan
 * Project: Faster Transformer Training
 * Department: ByteDance Data-AML
 * Email: {jiaxiaoying, wanchangyi}@bytedance.com
 */
#include "fastertransformer_v4/includes/transpose.h"
using namespace std;

namespace fastertransformerv4 {
const int WARP_SIZE = 32;

template <transposeType transpose_type>
__global__ void transpose4d_kernel(const float *input, float *output,
                                   const int dim_1, const int dim_2,
                                   const int dim_3, const int dim_4) {
  int warp_id = (threadIdx.x + blockIdx.x * blockDim.x) / WARP_SIZE;
  int total_data_blocks = dim_1 * dim_2 * dim_3;
  int total_warps = blockDim.x * gridDim.x / WARP_SIZE;

  int local_tid = threadIdx.x % WARP_SIZE;

  for (; warp_id < total_data_blocks; warp_id += total_warps) {
    int d1 = warp_id / (dim_2 * dim_3);
    int d2 = (warp_id % (dim_2 * dim_3)) / dim_3;
    int d3 = warp_id % dim_3;
    int source_row_id = warp_id;
    int target_row_id =
        transpose3d<transpose_type>(dim_1, dim_2, dim_3, d1, d2, d3);
    for (int tid = local_tid; tid < dim_4; tid += WARP_SIZE)
      output[target_row_id * dim_4 + tid] = input[source_row_id * dim_4 + tid];
  }
}

template <transposeType transpose_type>
__global__ void transpose4d_kernel(const __half *input, __half *output,
                                   const int dim_1, const int dim_2,
                                   const int dim_3, const int dim_4) {
  int warp_id = (threadIdx.x + blockIdx.x * blockDim.x) / WARP_SIZE;
  int total_data_blocks = dim_1 * dim_2 * dim_3;
  int total_warps = blockDim.x * gridDim.x / WARP_SIZE;

  int local_tid = threadIdx.x % WARP_SIZE;

  const half2 *input_ptr = (const half2 *)(input);
  half2 *output_ptr = (half2 *)(output);
  int dim_4_half = dim_4 / 2;

  for (; warp_id < total_data_blocks; warp_id += total_warps) {
    int d1 = warp_id / (dim_2 * dim_3);
    int d2 = (warp_id % (dim_2 * dim_3)) / dim_3;
    int d3 = warp_id % dim_3;
    int source_row_id = warp_id;
    int target_row_id =
        transpose3d<transpose_type>(dim_1, dim_2, dim_3, d1, d2, d3);
    for (int tid = local_tid; tid < dim_4_half; tid += WARP_SIZE)
      output_ptr[target_row_id * dim_4_half + tid] =
          input_ptr[source_row_id * dim_4_half + tid];
  }
}

template <OperationType OpType>
void Transpose<OpType>::forward(TransposeParam param) {
  dim3 grid(320), block(512);
  switch (param.transpose_type) {
  case TRANSPOSE0213:
    transpose4d_kernel<TRANSPOSE0213><<<grid, block, 0, param.stream>>>(
        param.input, param.output, param.dim_1, param.dim_2, param.dim_3,
        param.dim_4);
    break;
  case TRANSPOSE1203:
    transpose4d_kernel<TRANSPOSE1203><<<grid, block, 0, param.stream>>>(
        param.input, param.output, param.dim_1, param.dim_2, param.dim_3,
        param.dim_4);
    break;
  case TRANSPOSE2013:
    transpose4d_kernel<TRANSPOSE2013><<<grid, block, 0, param.stream>>>(
        param.input, param.output, param.dim_1, param.dim_2, param.dim_3,
        param.dim_4);
    break;
  }
}

template void Transpose<OperationType::FP32>::forward(TransposeParam param);
template void Transpose<OperationType::HALF>::forward(TransposeParam param);
} // namespace fastertransformerv4