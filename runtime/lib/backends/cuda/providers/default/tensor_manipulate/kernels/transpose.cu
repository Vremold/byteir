//===- transpose.cu -------------------------------------------*--- C++ -*-===//
//
// Copyright (c) ByteDance Inc. All rights reserved.
// Licensed under the Apache License, Version 2.0
//
//===----------------------------------------------------------------------===//


#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

namespace brt {
namespace cuda {
namespace kernel {

template <typename T>
__global__ void transpose_naive_2d_kernel(const T *input, T *output, int m,
                                          int n) {
  int ix = blockIdx.x * blockDim.x + threadIdx.x;
  int iy = blockIdx.y * blockDim.y + threadIdx.y;

  if (iy < m && ix < n) {
    int in_idx = iy * n + ix;
    int out_idx = ix * m + iy;
    output[out_idx] = input[in_idx];
  }
}

template <typename T>
void transpose_naive_2d(const T *input, T *output, int m, int n, dim3 grid,
                        dim3 block, hipStream_t stream) {
  transpose_naive_2d_kernel<T><<<grid, block, 0, stream>>>(input, output, m, n);
}

// instantiate
template void transpose_naive_2d<float>(const float *, float *, int, int, dim3,
                                        dim3, hipStream_t);
template void transpose_naive_2d<__half>(const __half *, __half *, int, int,
                                         dim3, dim3, hipStream_t);

} // namespace kernel
} // namespace cuda
} // namespace brt
