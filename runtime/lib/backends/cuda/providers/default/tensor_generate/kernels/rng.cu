#include "hip/hip_runtime.h"
//===- rng.cu -------------------------------------------------*--- C++ -*-===//
//
// Copyright (c) ByteDance Inc. All rights reserved.
// Licensed under the Apache License, Version 2.0
//
//===----------------------------------------------------------------------===//

#include "./rng.h"

#include <atomic>
#include <hiprand/hiprand_kernel.h>
#include <iostream>

// TODO: move to common header
#define DIVUP(x, y) (((x) + (y)-1) / (y))

namespace brt {
namespace cuda {
namespace kernel {
template <int NumElementsPerThread>
__global__ void _RngUniform(float *ptr, int32_t N, float base, float range,
                            size_t seed, size_t offset) {
  int32_t id = NumElementsPerThread * blockDim.x * blockIdx.x + threadIdx.x;

  hiprandState_t state;

  // initialize local state with 2^67 * sequence + offset steps
  hiprand_init(seed,   /* seed */
              id,     /* sequence */
              offset, /* offset */
              &state);

#pragma unroll
  for (int i = 0; i < NumElementsPerThread; i++) {
    if (id < N) {
      float value = hiprand_uniform(&state);
      ptr[id] = base + value * range;
      id += blockDim.x;
    }
  }
}

namespace details {
struct GlobalRngState {
public:
  GlobalRngState() : seed_(0) { offset_.store(0); }

  size_t next(size_t n) { return offset_.fetch_add(n); }
  size_t seed() { return seed_; }

  static GlobalRngState *inst() {
    static GlobalRngState _;
    return &_;
  }

private:
  size_t seed_;
  std::atomic<size_t> offset_;
};
} // namespace details

void RngUniform(hipStream_t stream, float *ptr, size_t length, float low,
                float high) {
  constexpr int maxThreadsPerBlock = 256;
  constexpr int maxElementsPerThread = 4;
  int blocksPerGrid = static_cast<int>(
      DIVUP(length, maxThreadsPerBlock * maxElementsPerThread));
  int32_t N = static_cast<int32_t>(length);
  auto globalState = details::GlobalRngState::inst();
  size_t seed = globalState->seed();
  size_t offset = globalState->next(maxElementsPerThread);
  _RngUniform<maxElementsPerThread>
      <<<blocksPerGrid, maxThreadsPerBlock, 0, stream>>>(
          ptr, N, low, high - low, seed, offset);
}

} // namespace kernel
} // namespace cuda
} // namespace brt