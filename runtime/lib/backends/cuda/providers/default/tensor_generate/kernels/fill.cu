#include "hip/hip_runtime.h"
//===- fill.cu ------------------------------------------------*--- C++ -*-===//
//
// Copyright (c) ByteDance Inc. All rights reserved.
// Licensed under the Apache License, Version 2.0
//
//===----------------------------------------------------------------------===//

#include "./fill.h"

// TODO: move to common header
#define DIVUP(x, y) (((x) + (y)-1) / (y))

namespace brt {
namespace cuda {
namespace kernel {
template <typename T, int NumThreadsPerBlock, int NumElementsPerThread>
__global__ void _Fill(T *output_data, T val, int32_t N) {
  int32_t id = NumElementsPerThread * blockDim.x * blockIdx.x + threadIdx.x;

#pragma unroll
  for (int i = 0; i < NumElementsPerThread; i++) {
    if (id < N) {
      output_data[id] = val;
      id += blockDim.x;
    }
  }
}

template <typename T>
void Fill(hipStream_t stream, T *output, T value, size_t count) {
  constexpr int maxThreadsPerBlock = 256;
  constexpr int maxElementsPerThread = 4;
  int blocksPerGrid =
      static_cast<int>(DIVUP(count, maxThreadsPerBlock * maxElementsPerThread));
  int32_t N = static_cast<int32_t>(count);
  _Fill<T, maxThreadsPerBlock, maxElementsPerThread>
      <<<blocksPerGrid, maxThreadsPerBlock, 0, stream>>>(output, value, N);
}

#define INST(T) template void Fill<T>(hipStream_t, T *, T, size_t);

INST(float)
INST(int64_t)
INST(double)
INST(__half)

#undef INST

} // namespace kernel
} // namespace cuda
} // namespace brt