#include "hip/hip_runtime.h"
/*
 * Author: Xiaoying Jia, Changyi Wan
 * Project: Faster Transformer Inference
 * Department: ByteDance Data-AML
 * Email: {jiaxiaoying, wanchangyi}@bytedance.com
 */
#include "fastertransformer_v3/includes/common.h"
#include "fastertransformer_v3/includes/fused_multi_head_attention.h"
#include "fastertransformer_v3/includes/operators.cuh"
#include "fastertransformer_v3/includes/utils.h"
#include <cmath>
#include <cstdio>
#include <cstdlib>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime_api.h>
using namespace std;

// #include <mma.h>
// using namespace nvcuda;

namespace fastertransformerv3 {

template <const int size_per_head>
__global__ void attention_kernel(const float *query, const float *query_bias,
                                 const float *key, const float *key_bias,
                                 const float *value, const float *value_bias,
                                 const float *key_padding_mask,
                                 float *attention_output, const int batch_size,
                                 const int num_heads, const int from_seq_len,
                                 const int to_seq_len, const float scaler) {
  const int max_from_seq_len = 32;
  const int max_to_seq_len = 32;

  __shared__ float s_logits[max_from_seq_len][max_to_seq_len];
  __shared__ float s_query[max_from_seq_len][size_per_head + 1];
  __shared__ float s_kv[max_to_seq_len][size_per_head + 1];

  const int bid = blockIdx.x / num_heads;
  const int head_id = blockIdx.x % num_heads;
  const int input_dim = num_heads * size_per_head;

  int ele_N = from_seq_len * size_per_head;
  for (int tid = threadIdx.x; tid < ele_N; tid += blockDim.x) {
    int seq_id = tid / size_per_head;
    int dim_id = tid % size_per_head;
    int offset = head_id * size_per_head + dim_id;
    int pos = seq_id * batch_size * input_dim + bid * input_dim + offset;
    s_query[seq_id][dim_id] = (__ldg(&query[pos]) + __ldg(&query_bias[offset]));
  }

  ele_N = to_seq_len * size_per_head;
  for (int tid = threadIdx.x; tid < ele_N; tid += blockDim.x) {
    int seq_id = tid / size_per_head;
    int dim_id = tid % size_per_head;
    int offset = head_id * size_per_head + dim_id;
    int pos = seq_id * batch_size * input_dim + bid * input_dim + offset;
    s_kv[seq_id][dim_id] = __ldg(&key[pos]) + __ldg(&key_bias[offset]);
  }

  __syncthreads();

  ele_N = from_seq_len * to_seq_len;
  for (int tid = threadIdx.x; tid < ele_N; tid += blockDim.x) {
    int from_id = tid / to_seq_len;
    int to_id = tid % to_seq_len;

    float tmp = 0.0f;
    for (int i = 0; i < size_per_head; ++i)
      tmp += s_query[from_id][i] * s_kv[to_id][i];

    float mask =
        (1.0f - __ldg(&key_padding_mask[bid * max_to_seq_len + to_id])) *
        -10000.0f; // query_mask.logical_not()
    s_logits[from_id][to_id] = tmp * scaler + mask;
  }

  __syncthreads();

  // softmax
  for (int from_id = (threadIdx.x >> 5); from_id < from_seq_len;
       from_id += (blockDim.x >> 5)) {
    float max_val = -1e20f;
    for (int to_id = (threadIdx.x & 0x1f); to_id < to_seq_len; to_id += 32)
      max_val = max(max_val, s_logits[from_id][to_id]);

    max_val = warpReduceMax(max_val);
    float sum_val = 0.0f;
    for (int to_id = (threadIdx.x & 0x1f); to_id < to_seq_len; to_id += 32) {
      float temp = __expf(s_logits[from_id][to_id] - max_val);
      s_logits[from_id][to_id] = temp;
      sum_val += temp;
    }
    sum_val = warpReduceSum(sum_val) + 1e-6f;
    for (int to_id = (threadIdx.x & 0x1f); to_id < to_seq_len; to_id += 32)
      s_logits[from_id][to_id] /= sum_val;
  }

  ele_N = to_seq_len * size_per_head;
  for (int tid = threadIdx.x; tid < ele_N; tid += blockDim.x) {
    int seq_id = tid / size_per_head;
    int dim_id = tid % size_per_head;
    int offset = head_id * size_per_head + dim_id;
    int pos = seq_id * batch_size * input_dim + bid * input_dim + offset;
    s_kv[seq_id][dim_id] = __ldg(&value[pos]) + __ldg(&value_bias[offset]);
  }

  __syncthreads();

  ele_N = from_seq_len * size_per_head;
  for (int tid = threadIdx.x; tid < ele_N; tid += blockDim.x) {
    int from_id = tid / size_per_head;
    int dim_id = tid % size_per_head;

    float tmp = 0.0f;
    for (int i = 0; i < to_seq_len; ++i)
      tmp += s_logits[from_id][i] * s_kv[i][dim_id];

    int pos = from_id * batch_size * input_dim + bid * input_dim +
              head_id * size_per_head + dim_id;
    attention_output[pos] = tmp;
  }
}

template <const int size_per_head>
__global__ void attention_kernel(
    const __half *Q_buf, const __half *query_bias_buf, const __half *K_buf,
    const __half *key_bias_buf, const __half *value, const __half *value_bias,
    const __half *key_padding_mask, __half *attention_output,
    const int batch_size, const int num_heads, const int from_seq_len,
    const int to_seq_len, const __half scaler) {
  const int half_size_per_head = size_per_head / 2;
  const int max_from_seq_len = 32;
  const int max_to_seq_len = 32;

  const half2 *query = (const half2 *)Q_buf;
  const half2 *query_bias = (const half2 *)query_bias_buf;
  const half2 *key = (const half2 *)K_buf;
  const half2 *key_bias = (const half2 *)key_bias_buf;

  __shared__ half2 s_query[max_from_seq_len][half_size_per_head];
  __shared__ half2 s_key[max_to_seq_len][half_size_per_head];
  __shared__ float s_logits[max_from_seq_len][max_to_seq_len];
  __shared__ __half s_value[max_to_seq_len][size_per_head + 1];

  const int bid = blockIdx.x / num_heads;
  const int head_id = blockIdx.x % num_heads;
  const int half_input_dim = num_heads * half_size_per_head;
  const int input_dim = num_heads * size_per_head;

  // loading Query
  int ele_N = from_seq_len * half_size_per_head;
  for (int tid = threadIdx.x; tid < ele_N; tid += blockDim.x) {
    int seq_id = tid / half_size_per_head;
    int dim_id = tid % half_size_per_head;
    int offset = head_id * half_size_per_head + dim_id;
    int pos =
        seq_id * batch_size * half_input_dim + bid * half_input_dim + offset;
    s_query[seq_id][dim_id] =
        __hadd2(__ldg(&query[pos]), __ldg(&query_bias[offset]));
  }

  // loading key
  ele_N = to_seq_len * half_size_per_head;
  for (int tid = threadIdx.x; tid < ele_N; tid += blockDim.x) {
    int seq_id = tid / half_size_per_head;
    int dim_id = tid % half_size_per_head;
    int offset = head_id * half_size_per_head + dim_id;
    int pos =
        seq_id * batch_size * half_input_dim + bid * half_input_dim + offset;
    s_key[seq_id][dim_id] = __hadd2(__ldg(&key[pos]), __ldg(&key_bias[offset]));
  }

  __syncthreads();

  ele_N = from_seq_len * to_seq_len;
  half2 zero_half = __float2half2_rn(0.0f);
  for (int tid = threadIdx.x; tid < ele_N; tid += blockDim.x) {
    int from_id = tid / to_seq_len;
    int to_id = tid % to_seq_len;

    half2 tmp = zero_half;
    for (int i = 0; i < half_size_per_head; ++i)
      tmp = __hfma2(s_query[from_id][i], s_key[to_id][i], tmp);

    __half mask = ((__half)1.0f -
                   __ldg(&key_padding_mask[bid * max_to_seq_len + to_id])) *
                  (__half)-10000.0f;
    s_logits[from_id][to_id] = (float)(__hadd(tmp.x, tmp.y) * scaler + mask);
  }

  __syncthreads();

  // softmax
  for (int from_id = (threadIdx.x >> 5); from_id < from_seq_len;
       from_id += (blockDim.x >> 5)) {
    float max_val = -1e20f;
    for (int to_id = (threadIdx.x & 0x1f); to_id < to_seq_len; to_id += 32)
      max_val = max(max_val, s_logits[from_id][to_id]);

    max_val = warpReduceMax(max_val);
    float sum_val = 0.0f;
    for (int to_id = (threadIdx.x & 0x1f); to_id < to_seq_len; to_id += 32) {
      float temp = __expf(s_logits[from_id][to_id] - max_val);
      s_logits[from_id][to_id] = temp;
      sum_val += temp;
    }
    sum_val = warpReduceSum(sum_val) + 1e-6f;
    for (int to_id = (threadIdx.x & 0x1f); to_id < to_seq_len; to_id += 32)
      s_logits[from_id][to_id] /= sum_val;
  }

  // loading V
  ele_N = to_seq_len * size_per_head;
  for (int tid = threadIdx.x; tid < ele_N; tid += blockDim.x) {
    int seq_id = tid / size_per_head;
    int dim_id = tid % size_per_head;
    int offset = head_id * size_per_head + dim_id;
    int pos = seq_id * batch_size * input_dim + bid * input_dim + offset;
    s_value[seq_id][dim_id] = __ldg(&value[pos]) + __ldg(&value_bias[offset]);
  }

  __syncthreads();

  //* V
  ele_N = from_seq_len * size_per_head;
  for (int tid = threadIdx.x; tid < ele_N; tid += blockDim.x) {
    int from_id = tid / size_per_head;
    int dim_id = tid % size_per_head;

    __half tmp = (__half)0.0f;
    for (int i = 0; i < to_seq_len; ++i)
      tmp += (__half)s_logits[from_id][i] * s_value[i][dim_id];

    int pos = from_id * batch_size * input_dim + bid * input_dim +
              head_id * size_per_head + dim_id;
    attention_output[pos] = tmp;
  }
}

template <OperationType OpType_>
void MultiHeadAttention<OpType_>::fused_infer(
    const DataType_ *query, const DataType_ *key, const DataType_ *value,
    const DataType_ *key_padding_mask, DataType_ *attn_output, void *buf,
    const int batch_size, const int from_seq_len, const int to_seq_len,
    hipblasHandle_t cublas_handle, hipStream_t stream) {
  DataType_ *Q_buf = (DataType_ *)buf;
  DataType_ *K_buf = (DataType_ *)Q_buf + q_buf_size_;
  DataType_ *V_buf = (DataType_ *)K_buf + k_buf_size_;
  DataType_ *dst_buf = (DataType_ *)V_buf + k_buf_size_;

  dense_layer_kernel_launcher(
      query, param_.query_weight, Q_buf, batch_size * from_seq_len, hidden_dim_,
      hidden_dim_, cublas_handle, stream, param_.cublas_Algo[0]);

  // dense_layer_kernel_launcher(
  //     query, param_.key_weight, K_buf,
  //     batch_size * to_seq_len, hidden_dim_, hidden_dim_, cublas_handle,
  //     stream, param_.cublas_Algo[0]);

  // dense_layer_kernel_launcher(
  //     query, param_.value_weight, V_buf,
  //     batch_size * to_seq_len, hidden_dim_, hidden_dim_, cublas_handle,
  //     stream, param_.cublas_Algo[0]);

  DataType_ alpha = (DataType_)1.0f, beta = (DataType_)0.0f;

  int M = batch_size * to_seq_len, K = hidden_dim_, N = hidden_dim_;
  check_cuda_error(hipblasGemmStridedBatchedEx(
      cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &alpha,
      param_.key_weight, Traits_::BType, N,
      param_.value_weight - param_.key_weight, key, Traits_::AType, K, 0, &beta,
      K_buf, Traits_::CType, N, k_buf_size_, 2, Traits_::computeType,
      static_cast<hipblasGemmAlgo_t>(param_.cublas_Algo[1])));

  // int size_per_head = hidden_dim_ / head_num_;
  DataType_ scaler = (DataType_)0.25f; //(1.0f / sqrt(size_per_head));

  dim3 grid(batch_size * head_num_);
  dim3 block;
  if (OpType_ == OperationType::FP32)
    block.x = 128;
  else
    block.x = 128;

  attention_kernel<16><<<grid, block, 0, stream>>>(
      Q_buf, param_.query_bias, K_buf, param_.key_bias, V_buf,
      param_.value_bias, key_padding_mask, dst_buf, batch_size, head_num_,
      from_seq_len, to_seq_len, scaler);

  dense_layer_kernel_launcher(
      dst_buf, param_.out_proj_weight, attn_output, batch_size * from_seq_len,
      hidden_dim_, hidden_dim_, cublas_handle, stream, param_.cublas_Algo[0]);

  grid.x = batch_size * from_seq_len;
  block.x = hidden_dim_; // assert block.x <= 1024

  add_bias_act<ActType::No, DataType_>
      <<<grid, block, 0, stream>>>(attn_output, param_.out_proj_bias,
                                   batch_size * from_seq_len, hidden_dim_);
}

template void MultiHeadAttention<OperationType::FP32>::fused_infer(
    const float *query, const float *key, const float *value,
    const float *key_padding_mask, float *attn_output, void *buf,
    const int batch_size, const int from_seq_len, const int to_seq_len,
    hipblasHandle_t cublas_handle, hipStream_t stream);

template void MultiHeadAttention<OperationType::HALF>::fused_infer(
    const __half *query, const __half *key, const __half *value,
    const __half *key_padding_mask, __half *attn_output, void *buf,
    const int batch_size, const int from_seq_len, const int to_seq_len,
    hipblasHandle_t cublas_handle, hipStream_t stream);
} // namespace fastertransformerv3
