#include "hip/hip_runtime.h"
/*
 * Author: Xiaoying Jia, Changyi Wan
 * Project: Faster Transformer Inference
 * Department: ByteDance Data-AML
 * Email: {jiaxiaoying, wanchangyi}@bytedance.com
 */
#include "fastertransformer_v3/includes/utils.h"
#include "fastertransformer_v3/includes/convolution.h"
#include <hip/hip_fp16.h>
using namespace std;

namespace fastertransformerv3
{
__global__
void add_bias_glu(const float *input, const float *bias, float *output, const int M, const int N)
{
    int row_offset = blockIdx.x * N;
    int tid = threadIdx.x;

    float front = __ldg(&input[row_offset + tid]) + __ldg(&bias[tid]);
    float back  = __ldg(&input[row_offset + (tid + blockDim.x)]) + __ldg(&bias[tid + blockDim.x]);

    output[row_offset / 2 + tid] = front / (1.0f + __expf(-back));
}

__global__
void add_bias_glu(const __half *input, const __half *bias, __half *output, const int M, const int N)
{
    const half2 *input_ptr = (half2 *)input;
    const half2 *bias_ptr = (const half2 *)bias;
    half2 *output_ptr = (half2 *)output;

    int row_offset = blockIdx.x * N / 2;
    int tid = threadIdx.x;

    float2 front = __half22float2(__hadd2(__ldg(&input_ptr[row_offset + tid]), __ldg(&bias_ptr[tid])));
    float2 back  = __half22float2(__hadd2(__ldg(&input_ptr[row_offset + (tid + blockDim.x)]), __ldg(&bias_ptr[tid + blockDim.x])));

    float2 result;
    result.x = (front.x) / (1.0f + __expf(-back.x));
    result.y = (front.y) / (1.0f + __expf(-back.y));

    output_ptr[row_offset / 2 + tid] = __float22half2_rn(result);
}

__global__
void transpose_to_NCL(const float *input, float *output, const int seq_len, const int hidden_dim)
{
    __shared__ float s_in[32][32 + 1];
    if(blockIdx.y * 32 + threadIdx.y < seq_len)
    {
        int offset = (blockIdx.z * seq_len * hidden_dim) + (blockIdx.y * 32 * hidden_dim) + (blockIdx.x * 32) + (threadIdx.y * hidden_dim);
        s_in[threadIdx.y][threadIdx.x] = __ldg(&input[offset + threadIdx.x]);
    }
    __syncthreads();
    if(blockIdx.y * 32 + threadIdx.x < seq_len)
    {
        int offset = (blockIdx.z * seq_len * hidden_dim) + (blockIdx.x * 32 * seq_len) + (blockIdx.y * 32) + (threadIdx.y * seq_len);
        output[offset + threadIdx.x] = s_in[threadIdx.x][threadIdx.y];
    }
}

__global__
void transpose_to_NCL(const __half *input, __half *output, const int seq_len, const int hidden_dim)
{
    __shared__ float s_in[32][32 + 1]; //todo: use half2
    if(blockIdx.y * 32 + threadIdx.y < seq_len)
    {
        int offset = (blockIdx.z * seq_len * hidden_dim) + (blockIdx.y * 32 * hidden_dim) + (blockIdx.x * 32) + (threadIdx.y * hidden_dim);
        s_in[threadIdx.y][threadIdx.x] = (float)__ldg(&input[offset + threadIdx.x]);
    }
    __syncthreads();
    if(blockIdx.y * 32 + threadIdx.x < seq_len)
    {
        int offset = (blockIdx.z * seq_len * hidden_dim) + (blockIdx.x * 32 * seq_len) + (blockIdx.y * 32) + (threadIdx.y * seq_len);
        output[offset + threadIdx.x] = (__half)s_in[threadIdx.x][threadIdx.y];
    }
}

__global__
void transpose_to_NLC(const float *input, float *output, const int hidden_dim, const int seq_len)
{
    __shared__ float s_in[32][32 + 1];
    if(blockIdx.x * 32 + threadIdx.x < seq_len)
    {
        int offset = (blockIdx.z * hidden_dim * seq_len) + (blockIdx.y * 32 * seq_len) + (blockIdx.x * 32) + (threadIdx.y * seq_len);
        s_in[threadIdx.y][threadIdx.x] = __ldg(&input[offset + threadIdx.x]);
    }
    __syncthreads();
    if(blockIdx.x * 32 + threadIdx.y < seq_len)
    {
        int offset = (blockIdx.z * seq_len * hidden_dim) + (blockIdx.x * 32 * hidden_dim) + (blockIdx.y * 32) + (threadIdx.y * hidden_dim);
        output[offset + threadIdx.x] = s_in[threadIdx.x][threadIdx.y];
    }
}

__global__
void transpose_to_NLC(const __half *input, __half *output, const int hidden_dim, const int seq_len)
{
    __shared__ float s_in[32][32 + 1]; //todo: use half2
    if(blockIdx.x * 32 + threadIdx.x < seq_len)
    {
        int offset = (blockIdx.z * hidden_dim * seq_len) + (blockIdx.y * 32 * seq_len) + (blockIdx.x * 32) + (threadIdx.y * seq_len);
        s_in[threadIdx.y][threadIdx.x] = (float)__ldg(&input[offset + threadIdx.x]);
    }
    __syncthreads();
    if(blockIdx.x * 32 + threadIdx.y < seq_len)
    {
        int offset = (blockIdx.z * seq_len * hidden_dim) + (blockIdx.x * 32 * hidden_dim) + (blockIdx.y * 32) + (threadIdx.y * hidden_dim);
        output[offset + threadIdx.x] = (__half)s_in[threadIdx.x][threadIdx.y];
    }
}

__global__
void depthwise_conv(const float *input, const float *conv_kernel, float *output, int seq_len, int N)
{
    __shared__ float s_conv_weight[31];

    int channel_id = blockIdx.x;
    int batch_id = blockIdx.y;

    if(threadIdx.x < 31)
        s_conv_weight[threadIdx.x] = __ldg(&conv_kernel[channel_id * 31 + threadIdx.x]);
    __syncthreads();

    if(threadIdx.x < seq_len)
    {
        int offset = (batch_id * N + channel_id) * seq_len + threadIdx.x;

        float sum = 0.0f;
        for(int i = -15; i <= 15; i++)
        {
            int pos = threadIdx.x + i;
            float in = (pos >= 0 && pos < seq_len) ? __ldg(&input[offset + i]) : 0.0f;
            sum += s_conv_weight[15 + i] * in;
        }

        output[offset] = sum;
    }
}

__global__
void depthwise_conv(const __half *input, const __half *conv_kernel, __half *output, int seq_len, int N)
{
    __shared__ float s_conv_weight[31];

    int channel_id = blockIdx.x;
    int batch_id = blockIdx.y;

    if(threadIdx.x < 31)
        s_conv_weight[threadIdx.x] = (float)__ldg(&conv_kernel[channel_id * 31 + threadIdx.x]);
    __syncthreads();

    if(threadIdx.x < seq_len)
    {
        int offset = (batch_id * N + channel_id) * seq_len + threadIdx.x;

        float sum = 0.0f;
        for(int i = -15; i <= 15; i++)
        {
            int pos = threadIdx.x + i;
            float in = (pos >= 0 && pos < seq_len) ? (float)__ldg(&input[offset + i]) : 0.0f;
            sum += s_conv_weight[15 + i] * in;
        }

        output[offset] = (__half)sum;
    }
}

__global__
void add_bias_batchnorm_swish(float *output, const float *bias, const float *mean, const float *var, const void *gamma, const void *beta, int m, int n, int hidden_dim, bool use_fp32)
{
    int tid = threadIdx.x;
    int offset = blockIdx.x * n + tid;
    float local_out = output[offset] + __ldg(&bias[tid]) - __ldg(&mean[tid]);
    float variance = rsqrtf(__ldg(&var[tid]) + 1e-5f);
    float result = local_out * variance * __ldg(&((float *)gamma)[tid]) + __ldg(&((float *)beta)[tid]);
    output[offset] = swish(result);
}

__global__
void add_bias_batchnorm_swish(__half *output, const __half *bias, const __half *mean, const __half *var, const void *gamma, const void *beta, int m, int n, int hidden_dim, bool use_fp32)
{
    int tid = threadIdx.x;
    int offset = blockIdx.x * n / 2 + tid;

    half2 *out_ptr = (half2 *)output;
    const half2 *bias_ptr = (const half2 *)bias;
    const half2 *mean_ptr = (const half2 *)mean;
    const half2 *var_ptr  = (const half2 *)var;

    float2 local_out_fp2 = __half22float2(__hsub2(__hadd2(out_ptr[offset], __ldg(&bias_ptr[tid])), __ldg(&mean_ptr[tid])));

    float2 variance = __half22float2(__ldg(&var_ptr[tid]));
    variance.x = rsqrtf(variance.x + 1e-5f);
    variance.y = rsqrtf(variance.y + 1e-5f);

    float2 gamma_val, beta_val;
    if(use_fp32)
    {
        gamma_val = __ldg(&((const float2 *)gamma)[tid]);
        beta_val  = __ldg(&((const float2 *)beta)[tid]);
    }
    else
    {
        gamma_val = __half22float2(__ldg(&((const half2 *)gamma)[tid]));
        beta_val  = __half22float2(__ldg(&((const half2 *)beta)[tid]));
    }

    local_out_fp2.x = local_out_fp2.x * variance.x * gamma_val.x + beta_val.x;
    local_out_fp2.y = local_out_fp2.y * variance.y * gamma_val.y + beta_val.y;
    half2 result = __float22half2_rn(local_out_fp2);
    out_ptr[offset] = swish(result);
}

}