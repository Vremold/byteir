#include "hip/hip_runtime.h"
/*
 * Author: Xiaoying Jia, Changyi Wan, Song Yu
 * Project: Faster Transformer Inference
 * Department: ByteDance Data-AML
 * Email: {jiaxiaoying, wanchangyi, yusong.andy}@bytedance.com
 */
#include "fastertransformer_v3/includes/attention.h"
#include "fastertransformer_v3/includes/common.h"
#include "fastertransformer_v3/includes/disentangle.h"
#include "fastertransformer_v3/includes/nofused_utils.h"
#include "fastertransformer_v3/includes/operators.cuh"
#include "fastertransformer_v3/includes/softmax_kernels.h"
#include <cmath>
#include <cstdio>
#include <cstdlib>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime_api.h>
using namespace std;

namespace fastertransformerv3 {

template <OperationType OpType>
void Attention<OpType>::nofused_infer(
    const DataType_ *query_in, const DataType_ *key_in,
    const DataType_ *value_in, const DataType_ *atten_mask,
    DataType_ *attention_output, void *buf, const int batch_size,
    const int seq_len, hipblasHandle_t cublas_handle, hipStream_t stream) {
  int input_tensor_size = batch_size * head_num_ * seq_len * size_per_head_;
  int qk_buf_size = ((batch_size * head_num_ * seq_len * seq_len + 15) >> 4)
                    << 4;

  DataType_ *query = (DataType_ *)buf + 0 * input_tensor_size;
  DataType_ *key = (DataType_ *)buf + 1 * input_tensor_size;
  DataType_ *value = (DataType_ *)buf + 2 * input_tensor_size;
  DataType_ *qk_buf = (DataType_ *)buf + 3 * input_tensor_size;
  DataType_ *transpose_dst = (DataType_ *)qk_buf + qk_buf_size;

  DataType_ *attn_score =
      is_deberta_ ? (transpose_dst + input_tensor_size) : nullptr;
  DataType_ *disentangled_buf =
      is_deberta_ ? (attn_score + qk_buf_size) : nullptr;

  int size_per_head_half = (OpType == OperationType::HALF)
                               ? size_per_head_ / 2
                               : size_per_head_; // Be careful.

  dim3 grid, block;

  grid.x = batch_size * seq_len;
  block.x = head_num_ * size_per_head_half;
  add_QKV_bias<<<grid, block, 0, stream>>>(
      query_in, param_.attr_bias_Q, key_in, param_.attr_bias_K, value_in,
      param_.attr_bias_V, query, key, value, batch_size, seq_len, head_num_,
      size_per_head_half);

  DataType_ alpha = (DataType_)1.0f, beta = (DataType_)0.0f;

  if (is_deberta_)
    alpha = (DataType_)(1.0f / sqrtf(size_per_head_ *
                                     param_.disentangle_param.scale * 1.0f));

  int M = seq_len, K = size_per_head_, N = seq_len;
  check_cuda_error(hipblasGemmStridedBatchedEx(
      cublas_handle, HIPBLAS_OP_T, HIPBLAS_OP_N, N, M, K, &alpha, key,
      Traits_::BType, K, K * N, query, Traits_::AType, K, M * K, &beta, qk_buf,
      Traits_::CType, N, M * N, batch_size * head_num_, Traits_::computeType,
      static_cast<hipblasGemmAlgo_t>(param_.cublas_Algo[0])));

  DataType_ *attr_probs;
  if (is_deberta_) {
    disentangle_layer_->infer(qk_buf, query, key, param_.attr_bias_Q,
                              param_.attr_bias_K, attn_score, disentangled_buf,
                              batch_size, seq_len, cublas_handle, stream);

    bool no_scale = true;

    softmax_kernelLauncher<OpType, DataType_>(attn_score, atten_mask,
                                              batch_size, seq_len, head_num_,
                                              size_per_head_, stream, no_scale);

    attr_probs = attn_score;
  } else {
    softmax_kernelLauncher<OpType, DataType_>(qk_buf, atten_mask, batch_size,
                                              seq_len, head_num_,
                                              size_per_head_, stream);
    attr_probs = qk_buf;
  }

  alpha = (DataType_)(1.0f);

  M = seq_len, K = seq_len, N = size_per_head_;
  check_cuda_error(hipblasGemmStridedBatchedEx(
      cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &alpha, value,
      Traits_::BType, N, K * N, attr_probs, Traits_::AType, K, M * K, &beta,
      transpose_dst, Traits_::CType, N, M * N, batch_size * head_num_,
      Traits_::computeType,
      static_cast<hipblasGemmAlgo_t>(param_.cublas_Algo[1])));

  grid.x = batch_size * seq_len;
  block.x = size_per_head_half, block.y = head_num_;
  transpose<<<grid, block, 0, stream>>>(transpose_dst, attention_output,
                                        batch_size, seq_len, head_num_,
                                        size_per_head_half);
}

template <OperationType OpType>
void Attention<OpType>::et_nofused_infer(
    const DataType_ *query_in, const DataType_ *key_in,
    const DataType_ *value_in, const DataType_ *atten_mask,
    DataType_ *attention_output, void *buf, const int batch_size,
    const int seq_len, hipblasHandle_t cublas_handle, hipStream_t stream,
    ET_Param et_param) {
  int input_tensor_size = batch_size * head_num_ * seq_len * size_per_head_;
  int qk_buf_size = ((batch_size * head_num_ * seq_len * seq_len + 15) >> 4)
                    << 4;

  DataType_ *query = (DataType_ *)buf + 0 * input_tensor_size;
  DataType_ *key = (DataType_ *)buf + 1 * input_tensor_size;
  DataType_ *value = (DataType_ *)buf + 2 * input_tensor_size;
  DataType_ *qk_buf = (DataType_ *)buf + 3 * input_tensor_size;
  DataType_ *transpose_dst = qk_buf + qk_buf_size;
  DataType_ *attn_score = transpose_dst + input_tensor_size;
  DataType_ *disentangled_buf = attn_score + qk_buf_size;

  int size_per_head_half = (OpType == OperationType::HALF)
                               ? size_per_head_ / 2
                               : size_per_head_; // Be careful.

  hipMemsetAsync(query, 0, 3 * input_tensor_size * sizeof(DataType_),
                  stream); // clean zero for batch_gemm

  dim3 grid, block;

  grid.x = et_param.valid_word_num;
  block.x = head_num_ * size_per_head_half;
  add_QKV_bias_padding<<<grid, block, 0, stream>>>(
      query_in, param_.attr_bias_Q, key_in, param_.attr_bias_K, value_in,
      param_.attr_bias_V, query, key, value, batch_size, seq_len, head_num_,
      size_per_head_half, et_param.batch_idx, et_param.word_idx);

  DataType_ alpha = (DataType_)1.0f, beta = (DataType_)0.0f;
  int M = seq_len, K = size_per_head_, N = seq_len;

  if (is_deberta_)
    alpha = (DataType_)(1.0f / sqrtf(size_per_head_ *
                                     param_.disentangle_param.scale * 1.0f));

  check_cuda_error(hipblasGemmStridedBatchedEx(
      cublas_handle, HIPBLAS_OP_T, HIPBLAS_OP_N, N, M, K, &alpha, key,
      Traits_::BType, K, K * N, query, Traits_::AType, K, M * K, &beta, qk_buf,
      Traits_::CType, N, M * N, batch_size * head_num_, Traits_::computeType,
      static_cast<hipblasGemmAlgo_t>(param_.cublas_Algo[0])));

  DataType_ *attr_probs;
  if (is_deberta_) {
    disentangle_layer_->infer(qk_buf, query, key, param_.attr_bias_Q,
                              param_.attr_bias_K, attn_score, disentangled_buf,
                              batch_size, seq_len, cublas_handle, stream);

    bool no_scale = true;
    softmax_kernelLauncher<OpType, DataType_>(attn_score, atten_mask,
                                              batch_size, seq_len, head_num_,
                                              size_per_head_, stream, no_scale);

    attr_probs = attn_score;

  } else {
    softmax_kernelLauncher<OpType, DataType_>(qk_buf, atten_mask, batch_size,
                                              seq_len, head_num_,
                                              size_per_head_, stream);
    attr_probs = qk_buf;
  }

  alpha = (DataType_)(1.0f);
  M = seq_len, K = seq_len, N = size_per_head_;
  check_cuda_error(hipblasGemmStridedBatchedEx(
      cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &alpha, value,
      Traits_::BType, N, K * N, attr_probs, Traits_::AType, K, M * K, &beta,
      transpose_dst, Traits_::CType, N, M * N, batch_size * head_num_,
      Traits_::computeType,
      static_cast<hipblasGemmAlgo_t>(param_.cublas_Algo[1])));

  grid.x = et_param.valid_word_num;
  block.x = size_per_head_half, block.y = head_num_;
  transpose_rm_padding<<<grid, block, 0, stream>>>(
      transpose_dst, attention_output, batch_size, seq_len, head_num_,
      size_per_head_half, et_param.batch_idx, et_param.word_idx);
}

template void Attention<OperationType::FP32>::nofused_infer(
    const float *query, const float *key, const float *value,
    const float *atten_mask, float *attention_output, void *buf,
    const int batch_size, const int seq_len, hipblasHandle_t cublas_handle,
    hipStream_t stream);

template void Attention<OperationType::HALF>::nofused_infer(
    const __half *query, const __half *key, const __half *value,
    const __half *atten_mask, __half *attention_output, void *buf,
    const int batch_size, const int seq_len, hipblasHandle_t cublas_handle,
    hipStream_t stream);

template void Attention<OperationType::FP32>::et_nofused_infer(
    const float *query, const float *key, const float *value,
    const float *atten_mask, float *attention_output, void *buf,
    const int batch_size, const int seq_len, hipblasHandle_t cublas_handle,
    hipStream_t stream, ET_Param et_param);

template void Attention<OperationType::HALF>::et_nofused_infer(
    const __half *query, const __half *key, const __half *value,
    const __half *atten_mask, __half *attention_output, void *buf,
    const int batch_size, const int seq_len, hipblasHandle_t cublas_handle,
    hipStream_t stream, ET_Param et_param);
} // namespace fastertransformerv3
