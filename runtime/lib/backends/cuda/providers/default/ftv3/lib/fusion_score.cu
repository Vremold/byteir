#include "hip/hip_runtime.h"
/*
 * Author: Xiaoying Jia, Changyi Wan
 * Project: Faster Transformer Inference
 * Department: ByteDance Data-AML
 * Email: {jiaxiaoying, wanchangyi}@bytedance.com
 */
#include "fastertransformer_v3/includes/common.h"
#include "fastertransformer_v3/includes/operators.cuh"
#include "fastertransformer_v3/includes/utils.h"
#include "fastertransformer_v3/includes/fusion_score.h"
#include <cstdio>
#include <cstdlib>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime_api.h>
#include <cmath>
using namespace std;

namespace fastertransformerv3
{

template <typename T>
__global__
void transpose_dim01(
                const T *input, T *output, const int dim0, const int dim1, const int num_units)
{
    int bid  = blockIdx.x;
    int idx0 = bid / dim1;
    int idx1 = bid % dim1;

    int source_offset = (idx0 * dim1 + idx1) * num_units;
    int target_offset = (idx1 * dim0 + idx0) * num_units;

    for(int tid = threadIdx.x; tid < num_units; tid += blockDim.x)
        output[target_offset + tid] = __ldg(&input[source_offset + tid]);
}

template <ActType act, typename T>
__global__
void matrix_mul_vector_add_bias_act(
                const T *input, const T *weight, const T *bias,
                T *output, const int M, const int K)
{
    int row_offset = blockIdx.x * K;

    T sum = (T) 0.0f;
    for(int tid = threadIdx.x; tid < K; tid += 32)
        sum += input[row_offset + tid] * weight[tid];

    sum = warpReduceSum(sum);

    if(threadIdx.x == 0)
    {
        T out = sum + __ldg(&bias[0]);
        output[blockIdx.x] = act_fun<act>(out);
    }
}

template<OperationType OpType_>
void FusionScore<OpType_>::fused_infer(
                const DataType_* query, const DataType_* key, const DataType_* value, const DataType_* key_padding_mask,
                DataType_* score, void *buf,
                const int batch_size, const int from_seq_len, const int to_seq_len, hipblasHandle_t cublas_handle, hipStream_t stream)
{
    DataType_* key_T = (DataType_ *)buf;
    DataType_* attention_output = (DataType_ *)key_T   + key_T_size_;
    DataType_* mid_out = (DataType_ *)attention_output + attention_output_size_;
    DataType_* multi_head_attention_buf = (DataType_ *)mid_out + mid_out_size_;

    dim3 grid(batch_size * to_seq_len);
    dim3 block(hidden_dim_); //assert block.x <= 1024

    transpose_dim01<<<grid, block, 0, stream>>>(key, key_T, batch_size, to_seq_len, hidden_dim_);

    multi_head_attention_layer_->infer(
                    query, key_T, key_T, key_padding_mask,
                    attention_output, multi_head_attention_buf,
                    batch_size, from_seq_len, to_seq_len, cublas_handle, stream);

    dense_layer_kernel_launcher(
                    attention_output, param_.linear1_weight, mid_out,
                    batch_size * from_seq_len, hidden_dim_, fc_hidden_size1_, cublas_handle, stream, param_.cublas_Algo[0]);

    grid.x = batch_size * from_seq_len;
    block.x = fc_hidden_size1_; //assert block.x <= 1024

    add_bias_act<ActType::Relu, DataType_><<<grid, block, 0, stream>>>(mid_out, param_.linear1_bias, batch_size * from_seq_len, fc_hidden_size1_);

    // dense_layer_kernel_launcher(
    //     mid_out, param_.linear2_weight, score,
    //     batch_size * from_seq_len, fc_hidden_size1_, fc_hidden_size2_, cublas_handle, stream, param_.cublas_Algo[1]);

    // grid.x = batch_size * from_seq_len;
    // block.x = 1; //assert block.x <= 1024

    // add_bias_act<ActType::Sigmoid, DataType_><<<grid, block, 0, stream>>>(score, param_.linear2_bias, batch_size * from_seq_len, fc_hidden_size2_);

    grid.x = batch_size * from_seq_len;
    block.x = 32; // fc_hidden_size1_ -> warpReduce
    matrix_mul_vector_add_bias_act<ActType::Sigmoid, DataType_><<<grid, block, 0, stream>>>(
                    mid_out, param_.linear2_weight, param_.linear2_bias, score,
                    batch_size * from_seq_len, fc_hidden_size1_);
}

template void FusionScore<OperationType::FP32>::fused_infer(
                const float *query, const float *key, const float *value, const float *key_padding_mask,
                float *attn_output, void *buf,
                const int batch_size, const int from_seq_len, const int to_seq_len, hipblasHandle_t cublas_handle, hipStream_t stream);

template void FusionScore<OperationType::HALF>::fused_infer(
                const __half *query, const __half *key, const __half *value, const __half *key_padding_mask,
                __half *attn_output, void *buf,
                const int batch_size, const int from_seq_len, const int to_seq_len, hipblasHandle_t cublas_handle, hipStream_t stream);
}
