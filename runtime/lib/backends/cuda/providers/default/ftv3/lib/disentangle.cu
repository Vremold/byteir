#include "hip/hip_runtime.h"
/*
 * Author: Xiaoying Jia, Song Yu
 * Project: Faster Transformer Inference
 * Department: ByteDance Data-AML
 * Email: {jiaxiaoying, yusong.andy}@bytedance.com
 */
#include "fastertransformer_v3/includes/common.h"
#include "fastertransformer_v3/includes/disentangle.h"
#include "fastertransformer_v3/includes/utils.h"
#include <hipblas.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>

namespace fastertransformerv3 {

template <typename T>
__global__ void relative_embedding_lookup(T *relative_embedding_out,
                                          const T *relative_embedding,
                                          const int start, const int end,
                                          const int dim) {
  int block_idx = blockIdx.x;
  if (block_idx + start < end) {
    const T *block_from = relative_embedding + dim * (block_idx + start);
    T *block_to = relative_embedding_out + dim * block_idx;
    int tid = threadIdx.x;
    for (int i = tid; i < dim; i += blockDim.x) {
      block_to[i] = block_from[i];
    }
  }
}

template <typename T>
void relative_embedding_lookup_kernel_launcher(T *relative_embedding_res,
                                               const T *relative_embedding,
                                               const int seq_len,
                                               const int max_relative_positions,
                                               const int hidden_dim,
                                               hipStream_t stream) {
  int start = max_relative_positions - seq_len,
      end = max_relative_positions + seq_len;
  dim3 grid(seq_len * 2);
  dim3 block(1024);
  relative_embedding_lookup<<<grid, block, 0, stream>>>(
      relative_embedding_res, relative_embedding, start, end, hidden_dim);
}

template <typename T>
__global__ void
add_bias_transpose(T *trans_pos_query_buf, T *trans_pos_key_buf,
                   const T *pos_query_buf, const T *pos_key_buf,
                   const T *bias_query, const T *bias_key,
                   // const T *pos_query_buf, const T *pos_key_buf, const T
                   // *bias_key, const T *bias_query,
                   const int batch_size, const int seq_len, const int head_num,
                   const int size_per_head) {
  int seq_id = blockIdx.x / head_num;
  int head_id = blockIdx.x % head_num;
  int dim_id = threadIdx.x;

  int src_id = blockIdx.x * blockDim.x + threadIdx.x;
  int bias_id = head_id * size_per_head + dim_id;
  int tgt_id =
      head_id * (seq_len * size_per_head) + seq_id * size_per_head + dim_id;

  T pos_query_value =
      __ldg(pos_query_buf + src_id) + __ldg(bias_query + bias_id);
  T pos_key_value = __ldg(pos_key_buf + src_id) + __ldg(bias_key + bias_id);

  for (int batch_id = 0; batch_id < batch_size; ++batch_id) {
    int batch_tgt_id = tgt_id + batch_id * (head_num * seq_len * size_per_head);
    trans_pos_query_buf[batch_tgt_id] = pos_query_value;
    trans_pos_key_buf[batch_tgt_id] = pos_key_value;
  }
}

template <typename T>
void add_bias_transpose_kernel_launcher(
    T *trans_pos_query_buf, const T *pos_query_buf, const T *bias_query,
    T *trans_pos_key_buf, const T *pos_key_buf, const T *bias_key,
    int batch_size, int seq_len, int head_num, int size_per_head,
    hipStream_t stream) {
  dim3 grid(1 * seq_len * head_num), block(size_per_head);
  add_bias_transpose<<<grid, block, 0, stream>>>(
      trans_pos_query_buf, trans_pos_key_buf, pos_query_buf, pos_key_buf,
      bias_query, bias_key, batch_size, seq_len, head_num, size_per_head);
}

template <typename T>
__global__ void gather_torch_kernel(const T *p2c, const T *c2p, const T *score,
                                    T *final, const int batch_size,
                                    const int num_heads, const int seq_len,
                                    const T scaler, const bool is_paper) {

  int bid = blockIdx.x;
  int offset = bid * seq_len * 2 * seq_len;

  for (int tid = threadIdx.x; tid < seq_len * seq_len; tid += blockDim.x) {
    int i = tid / seq_len;
    int j = tid % seq_len;

    int c2p_index = offset + i * seq_len * 2 + seq_len - 1 + i - j;
    int p2c_index =
        is_paper ? offset + j * seq_len * 2 + seq_len - 1 - i + j : c2p_index;

    T p2c_val = __ldg(&p2c[p2c_index]);
    T c2p_val = __ldg(&c2p[c2p_index]);

    T sum = (p2c_val + c2p_val) * scaler;
    sum += __ldg(&score[bid * seq_len * seq_len + tid]);
    final[bid * seq_len * seq_len + tid] = sum;
  }
}

template <OperationType OpType_>
void Disentangle<OpType_>::infer(
    const DataType_ *attn_score, const DataType_ *query_out,
    const DataType_ *key_out, const DataType_ *query_bias,
    const DataType_ *key_bias, DataType_ *attn_score_out, void *buf,
    const int batch_size, const int seq_len, hipblasHandle_t cublas_handle,
    hipStream_t stream) {
  const int input_tensor_size = seq_len * 2 * head_num_ * size_per_head_;
  const int middle_tensor_size =
      ((batch_size * head_num_ * seq_len * seq_len * 2 + 15) >> 4) << 4;

  DataType_ *relative_embedding_res = (DataType_ *)(buf);
  DataType_ *pos_query_buf = relative_embedding_res + input_tensor_size;
  DataType_ *pos_key_buf = pos_query_buf + input_tensor_size;
  DataType_ *trans_pos_query_buf = pos_key_buf + input_tensor_size;
  DataType_ *trans_pos_key_buf =
      trans_pos_query_buf + batch_size * input_tensor_size;
  DataType_ *c2p_att = trans_pos_key_buf + batch_size * input_tensor_size;
  DataType_ *p2c_att = c2p_att + middle_tensor_size;

  // [1, seq_len*2, head_num * size_per_head]
  relative_embedding_lookup_kernel_launcher(
      relative_embedding_res, param_.relative_embedding, seq_len,
      param_.max_pos, head_num_ * size_per_head_, stream);

  // [1, seq_len*2, head_num * size_per_head]

  int m = 1 * seq_len * 2;
  int k = head_num_ * size_per_head_;
  int n = k;
  dense_layer_kernel_launcher(relative_embedding_res, param_.attr_kernel_Q,
                              pos_query_buf, m, k, n, cublas_handle, stream,
                              param_.cublas_Algo[0]);

  dense_layer_kernel_launcher(relative_embedding_res, param_.attr_kernel_K,
                              pos_key_buf, m, k, n, cublas_handle, stream,
                              param_.cublas_Algo[0]);

  // print_vec(query_bias, "query_bias", 10);
  // print_vec(key_bias, "key_bias", 10);
  add_bias_transpose_kernel_launcher(trans_pos_query_buf, pos_query_buf,
                                     query_bias, trans_pos_key_buf, pos_key_buf,
                                     key_bias, batch_size, seq_len * 2,
                                     head_num_, size_per_head_, stream);
  // [batch_size, head_num, seq_len, seq_len*2]
  int N = seq_len * 2;
  int M = seq_len;
  int K = size_per_head_;
  DataType_ scaler =
      (DataType_)(1.0f / sqrtf(size_per_head_ * param_.scale * 1.0f));
  DataType_ pos_query_alpha = param_.is_paper ? (DataType_)(1.0f) : scaler;
  DataType_ pos_key_alpha = (DataType_)1.0f;
  DataType_ beta = (DataType_)0.0f;

  check_cuda_error(hipblasGemmStridedBatchedEx(
      cublas_handle, HIPBLAS_OP_T, HIPBLAS_OP_N, N, M, K, &pos_key_alpha,
      trans_pos_key_buf, Traits<OpType_>::BType, K, K * N, query_out,
      Traits<OpType_>::AType, K, M * K, &beta, c2p_att, Traits<OpType_>::CType,
      N, M * N, batch_size * head_num_, Traits<OpType_>::computeType,
      static_cast<hipblasGemmAlgo_t>(param_.cublas_Algo[1])));

  check_cuda_error(hipblasGemmStridedBatchedEx(
      cublas_handle, HIPBLAS_OP_T, HIPBLAS_OP_N, N, M, K, &pos_query_alpha,
      trans_pos_query_buf, Traits<OpType_>::BType, K, K * N, key_out,
      Traits<OpType_>::AType, K, M * K, &beta, p2c_att, Traits<OpType_>::CType,
      N, M * N, batch_size * head_num_, Traits<OpType_>::computeType,
      static_cast<hipblasGemmAlgo_t>(param_.cublas_Algo[1])));

  dim3 grid(batch_size * head_num_);
  dim3 block(1024);

  if (!param_.is_paper)
    scaler = (DataType_)(1.0f);

  gather_torch_kernel<<<grid, block, 0, stream>>>(
      p2c_att, c2p_att, attn_score, attn_score_out, batch_size, head_num_,
      seq_len, scaler, param_.is_paper);
}

template void Disentangle<OperationType::FP32>::infer(
    const float *attn_score, const float *query_out, const float *key_out,
    const float *query_bias, const float *key_bias, float *attn_score_out,
    void *buf, const int batch_size, const int seq_len,
    hipblasHandle_t cublas_handle, hipStream_t stream);

template void Disentangle<OperationType::HALF>::infer(
    const __half *attn_score, const __half *query_out, const __half *key_out,
    const __half *query_bias, const __half *key_bias, __half *attn_score_out,
    void *buf, const int batch_size, const int seq_len,
    hipblasHandle_t cublas_handle, hipStream_t stream);
} // namespace fastertransformerv3
