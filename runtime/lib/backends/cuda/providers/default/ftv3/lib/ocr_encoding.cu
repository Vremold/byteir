#include "hip/hip_runtime.h"
/*
 * Author: Xiaoying Jia, Changyi Wan
 * Project: Faster Transformer Inference
 * Department: ByteDance Data-AML
 * Email: {jiaxiaoying, wanchangyi}@bytedance.com
 */
#include "fastertransformer_v3/includes/common.h"
#include "fastertransformer_v3/includes/operators.cuh"
#include "fastertransformer_v3/includes/utils.h"
#include "fastertransformer_v3/includes/ocr_encoding.h"
#include <cstdio>
#include <cstdlib>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime_api.h>
#include <cmath>
using namespace std;

namespace fastertransformerv3
{
template<typename T>
__global__
void pos_encode(const T *src, const T *pe, T *dst)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int offset = blockIdx.y * gridDim.x * blockDim.x + tid;
    dst[offset] = __ldg(&src[offset]) + __ldg(&pe[tid]);
}

template<>
__global__
void pos_encode(const __half *src, const __half *pe, __half *dst)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int offset = blockIdx.y * gridDim.x * blockDim.x + tid;
    ((half2 *)dst)[offset] = __hadd2(__ldg(&((half2 *)src)[offset]), __ldg(&((half2 *)pe)[tid]));
}

template<OperationType OpType_>
void OCR_Conformer<OpType_>::infer(OCR_ConformerInferParam infer_param)
{
    int hidden_dim = head_num_ * size_per_head_;
    hidden_dim = (OpType_ == OperationType::HALF) ? (hidden_dim / 2) : hidden_dim;  // for float & half

    dim3 grid(infer_param.seq_len, infer_param.batch_size);
    dim3 block(hidden_dim); //assert block.x <= 1024
    pos_encode<<<grid, block, 0, infer_param.stream>>>(infer_param.input_tensor, param_.pos_encoder_src, infer_param.transformer_output);

    struct ConformerInferParam<DataType_> conformer_infer_param
    {
        infer_param.transformer_output, infer_param.atten_mask, infer_param.transformer_output, infer_param.buf,
                                        infer_param.batch_size, infer_param.seq_len, infer_param.cublas_handle, infer_param.stream
    };

    for(int i = 0; i < layers_; i++)
        conformer_layer_[i]->infer(conformer_infer_param);
}

template void OCR_Conformer<OperationType::FP32>::infer(OCR_ConformerInferParam infer_param);
template void OCR_Conformer<OperationType::HALF>::infer(OCR_ConformerInferParam infer_param);
}
