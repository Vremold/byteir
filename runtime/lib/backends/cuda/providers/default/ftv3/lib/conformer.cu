#include "hip/hip_runtime.h"
/*
 * Author: Xiaoying Jia, Changyi Wan
 * Project: Faster Transformer Inference
 * Department: ByteDance Data-AML
 * Email: {jiaxiaoying, wanchangyi}@bytedance.com
 */
#include "fastertransformer_v3/includes/common.h"
#include "fastertransformer_v3/includes/operators.cuh"
#include "fastertransformer_v3/includes/utils.h"
#include "fastertransformer_v3/includes/layernorm_kernels.h"
#include "fastertransformer_v3/includes/add_bias_half_input_out_layernorm_kernels.h"
#include "fastertransformer_v3/includes/add_bias_input_out_layernorm_kernels.h"
#include "fastertransformer_v3/includes/convolution.h"
#include "fastertransformer_v3/includes/conformer.h"
#include <cstdio>
#include <cstdlib>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime_api.h>
#include <cmath>
using namespace std;

namespace fastertransformerv3
{

template<OperationType OpType_>
void Conformer<OpType_>::infer(ConformerInferParam infer_param)
{
    const DataType_* from_tensor = infer_param.input_tensor;    //Todo: remove useless code
    const DataType_* atten_mask  = infer_param.atten_mask;
    DataType_* transformer_out   = infer_param.transformer_output;
    void *buf                    = infer_param.buf;
    const int batch_size         = infer_param.batch_size;
    const int seq_len            = infer_param.seq_len;
    hipblasHandle_t cublas_handle = infer_param.cublas_handle;
    hipStream_t stream          = infer_param.stream;

    int input_tensor_size = batch_size * head_num_ * seq_len * size_per_head_; //todo: set max_seq_len

    DataType_* attention_buf = (DataType_ *)((uint8_t *)buf + inner_buf_size_);
    DataType_* inner_buf     = (DataType_ *)buf;

    DataType_* layernorm_tensor  = inner_buf + 0 * input_tensor_size;

    DataType_* ffn1_inter_matmul_buf_ = inner_buf + 1 * input_tensor_size;
    DataType_* ffn1_out_matmul_buf_   = inner_buf + 2 * input_tensor_size;

    DataType_* new_from_tensor        = inner_buf + 3 * input_tensor_size;

    DataType_* query_buf_ = inner_buf + 4 * input_tensor_size;
    DataType_* key_buf_   = inner_buf + 5 * input_tensor_size;
    DataType_* value_buf_ = inner_buf + 6 * input_tensor_size;

    DataType_* attr_out_buf_     = inner_buf + 7 * input_tensor_size;
    DataType_* attr_matmul_buf_  = inner_buf + 8 * input_tensor_size;
    DataType_* inter_matmul_buf_ = inner_buf + 9 * input_tensor_size;
    DataType_* middle_tensor     = inner_buf + 10 * input_tensor_size;

    DataType_* conv_matmul_buf_  = inner_buf + 11 * input_tensor_size; // *2
    DataType_* glu_out_buf_      = inner_buf + 13 * input_tensor_size;

    DataType_* transpose_buf_    = inner_buf + 14 * input_tensor_size;

    int valid_word_num = batch_size * seq_len;

    int hidden_dim = head_num_ * size_per_head_;
    hidden_dim = (OpType_ == OperationType::HALF) ? (hidden_dim / 2) : hidden_dim;  // for float & half

    ET_Param et_param;
    if(is_remove_padding_)
    {
        et_param.word_idx  = (int *)(inner_buf + 15 * input_tensor_size);
        et_param.batch_idx = et_param.word_idx + batch_size * seq_len;

        build_sequence_length_padding_offset_kernelLauncher(
                        atten_mask, et_param.batch_idx, et_param.word_idx, &valid_word_num,
                        batch_size, seq_len, stream);

        et_param.valid_word_num = valid_word_num;

        input_compress_layernorm_kernel_launcher(
                        layernorm_tensor, from_tensor,
                        param_.ffn1_layernorm_gamma, param_.ffn1_layernorm_beta,
                        valid_word_num, head_num_ * size_per_head_, hidden_dim, stream, use_fp32_,
                        middle_tensor, et_param.batch_idx, et_param.word_idx);

        from_tensor = middle_tensor;        //1. compress from_tensor      -> middle_tensor

        DataType_* tmp  = transformer_out;  //2. compute  transformert_out -> inner_buf
        transformer_out = inner_buf;
        inner_buf = tmp;                    //3. restore  inner_buf        -> from_tensor (real transformer_out)
    }
    else
        input_layernorm_kernel_launcher(
                        layernorm_tensor, from_tensor,
                        param_.ffn1_layernorm_gamma, param_.ffn1_layernorm_beta,
                        valid_word_num, head_num_ * size_per_head_, hidden_dim, stream, use_fp32_);

    int m = valid_word_num;
    int k = head_num_ * size_per_head_;
    int n = k;

    dim3 grid(m);
    dim3 block(hidden_dim); //assert block.x <= 1024

    // FFN1 (pre-layernorm)
    dense_layer_kernel_launcher(
                    layernorm_tensor, param_.ffn1_inter_kernel, ffn1_inter_matmul_buf_,
                    m, k, n, cublas_handle, stream, param_.cublas_Algo[1]);

    add_bias_swish<<<grid, block, 0, stream>>>(
                    ffn1_inter_matmul_buf_, param_.ffn1_inter_bias, m, n);

    dense_layer_kernel_launcher(
                    ffn1_inter_matmul_buf_, param_.ffn1_output_kernel, ffn1_out_matmul_buf_,
                    m, k, n, cublas_handle, stream, param_.cublas_Algo[2]);

    add_bias_half_input_out_layernorm_kernel_launcher(
                    ffn1_out_matmul_buf_, from_tensor, param_.ffn1_output_bias, new_from_tensor,
                    param_.attr_output_layernorm_gamma, param_.attr_output_layernorm_beta, m, n, hidden_dim, stream, use_fp32_);

    //Multi-Head Self Attention
    dense_layer_kernel_launcher(
                    ffn1_out_matmul_buf_, param_.attr_kernel_Q, query_buf_,
                    m, k, n, cublas_handle, stream, param_.cublas_Algo[0]);

    dense_layer_kernel_launcher(
                    ffn1_out_matmul_buf_, param_.attr_kernel_K, key_buf_,
                    m, k, n, cublas_handle, stream, param_.cublas_Algo[0]);

    dense_layer_kernel_launcher(
                    ffn1_out_matmul_buf_, param_.attr_kernel_V, value_buf_,
                    m, k, n, cublas_handle, stream, param_.cublas_Algo[0]);

    attention_layer_->infer(
                    query_buf_, key_buf_, value_buf_, atten_mask, attr_out_buf_, attention_buf,
                    batch_size, seq_len, cublas_handle, stream,
                    et_param);

    dense_layer_kernel_launcher(
                    attr_out_buf_, param_.attr_output_kernel, attr_matmul_buf_,
                    m, k, n, cublas_handle, stream, param_.cublas_Algo[0]);

    add_bias_input_out_layernorm_kernel_launcher(
                    attr_matmul_buf_, new_from_tensor, param_.attr_output_bias, middle_tensor,
                    param_.conv_layernorm_gamma, param_.conv_layernorm_beta, m, n, hidden_dim, stream, use_fp32_);

    // Convolution
    dense_layer_kernel_launcher(
                    attr_matmul_buf_, param_.pointwise_conv_kernel_1, conv_matmul_buf_,
                    m, k, n * 2, cublas_handle, stream); //param_.cublas_Algo[5]

    add_bias_glu<<<grid, block, 0, stream>>>(
                    conv_matmul_buf_, param_.pointwise_conv_bias_1, glu_out_buf_, m, n * 2);

    transpose_to_NCL<<<dim3(n / 32, (seq_len + 31) / 32, batch_size), dim3(32, 32), 0, stream>>>(
                    glu_out_buf_, transpose_buf_, seq_len, n); //[N,L,C] -> [N,C,L] (assert L >= 32, padding for L)

    // add_bias_glu_transpose_dim12<<<dim3(n / 32, (seq_len + 31) / 32, batch_size), dim3(32, 32), 0, stream>>>(
    //    conv_matmul_buf_, param_.pointwise_conv_bias_1, transpose_buf_, seq_len, n);

    depthwise_conv<<<dim3(n, batch_size), max(seq_len, 32), 0, stream>>>(
                    transpose_buf_, param_.depthwise_conv_kernel, conv_matmul_buf_, seq_len, n); //todo: _normal & _et

    // add_bias_layernorm_swish_transpose_dim12<<<dim3((seq_len + 31) / 32, n / 32, batch_size), dim3(32, 32), 0, stream>>>(
    //     transpose_buf_, param_.depthwise_conv_bias,
    //     param_.batchnorm_mean, param_.batchnorm_var, param_.batchnorm_gamma, param_.batchnorm_beta,
    //     glu_out_buf_, n, seq_len); //[N,C,L] -> [N,L,C] remove padding of L

    // transpose_dim12_add_bias_layernorm_swish<<<dim3((seq_len + 31) / 32, n / 32, batch_size), dim3(32, 32), 0, stream>>>(
    //     transpose_buf_, param_.depthwise_conv_bias,
    //     param_.batchnorm_mean, param_.batchnorm_var, param_.batchnorm_gamma, param_.batchnorm_beta,
    //     glu_out_buf_, n, seq_len); //[N,C,L] -> [N,L,C] remove padding of L

    transpose_to_NLC<<<dim3((seq_len + 31) / 32, n / 32, batch_size), dim3(32, 32), 0, stream>>>(
                    conv_matmul_buf_, glu_out_buf_, n, seq_len); //[N,C,L] -> [N,L,C]

    add_bias_batchnorm_swish<<<grid, block, 0, stream>>>(
                    glu_out_buf_, param_.depthwise_conv_bias,
                    param_.batchnorm_mean, param_.batchnorm_var, param_.batchnorm_gamma, param_.batchnorm_beta,
                    m, n, hidden_dim, use_fp32_);

    dense_layer_kernel_launcher(
                    glu_out_buf_, param_.pointwise_conv_kernel_2, conv_matmul_buf_,
                    m, k, n, cublas_handle, stream); //param_.cublas_Algo[5]

    add_bias_input_out_layernorm_kernel_launcher(
                    conv_matmul_buf_, middle_tensor, param_.pointwise_conv_bias_2, new_from_tensor,
                    param_.output_layernorm_gamma, param_.output_layernorm_beta, m, n, hidden_dim, stream, use_fp32_);

    // FFN2 (pre-layernorm)
    dense_layer_kernel_launcher(
                    conv_matmul_buf_, param_.inter_kernel, inter_matmul_buf_,
                    m, k, n, cublas_handle, stream, param_.cublas_Algo[1]);

    add_bias_swish<<<grid, block, 0, stream>>>(
                    inter_matmul_buf_, param_.inter_bias, m, n);

    if(is_remove_padding_)
        hipMemsetAsync(inner_buf,
                        0, batch_size * seq_len * head_num_ * size_per_head_ * sizeof(DataType_), stream);

    dense_layer_kernel_launcher(
                    inter_matmul_buf_, param_.output_kernel, transformer_out,
                    m, k, n, cublas_handle, stream, param_.cublas_Algo[2]);

    if(is_remove_padding_)
        add_bias_half_input_layernorm_restore_output_kernel_launcher(
                        transformer_out, new_from_tensor, param_.output_bias,
                        param_.last_layernorm_gamma, param_.last_layernorm_beta, m, n, hidden_dim, stream, use_fp32_,
                        inner_buf, et_param.batch_idx, et_param.word_idx);
    else
        add_bias_half_input_layernorm_kernel_launcher(
                        transformer_out, new_from_tensor, param_.output_bias,
                        param_.last_layernorm_gamma, param_.last_layernorm_beta, m, n, hidden_dim, stream, use_fp32_);
}

template void Conformer<OperationType::FP32>::infer(ConformerInferParam infer_param);
template void Conformer<OperationType::HALF>::infer(ConformerInferParam infer_param);
}
