#include "hip/hip_runtime.h"
/*
 * Author: Xiaoying Jia, Changyi Wan
 * Project: Faster Transformer Inference
 * Department: ByteDance Data-AML
 * Email: {jiaxiaoying, wanchangyi}@bytedance.com
 */
#include "fastertransformer_v3/includes/utils.h"
#include <hip/hip_fp16.h>

namespace fastertransformerv3 {
void dense_layer_kernel_launcher(const float *in, const float *weight,
                                 float *out, const int M, const int K,
                                 const int N, hipblasHandle_t cublas_handle,
                                 hipStream_t stream, int cublasAlgo) {
  const float alpha = 1.0f, beta = 0.0f;
  check_cuda_error(hipblasGemmEx(
      cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &alpha, weight,
      HIP_R_32F, N, in, HIP_R_32F, K, &beta, out, HIP_R_32F, N, HIP_R_32F,
      static_cast<hipblasGemmAlgo_t>(cublasAlgo)));
}

void dense_layer_kernel_launcher(const __half *in, const __half *weight,
                                 __half *out, const int M, const int K,
                                 const int N, hipblasHandle_t cublas_handle,
                                 hipStream_t stream, int cublasAlgo) {
  const __half alpha = (__half)1.0f, beta = (__half)0.0f;
  check_cuda_error(hipblasGemmEx(
      cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &alpha, weight,
      HIP_R_16F, N, in, HIP_R_16F, K, &beta, out, HIP_R_16F, N, HIP_R_16F,
      static_cast<hipblasGemmAlgo_t>(cublasAlgo)));
}

__global__ void add_bias_gelu(float *output, const float *bias, const int M,
                              const int N) {
  int row_offset = blockIdx.x * N;
  for (int tid = threadIdx.x; tid < N; tid += blockDim.x) {
    float out = output[row_offset + tid] + __ldg(&bias[tid]);
    output[row_offset + tid] = gelu(out);
  }
}

__global__ void add_bias_gelu(__half *output, const __half *bias, const int M,
                              const int N) {
  half2 *output_ptr = (half2 *)output;
  const half2 *bias_ptr = (const half2 *)bias;

  int row_offset = blockIdx.x * N / 2;
  for (int tid = threadIdx.x; tid < N / 2; tid += blockDim.x) {
    half2 out = __hadd2(output_ptr[row_offset + tid], __ldg(&bias_ptr[tid]));
    output_ptr[row_offset + tid] = gelu(out);
  }
}

__global__ void add_bias_swish(float *output, const float *bias, const int M,
                               const int N) {
  int row_offset = blockIdx.x * N;
  for (int tid = threadIdx.x; tid < N; tid += blockDim.x) {
    float out = output[row_offset + tid] + __ldg(&bias[tid]);
    output[row_offset + tid] = swish(out);
  }
}

__global__ void add_bias_swish(__half *output, const __half *bias, const int M,
                               const int N) {
  half2 *output_ptr = (half2 *)output;
  const half2 *bias_ptr = (const half2 *)bias;

  int row_offset = blockIdx.x * N / 2;
  for (int tid = threadIdx.x; tid < N / 2; tid += blockDim.x) {
    half2 out = __hadd2(output_ptr[row_offset + tid], __ldg(&bias_ptr[tid]));
    output_ptr[row_offset + tid] = swish(out);
  }
}

__global__ void add_bias_input(float *out, const float *input,
                               const float *bias, int m, int n) {
  int offset = blockIdx.x * n;
  for (int i = threadIdx.x; i < n; i += blockDim.x) {
    int index = offset + i;
    out[index] = out[index] + __ldg(&input[index]) + __ldg(&bias[i]);
  }
}

__global__ void add_bias_input(__half *out, const __half *input,
                               const __half *bias, int m, int n) {
  half2 *out_ptr = (half2 *)out;
  const half2 *input_ptr = (const half2 *)input;
  const half2 *bias_ptr = (const half2 *)bias;

  int id = blockIdx.x * n / 2 + threadIdx.x;
  out_ptr[id] = __hadd2(__hadd2(out_ptr[id], __ldg(&input_ptr[id])),
                        __ldg(&bias_ptr[threadIdx.x]));
}

__global__ void add_bias_input_restore_output(const float *out,
                                              const float *input,
                                              const float *bias, int m, int n,
                                              float *out2, const int *batch_idx,
                                              const int *word_idx) {
  int input_offset = blockIdx.x * n;
  int output_offset = __ldg(&word_idx[blockIdx.x]) * n;
  for (int i = threadIdx.x; i < n; i += blockDim.x) {
    int index = input_offset + i;
    out2[output_offset + i] =
        out[index] + __ldg(&input[index]) + __ldg(&bias[i]);
  }
}

__global__ void
add_bias_input_restore_output(const __half *out, const __half *input,
                              const __half *bias, int m, int n, __half *out2,
                              const int *batch_idx, const int *word_idx) {
  half2 *out_ptr = (half2 *)out;
  const half2 *input_ptr = (const half2 *)input;
  const half2 *bias_ptr = (const half2 *)bias;

  int offset = __ldg(&word_idx[blockIdx.x]);
  int id = blockIdx.x * n / 2 + threadIdx.x;
  ((half2 *)out2)[offset * n / 2 + threadIdx.x] =
      __hadd2(__hadd2(out_ptr[id], __ldg(&input_ptr[id])),
              __ldg(&bias_ptr[threadIdx.x]));
}

__global__ void add_bias_half_input(float *out, const float *input,
                                    const float *bias, int m, int n) {
  int offset = blockIdx.x * n;
  for (int i = threadIdx.x; i < n; i += blockDim.x) {
    int index = offset + i;
    out[index] = __ldg(&input[index]) + (out[index] + __ldg(&bias[i])) * 0.5f;
  }
}

__global__ void add_bias_half_input(__half *out, const __half *input,
                                    const __half *bias, int m, int n) {
  half2 *out_ptr = (half2 *)out;
  const half2 *input_ptr = (const half2 *)input;
  const half2 *bias_ptr = (const half2 *)bias;

  int id = blockIdx.x * n / 2 + threadIdx.x;
  out_ptr[id] =
      __hadd2(__ldg(&input_ptr[id]),
              __hmul2(__hadd2(out_ptr[id], __ldg(&bias_ptr[threadIdx.x])),
                      half2(0.5f, 0.5f)));
}

__global__ void
add_bias_half_input_restore_output(const float *out, const float *input,
                                   const float *bias, int m, int n, float *out2,
                                   const int *batch_idx, const int *word_idx) {
  int input_offset = blockIdx.x * n;
  int output_offset = __ldg(&word_idx[blockIdx.x]) * n;
  for (int i = threadIdx.x; i < n; i += blockDim.x) {
    int index = input_offset + i;
    out2[output_offset + i] =
        __ldg(&input[index]) + (__ldg(&out[index]) + __ldg(&bias[i])) * 0.5f;
  }
}

__global__ void add_bias_half_input_restore_output(
    const __half *out, const __half *input, const __half *bias, int m, int n,
    __half *out2, const int *batch_idx, const int *word_idx) {
  half2 *out_ptr = (half2 *)out;
  const half2 *input_ptr = (const half2 *)input;
  const half2 *bias_ptr = (const half2 *)bias;

  int offset = __ldg(&word_idx[blockIdx.x]);
  int id = blockIdx.x * n / 2 + threadIdx.x;
  ((half2 *)out2)[offset * n / 2 + threadIdx.x] = __hadd2(
      __ldg(&input_ptr[id]),
      __hmul2(__hadd2(__ldg(&out_ptr[id]), __ldg(&bias_ptr[threadIdx.x])),
              half2(0.5f, 0.5f)));
}

} // namespace fastertransformerv3
