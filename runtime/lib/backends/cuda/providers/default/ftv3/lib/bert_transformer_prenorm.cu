/*
* Author: Xiaoying Jia, Changyi Wan
* Project: Faster Transformer Inference
* Department: ByteDance Data-AML
* Email: {jiaxiaoying, wanchangyi}@bytedance.com
*/
#include "fastertransformer_v3/includes/common.h"
#include "fastertransformer_v3/includes/operators.cuh"
#include "fastertransformer_v3/includes/utils.h"
#include "fastertransformer_v3/includes/layernorm_kernels.h"
#include "fastertransformer_v3/includes/add_bias_input_out_layernorm_kernels.h"
#include "fastertransformer_v3/includes/bert_transformer.h"
#include <cstdio>
#include <cstdlib>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime_api.h>
#include <cmath>
using namespace std;

namespace fastertransformerv3
{

template<OperationType OpType_>
void BertTransformer<OpType_>::prenorm_bert_infer(BertTransformerInferParam infer_param)
{
    const DataType_* from_tensor = infer_param.input_tensor;    //Todo: remove useless code
    const DataType_* atten_mask  = infer_param.atten_mask;
    DataType_* transformer_out   = infer_param.transformer_output;
    void *buf                    = infer_param.buf;
    const int batch_size         = infer_param.batch_size;
    const int seq_len            = infer_param.seq_len;
    hipblasHandle_t cublas_handle = infer_param.cublas_handle;
    hipStream_t stream          = infer_param.stream;

    int input_tensor_size = batch_size * head_num_ * seq_len * size_per_head_;

    DataType_* attention_buf = (DataType_ *)((uint8_t *)buf + inner_buf_size_);
    DataType_* inner_buf     = (DataType_ *)buf;

    DataType_* query_buf_ = inner_buf + 0 * input_tensor_size;
    DataType_* key_buf_   = inner_buf + 1 * input_tensor_size;
    DataType_* value_buf_ = inner_buf + 2 * input_tensor_size;

    DataType_* attr_out_buf_     = inner_buf + 3 * input_tensor_size;
    DataType_* attr_matmul_buf_  = inner_buf + 1 * input_tensor_size;
    DataType_* inter_matmul_buf_ = inner_buf + 5 * input_tensor_size;

    DataType_* layernorm_tensor  = inner_buf + 3 * input_tensor_size;
    DataType_* middle_tensor     = inner_buf + 4 * input_tensor_size;

    int valid_word_num = batch_size * seq_len;

    int hidden_dim = head_num_ * size_per_head_;
    hidden_dim = (OpType_ == OperationType::HALF) ? (hidden_dim / 2) : hidden_dim;  // for float & half

    ET_Param et_param;
    if(is_remove_padding_)
    {
        et_param.word_idx  = (int *)(inter_matmul_buf_ + 4 * input_tensor_size);
        et_param.batch_idx = et_param.word_idx + batch_size * seq_len;

        build_sequence_length_padding_offset_kernelLauncher(
                        atten_mask, et_param.batch_idx, et_param.word_idx, &valid_word_num,
                        batch_size, seq_len, stream);

        et_param.valid_word_num = valid_word_num;

        input_compress_layernorm_kernel_launcher(
                        layernorm_tensor, from_tensor,
                        param_.attr_output_layernorm_gamma, param_.attr_output_layernorm_beta,
                        valid_word_num, head_num_ * size_per_head_, hidden_dim, stream, use_fp32_,
                        middle_tensor, et_param.batch_idx, et_param.word_idx);

        from_tensor = middle_tensor;        //1. compress from_tensor      -> middle_tensor

        DataType_* tmp  = transformer_out;  //2. compute  transformert_out -> inner_buf
        transformer_out = inner_buf;
        inner_buf = tmp;                    //3. restore  inner_buf        -> from_tensor (real transformer_out)
    }
    else
        input_layernorm_kernel_launcher(
                        layernorm_tensor, from_tensor,
                        param_.attr_output_layernorm_gamma, param_.attr_output_layernorm_beta,
                        valid_word_num, head_num_ * size_per_head_, hidden_dim, stream, use_fp32_);

    int m = valid_word_num;
    int k = head_num_ * size_per_head_;
    int n = k;

    dim3 grid(m);
    dim3 block(hidden_dim); //assert block.x <= 1024

    dense_layer_kernel_launcher(
                    layernorm_tensor, param_.attr_kernel_Q, query_buf_,
                    m, k, n, cublas_handle, stream, param_.cublas_Algo[0]);

    dense_layer_kernel_launcher(
                    layernorm_tensor, param_.attr_kernel_K, key_buf_,
                    m, k, n, cublas_handle, stream, param_.cublas_Algo[0]);

    dense_layer_kernel_launcher(
                    layernorm_tensor, param_.attr_kernel_V, value_buf_,
                    m, k, n, cublas_handle, stream, param_.cublas_Algo[0]);

    attention_layer_->infer(
                    query_buf_, key_buf_, value_buf_, atten_mask, attr_out_buf_, attention_buf,
                    batch_size, seq_len, cublas_handle, stream,
                    et_param);

    dense_layer_kernel_launcher(
                    attr_out_buf_, param_.attr_output_kernel, attr_matmul_buf_,
                    m, k, n, cublas_handle, stream, param_.cublas_Algo[0]);

    add_bias_input_out_layernorm_kernel_launcher(
                    attr_matmul_buf_, from_tensor, param_.attr_output_bias, middle_tensor,
                    param_.output_layernorm_gamma, param_.output_layernorm_beta, m, n, hidden_dim, stream, use_fp32_);

    dense_layer_kernel_launcher(
                    attr_matmul_buf_, param_.inter_kernel, inter_matmul_buf_,
                    m, k, n * 4, cublas_handle, stream, param_.cublas_Algo[1]);

    add_bias_gelu<<<grid, block, 0, stream>>>(
                    inter_matmul_buf_, param_.inter_bias, m, n * 4);

    if(is_remove_padding_)
        hipMemsetAsync(inner_buf,
                        0, batch_size * seq_len * head_num_ * size_per_head_ * sizeof(DataType_), stream);

    dense_layer_kernel_launcher(
                    inter_matmul_buf_, param_.output_kernel, transformer_out,
                    m, k * 4, n, cublas_handle, stream, param_.cublas_Algo[2]);

    if(is_remove_padding_)
        add_bias_input_restore_output<<<grid, block, 0, stream>>>(
                        transformer_out, middle_tensor, param_.output_bias, m, n,
                        inner_buf, et_param.batch_idx, et_param.word_idx);
    else
        add_bias_input<<<grid, block, 0, stream>>>(
                        transformer_out, middle_tensor, param_.output_bias, m, n);
}

template void BertTransformer<OperationType::FP32>::prenorm_bert_infer(BertTransformerInferParam infer_param);
template void BertTransformer<OperationType::HALF>::prenorm_bert_infer(BertTransformerInferParam infer_param);
}
