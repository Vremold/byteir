#include "hip/hip_runtime.h"
/*
 * Author: Xiaoying Jia, Changyi Wan
 * Project: Faster Transformer Inference
 * Department: ByteDance Data-AML
 * Email: {jiaxiaoying, wanchangyi}@bytedance.com
 */
#include "fastertransformer_v3/includes/attention.h"
#include "fastertransformer_v3/includes/common.h"
#include "fastertransformer_v3/includes/operators.cuh"
#include <cstdio>
#include <cstdlib>
#include <ctime>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime_api.h>
//#include <sys/time.h>
#include <cmath>
using namespace std;

#include <mma.h>
using namespace nvcuda;

namespace fastertransformerv3 {

#define SKEW_HALF 8 // offset for avoding bank conflict
#define WMMA_M 16
#define WMMA_N 16
#define WMMA_K 16

template <const int max_seq_len, const int size_per_head>
__global__
    // __launch_bounds__(512,4)//THREADS_PER_BLOCK
    void
    wmma_attention_kernel(const half2 *query, const half2 *query_bias,
                          const half2 *key, const half2 *key_bias,
                          const half2 *value, const half2 *value_bias,
                          const __half *attention_mask,
                          __half *attention_output, const int batch_size,
                          const int head_num, const int seq_len,
                          const half2 scaler) {
  const int half_size_per_head = size_per_head / 2;
  const int from_size = max_seq_len / 16;
  const int to_size = max_seq_len / 16;

  __shared__ __half s_kv[max_seq_len][size_per_head + SKEW_HALF];
  __shared__ __half s_query[max_seq_len][size_per_head + SKEW_HALF];
  __shared__ __half s_logits[max_seq_len][max_seq_len + SKEW_HALF];

  const int warpNums = (blockDim.x >> 5);
  const int warpId = (threadIdx.x >> 5);
  const int warp_tid = (threadIdx.x & 0x1f);

  const int bid = blockIdx.x / head_num;
  const int head_id = blockIdx.x % head_num;

  const int half_hidden_dim = head_num * half_size_per_head;
  const int head_offset = head_id * half_size_per_head;
  const int bias_id = head_offset + warp_tid;

  // loading Query & Key
  for (int seq_id = warpId; seq_id < seq_len; seq_id += warpNums) {
    int pos =
        (bid * seq_len + seq_id) * half_hidden_dim + head_offset + warp_tid;
    half2 tmp = __hadd2(__ldg(&query[pos]), __ldg(&query_bias[bias_id]));
    int offset = seq_id * (size_per_head + SKEW_HALF) + (warp_tid << 1);
    *(__half2 *)(*s_query + offset) = __hmul2(tmp, scaler);

    *(__half2 *)(*s_kv + offset) =
        __hadd2(__ldg(&key[pos]), __ldg(&key_bias[bias_id]));
  }

  __syncthreads();

  if (warpId < from_size * to_size) {
    wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half,
                   wmma::row_major>
        Q_mat;
    wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half,
                   wmma::col_major>
        K_mat;
    wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, half> QK_mat;
    wmma::fill_fragment(QK_mat, 0.0f);
    const int warp_from_offset = (warpId / to_size) << 4;
    const int warp_to_offset = (warpId % to_size) << 4;

#pragma unroll
    for (int k = 0; k < 4; k++) {
      wmma::load_matrix_sync(Q_mat, s_query[warp_from_offset] + k * WMMA_K,
                             size_per_head + SKEW_HALF);
      wmma::load_matrix_sync(K_mat, s_kv[warp_to_offset] + k * WMMA_K,
                             size_per_head + SKEW_HALF);
      wmma::mma_sync(QK_mat, Q_mat, K_mat, QK_mat);
    }
    wmma::store_matrix_sync(s_logits[warp_from_offset] + warp_to_offset, QK_mat,
                            max_seq_len + SKEW_HALF, wmma::mem_row_major);
  }

  __syncthreads();

  // softmax
  for (int from_id = warpId; from_id < seq_len; from_id += warpNums) {
    float max_val = -1e20f;

    const int n = (max_seq_len + 31) / 32;
    float logits[n];
    int to_id[n];

#pragma unroll
    for (int i = 0; i < n; i++) {
      to_id[i] = warp_tid + (i << 5);

      float mask =
          to_id[i] < seq_len
              ? (float)__ldg(&attention_mask[bid * seq_len * seq_len +
                                             from_id * seq_len + to_id[i]])
              : 0.0f;
      mask = (1.0f - mask) * (-10000.0f);

      logits[i] = to_id[i] < seq_len
                      ? (float)(s_logits[from_id][to_id[i]]) + mask
                      : -1e20f;
      max_val = max(max_val, logits[i]);
    }

    max_val = warpReduceMax(max_val);

    float sum_val = 0.0f;
#pragma unroll
    for (int i = 0; i < n; i++) {
      logits[i] = __expf(logits[i] - max_val);
      sum_val += (to_id[i] < seq_len) ? logits[i] : 0.0f;
    }

    sum_val = warpReduceSum(sum_val);

#pragma unroll
    for (int i = 0; i < n; i++)
      if (to_id[i] < max_seq_len)
        s_logits[from_id][to_id[i]] =
            (__half)(to_id[i] < seq_len ? (logits[i] / (sum_val + 1e-6f))
                                        : 0.0f);
  }

  // loading Value
  for (int seq_id = warpId; seq_id < seq_len; seq_id += warpNums) {
    int pos =
        (bid * seq_len + seq_id) * half_hidden_dim + head_offset + warp_tid;
    ((__half2 *)(s_kv[seq_id]))[warp_tid] =
        __hadd2(__ldg(&value[pos]), __ldg(&value_bias[bias_id]));
  }

  // K dim clear 0
  for (int seq_id = seq_len + warpId; seq_id < max_seq_len; seq_id += warpNums)
    ((float *)(s_kv[seq_id]))[warp_tid] = 0.0f;

  __syncthreads();

  //* V
  if (warpId < (from_size << 2)) {
    wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half,
                   wmma::row_major>
        Logits_mat;
    wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half,
                   wmma::row_major>
        V_mat;
    wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, half> QKV_mat;
    wmma::fill_fragment(QKV_mat, 0.0f);
    const int warp_from_offset = (warpId >> 2) << 4;
    const int warp_to_offset = (warpId & 0x3) * WMMA_K;

#pragma unroll
    for (int k = 0; k < to_size; k++) {
      wmma::load_matrix_sync(Logits_mat,
                             s_logits[warp_from_offset] + k * WMMA_K,
                             max_seq_len + SKEW_HALF);
      wmma::load_matrix_sync(V_mat, s_kv[k * WMMA_K] + warp_to_offset,
                             size_per_head + SKEW_HALF);
      wmma::mma_sync(QKV_mat, Logits_mat, V_mat, QKV_mat);
    }
    wmma::store_matrix_sync(s_query[warp_from_offset] + warp_to_offset, QKV_mat,
                            size_per_head + SKEW_HALF, wmma::mem_row_major);
  }

  __syncthreads();

  for (int from_id = warpId; from_id < seq_len; from_id += warpNums) {
    int pos =
        (bid * seq_len + from_id) * half_hidden_dim + head_offset + warp_tid;
    ((__half2 *)(attention_output))[pos] =
        ((__half2 *)(s_query[from_id]))[warp_tid];
  }
}

template <const int max_seq_len, const int size_per_head>
__global__
    // __launch_bounds__(256)//THREADS_PER_BLOCK
    void
    wmma_attention_kernel_LE32(const half2 *query, const half2 *query_bias,
                               const half2 *key, const half2 *key_bias,
                               const half2 *value, const half2 *value_bias,
                               const __half *attention_mask,
                               __half *attention_output, const int batch_size,
                               const int head_num, const int seq_len,
                               const half2 scaler) {
  const int half_size_per_head = size_per_head / 2;
  const int from_size = max_seq_len / 16;
  const int to_size = max_seq_len / 16;

  __shared__ __half s_kv[max_seq_len][size_per_head + SKEW_HALF];
  __shared__ __half s_query[max_seq_len][size_per_head + SKEW_HALF];
  __shared__ __half s_logits[max_seq_len][max_seq_len + SKEW_HALF];
  __shared__ __half s_value[max_seq_len][size_per_head + SKEW_HALF];

  const int warpNums = (blockDim.x >> 5);
  const int warpId = (threadIdx.x >> 5);
  const int warp_tid = (threadIdx.x & 0x1f);

  const int bid = blockIdx.x / head_num;
  const int head_id = blockIdx.x % head_num;

  const int half_hidden_dim = head_num * half_size_per_head;
  const int head_offset = head_id * half_size_per_head;
  const int bias_id = head_offset + warp_tid;

  // loading Query & Key
  for (int seq_id = warpId; seq_id < seq_len; seq_id += warpNums) {
    int pos =
        (bid * seq_len + seq_id) * half_hidden_dim + head_offset + warp_tid;
    half2 tmp = __hadd2(__ldg(&query[pos]), __ldg(&query_bias[bias_id]));
    int offset = seq_id * (size_per_head + SKEW_HALF) + (warp_tid << 1);
    *(__half2 *)(*s_query + offset) = __hmul2(tmp, scaler);
    *(__half2 *)(*s_kv + offset) =
        __hadd2(__ldg(&key[pos]), __ldg(&key_bias[bias_id]));
    *(__half2 *)(*s_value + offset) =
        __hadd2(__ldg(&value[pos]), __ldg(&value_bias[bias_id]));
  }

  // K dim clear 0
  for (int seq_id = seq_len + warpId; seq_id < max_seq_len; seq_id += warpNums)
    ((float *)(s_value[seq_id]))[warp_tid] = 0.0f;

  __syncthreads();

  if (warpId < from_size * to_size) {
    wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half,
                   wmma::row_major>
        Q_mat;
    wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half,
                   wmma::col_major>
        K_mat;
    wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, half> QK_mat;
    wmma::fill_fragment(QK_mat, 0.0f);

    const int warp_from_offset = (warpId / to_size) << 4;
    const int warp_to_offset = (warpId % to_size) << 4;

#pragma unroll
    for (int k = 0; k < 4; k++) {
      wmma::load_matrix_sync(Q_mat, s_query[warp_from_offset] + k * WMMA_K,
                             size_per_head + SKEW_HALF);
      wmma::load_matrix_sync(K_mat, s_kv[warp_to_offset] + k * WMMA_K,
                             size_per_head + SKEW_HALF);
      wmma::mma_sync(QK_mat, Q_mat, K_mat, QK_mat);
    }
    wmma::store_matrix_sync(s_logits[warp_from_offset] + warp_to_offset, QK_mat,
                            max_seq_len + SKEW_HALF, wmma::mem_row_major);
  }

  __syncthreads();

  // softmax
  for (int from_id = warpId; from_id < seq_len; from_id += warpNums) {
    float max_val = -1e20f;
    int to_id = warp_tid;

    float mask = to_id < seq_len
                     ? (float)__ldg(&attention_mask[bid * seq_len * seq_len +
                                                    from_id * seq_len + to_id])
                     : 0.0f;
    mask = (1.0f - mask) * (-10000.0f);
    float logits =
        to_id < seq_len ? (float)(s_logits[from_id][to_id]) + mask : max_val;
    max_val = warpReduceMax(logits);

    logits = __expf(logits - max_val);
    float sum_val = (to_id < seq_len) ? logits : 0.0f;
    sum_val = warpReduceSum(sum_val);

    if (to_id < max_seq_len)
      s_logits[from_id][to_id] =
          to_id < seq_len ? logits / (sum_val + 1e-6f) : 0.0f;
  }

  __syncthreads();

  //* V
  if (warpId < (from_size << 2)) {
    wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half,
                   wmma::row_major>
        Logits_mat;
    wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half,
                   wmma::row_major>
        V_mat;
    wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, half> QKV_mat;
    wmma::fill_fragment(QKV_mat, 0.0f);
    const int warp_from_offset = (warpId >> 2) << 4;
    const int warp_to_offset = (warpId & 0x3) * WMMA_N;
#pragma unroll
    for (int k = 0; k < to_size; k++) {
      wmma::load_matrix_sync(Logits_mat,
                             s_logits[warp_from_offset] + k * WMMA_K,
                             max_seq_len + SKEW_HALF);
      wmma::load_matrix_sync(V_mat, s_value[k * WMMA_K] + warp_to_offset,
                             size_per_head + SKEW_HALF);
      wmma::mma_sync(QKV_mat, Logits_mat, V_mat, QKV_mat);
    }
    wmma::store_matrix_sync(s_query[warp_from_offset] + warp_to_offset, QKV_mat,
                            size_per_head + SKEW_HALF, wmma::mem_row_major);
  }

  __syncthreads();

  for (int from_id = warpId; from_id < seq_len; from_id += warpNums) {
    int pos =
        (bid * seq_len + from_id) * half_hidden_dim + head_offset + warp_tid;
    ((__half2 *)(attention_output))[pos] =
        ((__half2 *)(s_query[from_id]))[warp_tid];
  }
}

template <OperationType OpType_>
void Attention<OpType_>::fused_infer(
    const DataType_ *query, const DataType_ *key, const DataType_ *value,
    const DataType_ *atten_mask, DataType_ *attention_output, void *buf,
    const int batch_size, const int seq_len, hipblasHandle_t cublas_handle,
    hipStream_t stream) {
  dim3 grid(batch_size * head_num_);
  dim3 block;

  if (OpType_ == OperationType::HALF) {
    const half2 *query_ptr = (const half2 *)query;
    const half2 *query_bias_ptr = (const half2 *)param_.attr_bias_Q;
    const half2 *key_ptr = (const half2 *)key;
    const half2 *key_bias_ptr = (const half2 *)param_.attr_bias_K;
    const half2 *value_ptr = (const half2 *)value;
    const half2 *value_bias_ptr = (const half2 *)param_.attr_bias_V;
    half2 scaler;
    scaler.x = (__half)(0.125f), scaler.y = (__half)(0.125f);

    if (seq_len <= 16) {
      block.x = 128;
      wmma_attention_kernel_LE32<16, 64><<<grid, block, 0, stream>>>(
          query_ptr, query_bias_ptr, key_ptr, key_bias_ptr, value_ptr,
          value_bias_ptr, (__half *)atten_mask, (__half *)attention_output,
          batch_size, head_num_, seq_len, scaler);
    } else if (seq_len <= 32) {
      block.x = 256;
      wmma_attention_kernel_LE32<32, 64><<<grid, block, 0, stream>>>(
          query_ptr, query_bias_ptr, key_ptr, key_bias_ptr, value_ptr,
          value_bias_ptr, (__half *)atten_mask, (__half *)attention_output,
          batch_size, head_num_, seq_len, scaler);
    } else if (seq_len <= 48) {
      block.x = 384;
      wmma_attention_kernel<48, 64><<<grid, block, 0, stream>>>(
          query_ptr, query_bias_ptr, key_ptr, key_bias_ptr, value_ptr,
          value_bias_ptr, (__half *)atten_mask, (__half *)attention_output,
          batch_size, head_num_, seq_len, scaler);
    } else if (seq_len <= 64) {
      block.x = 512;
      wmma_attention_kernel<64, 64><<<grid, block, 0, stream>>>(
          query_ptr, query_bias_ptr, key_ptr, key_bias_ptr, value_ptr,
          value_bias_ptr, (__half *)atten_mask, (__half *)attention_output,
          batch_size, head_num_, seq_len, scaler);
    }
  }
}

template void Attention<OperationType::FP32>::fused_infer(
    const float *query, const float *key, const float *value,
    const float *atten_mask, float *attention_output, void *buf,
    const int batch_size, const int seq_len, hipblasHandle_t cublas_handle,
    hipStream_t stream);

template void Attention<OperationType::HALF>::fused_infer(
    const __half *query, const __half *key, const __half *value,
    const __half *atten_mask, __half *attention_output, void *buf,
    const int batch_size, const int seq_len, hipblasHandle_t cublas_handle,
    hipStream_t stream);
} // namespace fastertransformerv3
