#include "hip/hip_runtime.h"
/*
 * Author: Xiaoying Jia, Changyi Wan
 * Project: Faster Transformer Inference
 * Department: ByteDance Data-AML
 * Email: {jiaxiaoying, wanchangyi}@bytedance.com
 */
#include "fastertransformer_v3/includes/layernorm_kernels.h"
#include "fastertransformer_v3/includes/utils.h"
#include <hip/hip_fp16.h>

namespace fastertransformerv3
{

__global__
void input_layernorm(
                float *out, const float *input,
                const void *gamma, const void *beta, int m, int n, bool use_fp32)
{
    int tid = threadIdx.x;

    __shared__ float s_mean;
    __shared__ float s_variance;

    float local_out = __ldg(&input[blockIdx.x * n + tid]);

    float mean = blockReduceSum<float>(local_out);
    if(threadIdx.x == 0)
        s_mean = mean / n;
    __syncthreads();

    local_out -= s_mean;
    float variance = blockReduceSum<float>(local_out * local_out);
    if(threadIdx.x == 0)
        s_variance = rsqrtf(variance / n + 1e-6f);
    __syncthreads();

    for(int i = tid; i < n; i += blockDim.x)
        out[blockIdx.x * n + i] = local_out * s_variance * __ldg(&((float *)gamma)[i]) + __ldg(&((float *)beta)[i]);
}

__global__
void input_layernorm(
                __half *out, const __half *input,
                const void *gamma, const void *beta, int m, int n, bool use_fp32)
{
    int tid = threadIdx.x;
    __shared__ float s_mean;
    __shared__ float s_variance;

    half2 *out_ptr = (half2 *)out;
    const half2 *input_ptr = (const half2 *)input;

    int id = blockIdx.x * n / 2 + tid;
    float2 local_out_fp2 = __half22float2(__ldg(&input_ptr[id]));

    float local_out = local_out_fp2.x + local_out_fp2.y;

    float mean = blockReduceSum<float>(local_out);
    if(threadIdx.x == 0)
        s_mean = mean / n;
    __syncthreads();

    local_out_fp2.x -= s_mean;
    local_out_fp2.y -= s_mean;
    float variance = local_out_fp2.x * local_out_fp2.x + local_out_fp2.y * local_out_fp2.y;
    variance = blockReduceSum<float>(variance);
    if(threadIdx.x == 0)
        s_variance = rsqrtf(variance / n + 1e-6f);
    __syncthreads();

    float2 gamma_val, beta_val;
    if(use_fp32)
    {
        gamma_val = __ldg(&((const float2 *)gamma)[tid]);
        beta_val  = __ldg(&((const float2 *)beta)[tid]);
    }
    else
    {
        gamma_val = __half22float2(__ldg(&((const half2 *)gamma)[tid]));
        beta_val  = __half22float2(__ldg(&((const half2 *)beta)[tid]));
    }

    local_out_fp2.x = local_out_fp2.x * s_variance * gamma_val.x + beta_val.x;
    local_out_fp2.y = local_out_fp2.y * s_variance * gamma_val.y + beta_val.y;
    out_ptr[id] = __float22half2_rn(local_out_fp2);
}

__global__
void input_compress_layernorm(
                float *out, const float *input,
                const void *gamma, const void *beta, int m, int n, bool use_fp32,
                float *out2, const int *batch_idx, const int *word_idx)
{
    int tid = threadIdx.x;

    __shared__ float s_mean;
    __shared__ float s_variance;

    int offset = __ldg(&word_idx[blockIdx.x]);

    float local_out = __ldg(&input[offset * n + tid]);
    out[blockIdx.x * n + tid] = local_out;

    float mean = blockReduceSum<float>(local_out);
    if(threadIdx.x == 0)
        s_mean = mean / n;
    __syncthreads();

    local_out -= s_mean;
    float variance = blockReduceSum<float>(local_out * local_out);
    if(threadIdx.x == 0)
        s_variance = rsqrtf(variance / n + 1e-6f);
    __syncthreads();

    out2[blockIdx.x * n + tid] = local_out * s_variance * __ldg(&((float *)gamma)[tid]) + __ldg(&((float *)beta)[tid]);
}

__global__
void input_compress_layernorm(
                __half *out, const __half *input,
                const void *gamma, const void *beta, int m, int n, bool use_fp32,
                __half *out2, const int *batch_idx, const int *word_idx)
{
    int tid = threadIdx.x;
    __shared__ float s_mean;
    __shared__ float s_variance;

    const half2 *input_ptr = (const half2 *)input;

    int offset = __ldg(&word_idx[blockIdx.x]);

    int id = offset * n / 2 + tid;
    half2 temp = __ldg(&input_ptr[id]);
    ((half2 *)out)[blockIdx.x * n / 2 + tid] = temp;
    float2 local_out_fp2 = __half22float2(temp);

    float local_out = local_out_fp2.x + local_out_fp2.y;

    float mean = blockReduceSum<float>(local_out);
    if(threadIdx.x == 0)
        s_mean = mean / n;
    __syncthreads();

    local_out_fp2.x -= s_mean;
    local_out_fp2.y -= s_mean;
    float variance = local_out_fp2.x * local_out_fp2.x + local_out_fp2.y * local_out_fp2.y;
    variance = blockReduceSum<float>(variance);
    if(threadIdx.x == 0)
        s_variance = rsqrtf(variance / n + 1e-6f);
    __syncthreads();

    float2 gamma_val, beta_val;
    if(use_fp32)
    {
        gamma_val = __ldg(&((const float2 *)gamma)[tid]);
        beta_val  = __ldg(&((const float2 *)beta)[tid]);
    }
    else
    {
        gamma_val = __half22float2(__ldg(&((const half2 *)gamma)[tid]));
        beta_val  = __half22float2(__ldg(&((const half2 *)beta)[tid]));
    }

    local_out_fp2.x = local_out_fp2.x * s_variance * gamma_val.x + beta_val.x;
    local_out_fp2.y = local_out_fp2.y * s_variance * gamma_val.y + beta_val.y;

    ((half2 *)out2)[blockIdx.x * n / 2 + tid] = __float22half2_rn(local_out_fp2);
}

__global__
void add_bias_input_layernorm(
                float *out, const float *input, const float *bias,
                const void *gamma, const void *beta, int m, int n, bool use_fp32)
{
    int tid = threadIdx.x;
    int bid = blockIdx.x;

    __shared__ float s_mean;
    __shared__ float s_variance;

    float local_out = (float)(out[bid * n + tid] + __ldg(&input[bid * n + tid]) + __ldg(&bias[tid]));

    float mean = blockReduceSum<float>(local_out);
    if(threadIdx.x == 0)
        s_mean = mean / n;
    __syncthreads();

    local_out -= s_mean;
    float variance = blockReduceSum<float>(local_out * local_out);
    if(threadIdx.x == 0)
        s_variance = rsqrtf(variance / n + 1e-6f);
    __syncthreads();

    out[bid * n + tid] = local_out * s_variance * __ldg(&((float *)gamma)[tid]) + __ldg(&((float *)beta)[tid]);
}

__global__
void add_bias_input_layernorm(
                __half *out, const __half *input, const __half *bias,
                const void *gamma, const void *beta, int m, int n, bool use_fp32)
{
    int tid = threadIdx.x;
    __shared__ float s_mean;
    __shared__ float s_variance;

    half2 *out_ptr = (half2 *)out;
    const half2 *input_ptr = (const half2 *)input;
    const half2 *bias_ptr  = (const half2 *)bias;

    int id = blockIdx.x * n / 2 + tid;
    float2 local_out_fp2 = __half22float2(
                                           __hadd2(__hadd2(out_ptr[id], __ldg(&input_ptr[id])), __ldg(&bias_ptr[tid])));
    float local_out = local_out_fp2.x + local_out_fp2.y;

    float mean = blockReduceSum<float>(local_out);
    if(threadIdx.x == 0)
        s_mean = mean / n;
    __syncthreads();

    local_out_fp2.x -= s_mean;
    local_out_fp2.y -= s_mean;
    float variance = local_out_fp2.x * local_out_fp2.x + local_out_fp2.y * local_out_fp2.y;
    variance = blockReduceSum<float>(variance);
    if(threadIdx.x == 0)
        s_variance = rsqrtf(variance / n + 1e-6f);
    __syncthreads();

    float2 gamma_val, beta_val;
    if(use_fp32)
    {
        gamma_val = __ldg(&((const float2 *)gamma)[tid]);
        beta_val  = __ldg(&((const float2 *)beta)[tid]);
    }
    else
    {
        gamma_val = __half22float2(__ldg(&((const half2 *)gamma)[tid]));
        beta_val  = __half22float2(__ldg(&((const half2 *)beta)[tid]));
    }

    local_out_fp2.x = local_out_fp2.x * s_variance * gamma_val.x + beta_val.x;
    local_out_fp2.y = local_out_fp2.y * s_variance * gamma_val.y + beta_val.y;
    out_ptr[id] = __float22half2_rn(local_out_fp2);
}

__global__
void add_bias_input_layernorm_restore_output(
                const float *out, const float *input, const float *bias,
                const void *gamma, const void *beta, int m, int n, bool use_fp32,
                float *out2, const int *batch_idx, const int *word_idx)
{
    int tid = threadIdx.x;

    __shared__ float s_mean;
    __shared__ float s_variance;

    float local_out = (float)(out[blockIdx.x * n + tid] + __ldg(&input[blockIdx.x * n + tid]) + __ldg(&bias[tid]));

    float mean = blockReduceSum<float>(local_out);
    if(threadIdx.x == 0)
        s_mean = mean / n;
    __syncthreads();

    local_out -= s_mean;
    float variance = blockReduceSum<float>(local_out * local_out);
    if(threadIdx.x == 0)
        s_variance = rsqrtf(variance / n + 1e-6f);
    __syncthreads();

    int offset = __ldg(&word_idx[blockIdx.x]);
    out2[offset * n + tid] = local_out * s_variance * __ldg(&((float *)gamma)[tid]) + __ldg(&((float *)beta)[tid]);
}

__global__
void add_bias_input_layernorm_restore_output(
                const __half *out, const __half *input, const __half *bias,
                const void *gamma, const void *beta, int m, int n, bool use_fp32,
                __half *out2, const int *batch_idx, const int *word_idx)
{
    int tid = threadIdx.x;
    __shared__ float s_mean;
    __shared__ float s_variance;

    half2 *out_ptr = (half2 *)out;
    const half2 *input_ptr = (const half2 *)input;
    const half2 *bias_ptr  = (const half2 *)bias;

    int id = blockIdx.x * n / 2 + tid;
    float2 local_out_fp2 = __half22float2(
                                           __hadd2(__hadd2(out_ptr[id], __ldg(&input_ptr[id])), __ldg(&bias_ptr[tid])));
    float local_out = local_out_fp2.x + local_out_fp2.y;

    float mean = blockReduceSum<float>(local_out);
    if(threadIdx.x == 0)
        s_mean = mean / n;
    __syncthreads();

    local_out_fp2.x -= s_mean;
    local_out_fp2.y -= s_mean;
    float variance = local_out_fp2.x * local_out_fp2.x + local_out_fp2.y * local_out_fp2.y;
    variance = blockReduceSum<float>(variance);
    if(threadIdx.x == 0)
        s_variance = rsqrtf(variance / n + 1e-6f);
    __syncthreads();

    float2 gamma_val, beta_val;
    if(use_fp32)
    {
        gamma_val = __ldg(&((const float2 *)gamma)[tid]);
        beta_val  = __ldg(&((const float2 *)beta)[tid]);
    }
    else
    {
        gamma_val = __half22float2(__ldg(&((const half2 *)gamma)[tid]));
        beta_val  = __half22float2(__ldg(&((const half2 *)beta)[tid]));
    }

    local_out_fp2.x = local_out_fp2.x * s_variance * gamma_val.x + beta_val.x;
    local_out_fp2.y = local_out_fp2.y * s_variance * gamma_val.y + beta_val.y;

    int offset = __ldg(&word_idx[blockIdx.x]);
    ((half2 *)out2)[offset * n / 2 + tid] = __float22half2_rn(local_out_fp2);
}

}
