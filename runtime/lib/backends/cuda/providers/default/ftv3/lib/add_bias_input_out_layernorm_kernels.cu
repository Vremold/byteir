#include "hip/hip_runtime.h"
/*
 * Author: Xiaoying Jia, Changyi Wan
 * Project: Faster Transformer Inference
 * Department: ByteDance Data-AML
 * Email: {jiaxiaoying, wanchangyi}@bytedance.com
 */
#include "fastertransformer_v3/includes/add_bias_input_out_layernorm_kernels.h"
#include "fastertransformer_v3/includes/utils.h"
#include <hip/hip_fp16.h>

namespace fastertransformerv3
{

__global__
void add_bias_input_out_layernorm(
                float *out, const float *input, const float *bias, float *out2,
                const void *gamma, const void *beta, int m, int n, bool use_fp32)
{
    int tid = threadIdx.x;

    __shared__ float s_mean;
    __shared__ float s_variance;

    int offset = blockIdx.x * n + tid;
    float local_out = out[offset] + __ldg(&input[offset]) + __ldg(&bias[tid]);
    out2[offset] = local_out;

    float mean = blockReduceSum<float>(local_out);
    if(threadIdx.x == 0)
        s_mean = mean / n;
    __syncthreads();

    local_out -= s_mean;
    float variance = blockReduceSum<float>(local_out * local_out);
    if(threadIdx.x == 0)
        s_variance = rsqrtf(variance / n + 1e-6f);
    __syncthreads();

    out[blockIdx.x * n + tid] = local_out * s_variance * __ldg(&((float *)gamma)[tid]) + __ldg(&((float *)beta)[tid]);
}

__global__
void add_bias_input_out_layernorm(
                __half *out, const __half *input, const __half *bias, __half *out2,
                const void *gamma, const void *beta, int m, int n, bool use_fp32)
{
    int tid = threadIdx.x;
    __shared__ float s_mean;
    __shared__ float s_variance;

    half2 *out_ptr  = (half2 *)out;
    half2 *out2_ptr = (half2 *)out2;
    const half2 *input_ptr = (const half2 *)input;
    const half2 *bias_ptr  = (const half2 *)bias;

    int id = blockIdx.x * n / 2 + tid;
    half2 temp = __hadd2(__hadd2(out_ptr[id], __ldg(&input_ptr[id])), __ldg(&bias_ptr[tid]));
    out2_ptr[id] = temp;
    float2 local_out_fp2 = __half22float2(temp);
    float local_out = local_out_fp2.x + local_out_fp2.y;

    float mean = blockReduceSum<float>(local_out);
    if(threadIdx.x == 0)
        s_mean = mean / n;
    __syncthreads();

    local_out_fp2.x -= s_mean;
    local_out_fp2.y -= s_mean;
    float variance = local_out_fp2.x * local_out_fp2.x + local_out_fp2.y * local_out_fp2.y;
    variance = blockReduceSum<float>(variance);
    if(threadIdx.x == 0)
        s_variance = rsqrtf(variance / n + 1e-6f);
    __syncthreads();

    float2 gamma_val, beta_val;
    if(use_fp32)
    {
        gamma_val = __ldg(&((const float2 *)gamma)[tid]);
        beta_val  = __ldg(&((const float2 *)beta)[tid]);
    }
    else
    {
        gamma_val = __half22float2(__ldg(&((const half2 *)gamma)[tid]));
        beta_val  = __half22float2(__ldg(&((const half2 *)beta)[tid]));
    }

    local_out_fp2.x = local_out_fp2.x * s_variance * gamma_val.x + beta_val.x;
    local_out_fp2.y = local_out_fp2.y * s_variance * gamma_val.y + beta_val.y;
    out_ptr[id] = __float22half2_rn(local_out_fp2);
}
}