//===- cutlass_blas.cu ----------------------------------------*--- C++ -*-===//
//
// Copyright (c) ByteDance Inc. All rights reserved.
// Licensed under the Apache License, Version 2.0
//
//===----------------------------------------------------------------------===//

#include "cutlass/cutlass.h"
#pragma GCC diagnostic push
#pragma GCC diagnostic ignored "-Wstrict-aliasing"
#pragma GCC diagnostic ignored "-Wsign-compare"
#include "cutlass/gemm/device/gemm_batched.h"
#pragma GCC diagnostic pop
#include "cutlass/layout/matrix.h"

namespace brt {
namespace cuda {
namespace kernel {

// cutlass batch matmul implementation
template <typename T>
cutlass::Status
cutlass_batch_matmul(const T *A, int lda, long long int batch_stride_A,
                     const T *B, int ldb, long long int batch_stride_B, T *C,
                     int ldc, long long int batch_stride_C, int batch_count,
                     int m, int n, int k, T alpha, T beta,
                     hipStream_t stream = nullptr) {
  using Gemm = cutlass::gemm::device::GemmBatched<T, cutlass::layout::RowMajor,
                                                  T, cutlass::layout::RowMajor,
                                                  T, cutlass::layout::RowMajor>;
  Gemm gemm_op;
  return gemm_op({{m, n, k},
                  {A, lda},
                  batch_stride_A,
                  {B, ldb},
                  batch_stride_B,
                  {C, ldc},
                  batch_stride_C,
                  {C, ldc},
                  batch_stride_C,
                  {alpha, beta},
                  batch_count},
                 nullptr, stream);
}

// instantiate
template cutlass::Status
cutlass_batch_matmul<float>(const float *, int, long long int, const float *,
                            int, long long int, float *, int, long long int,
                            int, int, int, int, float, float, hipStream_t);

} // namespace kernel
} // namespace cuda
} // namespace brt
