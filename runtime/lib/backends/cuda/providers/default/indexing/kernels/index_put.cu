#include "hip/hip_runtime.h"
//===- index_put.cu -------------------------------------------*--- C++ -*-===//
//
// Copyright (c) ByteDance Inc. All rights reserved.
// Licensed under the Apache License, Version 2.0
//
//===----------------------------------------------------------------------===//

#include "brt/backends/cuda/device/common/cuda_call.h"
#include "index_put.h"
#include <algorithm>

namespace brt {
namespace cuda {
namespace kernel {

// a native_input_put putting entire inner_loop (feature dim) based outer_loop
// (embedding dim)
template <typename T, bool Accum>
__global__ void naive_index_put_kernel(T *inout, const int64_t *indices,
                                       const T *update,
                                       const int feature_bound) {
  int out_offset = indices[blockIdx.x];
  for (int idx = threadIdx.x; idx < feature_bound; idx += blockDim.x) {
    int in_idx = blockIdx.x * feature_bound + idx;
    int out_idx = out_offset * feature_bound + idx;
    T value = update[in_idx];
    if (Accum) {
      atomicAdd((T *)(inout + out_idx), value);
    } else {
      inout[out_idx] = value;
    }
  }
}

template <typename T, bool Accum>
void index_put(const T *input, const int64_t *indices, const T *update,
               T *output, const int index_count, const int feature_bound,
               const int size, hipStream_t stream) {
  BRT_CUDA_CHECK(hipMemcpyAsync(output, input, size * sizeof(T),
                                 hipMemcpyDeviceToDevice, stream));
  dim3 grid = index_count;
  dim3 block = std::min(256, feature_bound);
  naive_index_put_kernel<T, Accum>
      <<<grid, block, 0, stream>>>(output, indices, update, feature_bound);
}

template void index_put<float, true>(const float *, const int64_t *,
                                     const float *, float *, const int,
                                     const int, const int, hipStream_t);

} // namespace kernel
} // namespace cuda
} // namespace brt
