#include "hip/hip_runtime.h"
//===- index_select.cu ----------------------------------------*--- C++ -*-===//
//
// Copyright (c) ByteDance Inc. All rights reserved.
// Licensed under the Apache License, Version 2.0
//
//===----------------------------------------------------------------------===//

#include "./index_select.h"
#include <algorithm>

namespace brt {
namespace cuda {
namespace kernel {

template <typename T>
__global__ void naive_index_select_kernel(const T *input, const uint32_t *index,
                                          T *output, const int A, const int IB,
                                          const int OB, const int C) {
  for (int outIdx = blockIdx.x * blockDim.x + threadIdx.x; outIdx < A * OB * C;
       outIdx += gridDim.x * blockDim.x) {
    const int ind = outIdx / C % OB;
    const int inpIdx =
        outIdx / (OB * C) * (IB * C) + index[ind] * C + outIdx % C;
    output[outIdx] = input[inpIdx];
  }
}

template <typename T>
void index_select(const T *input, const uint32_t *index, T *output, const int A,
                  const int IB, const int OB, const int C,
                  hipStream_t stream) {
  dim3 grid = std::min(256, (A * OB * C + 63) / 64);
  dim3 block = std::min(64, A * OB * C);
  naive_index_select_kernel<<<grid, block, 0, stream>>>(input, index, output, A,
                                                        IB, OB, C);
}

template void index_select<float>(const float *, const uint32_t *, float *,
                                  const int, const int, const int, const int,
                                  hipStream_t);

} // namespace kernel
} // namespace cuda
} // namespace brt
